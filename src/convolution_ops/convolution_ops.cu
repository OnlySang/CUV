#include "hip/hip_runtime.h"
#include "convolution_ops.hpp"

#include <convert.hpp>
#include <nvmatrix.cuh>
#include <conv.cuh>
#include <conv_util.cuh>

namespace cuv{

template<>
	void convolve(dev_dense_matrix<float,row_major>& dst,
			  dev_dense_matrix<float,row_major>&   img,
			  dev_dense_matrix<float,row_major>&   filter){

	// some preliminary checks to ensure compatibility
	int numFilters = filter.h();
	int filterSize = sqrt(filter.w());
	int imgSize = sqrt(img.w());
	int dstSize = sqrt(dst.w()/numFilters);
	cuvAssert(numFilters%16 == 0);
	cuvAssert(filterSize*filterSize == filter.w());
	cuvAssert(imgSize*imgSize == img.w());
	cuvAssert(dstSize == imgSize - filterSize + 1);

	// make NVMatrices with this data
	NVMatrix nv_dst(dst.ptr(), dst.h(), dst.w(), false);
	NVMatrix nv_img(img.ptr(), img.h(), img.w(), false);
	NVMatrix nv_filter(filter.ptr(), filter.h(), filter.w(), false);

	// execute convolution
	convolve_bw(&nv_img, &nv_filter, &nv_dst);
	cuvSafeCall(hipDeviceSynchronize());
	}

/*
 * img		contains one input pattern in each row
 * filters	contains one filter in each row, number of filters must
 * 			be multiples of 16.
 * dst		holds the target images of the convolution. one row for each
 *			input image. width = dstSize^2 * numFilters
 */
template<>
void convolve(host_dense_matrix<float,row_major>& dst,
		  host_dense_matrix<float,row_major>&   img,
		  host_dense_matrix<float,row_major>&   filter) {

	int numImages = img.h();
	int numFilters = filter.h();

	int filterSize = sqrt(filter.w());
	int imgSize = sqrt(img.w());
	int dstSize = sqrt(dst.w()/numFilters);

	int dstPixels = dstSize * dstSize;

// Alex' host convolution is ~25% faster due to better memory access patterns
//	convCPU(img.ptr(), filter.ptr(), dst.ptr(), imgSize, filterSize, numImages, numFilters);

	for(int i=0; i<numImages; i++)
		for(int f=0; f<numFilters; f++)
			for(int r=0; r<dstSize; r++)
				for(int c=0; c<dstSize; c++) {
					float sum = 0.0f;
					for(int y=0; y<filterSize; y++) {
						float subsum = 0.0f;
						for(int x=0; x<filterSize; x++)
							subsum += img(i, (r+y)*imgSize + (c+x) ) *  filter(f, y * filterSize + x);
						sum += subsum;
					}
					sum += dst(i, f*dstPixels + r*dstSize + c);
					dst.set(i, f*dstPixels + r*dstSize + c, sum);
				}
}

template<>
void localMaximum(dev_dense_matrix<float,row_major>& dst,
		  dev_dense_matrix<float,row_major>&   img,
		  int poolSize) {
	int numImages = img.h();
	int imgPixels = img.w();
	int regionsPerImage = imgPixels / (poolSize * poolSize);
	int imgSize = sqrt(img.w());
	int dstSize = sqrt(dst.w());

	// some preliminary checks
	cuvAssert(imgSize*imgSize == img.w());
	cuvAssert(dstSize*dstSize == dst.w());
	cuvAssert(img.h() == dst.h());

	// make nvMatrices with this data
	NVMatrix nv_img(img.ptr(), numImages, imgPixels, false);
	NVMatrix nv_trans(numImages * regionsPerImage, poolSize * poolSize, false);
	NVMatrix nv_dst(dst.ptr(), numImages * regionsPerImage, 1, false);
	nv_trans.apply(NVMatrix::ZERO);
	nv_dst.apply(NVMatrix::ZERO);

	// transform and calculate maximum
	gridToMatrix(&nv_img, &nv_trans, poolSize, true);
	nv_trans.max(1, nv_dst);

	hipDeviceSynchronize();
}

template<>
void localMaximum(host_dense_matrix<float,row_major>& dst,
		  host_dense_matrix<float,row_major>&   img,
		  int poolSize) {
	int numImages = img.h();
	int imgPixels = img.w();
	int imgSize = sqrt(img.w());
	int dstSize = sqrt(dst.w());

	for(int i=0; i < img.h(); i++)
		for(int r=0; r<dstSize; r++)
			for(int c=0; c<dstSize; c++) {
				float maxi = -1000.f;
				for(int y=0; y<poolSize; y++)
					for(int x=0; x<poolSize; x++) {
						float val = img(i, (r*poolSize+y)*imgSize + c*poolSize+x);
						if(maxi < val)
							maxi = val;
					}
				dst.set(i, r*dstSize+c, maxi);
			}
}

}
