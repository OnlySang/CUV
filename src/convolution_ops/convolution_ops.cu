#include "hip/hip_runtime.h"
#include <float.h>

#include "convolution_ops.hpp"

#include <convert/convert.hpp>
#include <matrix_ops/matrix_ops.hpp>
#include <conv_util.cuh>
#include <random/random.hpp>
#include <nvmatrix.cuh>
#include <conv.cuh>
#include <conv2.cuh>
#include <conv3.cuh>
#include <conv_util.cuh>
#include <convCPU.h>

namespace cuv{

/* Convolve N patterns (images) with F filters, resulting in N*F target images
 *
 * img		contains one input pattern in each row
 * filters	contains one filter in each row, number of filters must
 * 			be multiples of 16.
 * dst		holds the target images of the convolution. one row for each
 *			input image. width = dstSize^2 * numFilters
 */
template<>
	void convolve(dev_dense_matrix<float,row_major>& dst,
			  dev_dense_matrix<float,row_major>&   img,
			  dev_dense_matrix<float,row_major>&   filter){

	// some preliminary checks to ensure compatibility
	int numFilters = filter.h();
	int filterSize = sqrt(filter.w());
	int imgSize = sqrt(img.w());
	int dstSize = sqrt(dst.w()/numFilters);
	cuvAssert(numFilters%16 == 0);
	cuvAssert(filterSize*filterSize == filter.w());
	cuvAssert(imgSize*imgSize == img.w());
	cuvAssert(dstSize == imgSize - filterSize + 1);

	// make NVMatrices with this data
	NVMatrix nv_dst(dst.ptr(), dst.h(), dst.w(), false);
	NVMatrix nv_img(img.ptr(), img.h(), img.w(), false);
	NVMatrix nv_filter(filter.ptr(), filter.h(), filter.w(), false);

	// execute convolution
	convolve_bw(&nv_img, &nv_filter, &nv_dst);
	cuvSafeCall(hipDeviceSynchronize());
	}

template<>
void convolve(host_dense_matrix<float,row_major>& dst,
		  host_dense_matrix<float,row_major>&   img,
		  host_dense_matrix<float,row_major>&   filter) {

	int numImages = img.h();
	int numFilters = filter.h();

	int filterSize = sqrt(filter.w());
	int imgSize = sqrt(img.w());
	int dstSize = sqrt(dst.w()/numFilters);

	int dstPixels = dstSize * dstSize;

	float* images = img.ptr();
	float* targets = dst.ptr();

	for(int i=0; i<numImages; i++) {
		float* filters = filter.ptr();
		for(int f=0; f<numFilters; f++) {
			for(int r=0; r<dstSize; r++)
				for(int c=0; c<dstSize; c++) {
					float sum = 0.0f;
					for(int y=0; y<filterSize; y++) {
						float subsum = 0.0f;
						for(int x=0; x<filterSize; x++)
							subsum += images[(r+y)*imgSize + (c+x)] * filters[y * filterSize + x];
						sum += subsum;
					}
					targets[f*dstPixels + r*dstSize + c] += sum;
				}
			filters += filter.w();
		}
		targets += dst.w();
		images += img.w();
	}
}

/* Convolve N patterns (images), each with a different set of F filters,
 * resulting in N*F target images
 *
 * img		contains one input pattern in each row
 * filters	contains F filters in each row, number of filters must
 * 			be multiples of 16.
 * dst		holds the target images of the convolution. one row for each
 *			input image. width = dstSize^2 * numFilters
 *
 * This routine can be used to compute the weight gradients: img contains the
 * activations from the lower layers filters are the error maps from the upper
 * layer. dst will then contain weight gradients for each pattern per row (sum
 * each column up).
 */
template<>
	void convolve2(dev_dense_matrix<float,row_major>& dst,
			  dev_dense_matrix<float,row_major>&   img,
			  dev_dense_matrix<float,row_major>&   filter,
			  int numFilters) {
	int imgSize = sqrt(img.w());
	int numImages = img.h();
	int filterSize = sqrt(filter.w()/numFilters);
	int dstSize = sqrt(dst.w()/numFilters);

	// some preliminary checks to ensure compatibility
	cuvAssert(numFilters%16 == 0);
	cuvAssert(filterSize*filterSize*numFilters == filter.w());
	cuvAssert(imgSize*imgSize == img.w());
	cuvAssert(dstSize == imgSize - filterSize + 1);

	// make NVMatrices with this data
	NVMatrix nv_dst(dst.ptr(), dst.h(), dst.w(), false);
	NVMatrix nv_img(img.ptr(), img.h(), img.w(), false);
	NVMatrix nv_filter(filter.ptr(), filter.h(), filter.w(), false);

	// execute convolution
    convolve2_bw(&nv_img, &nv_filter, &nv_dst, filterSize);
	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void convolve2(host_dense_matrix<float,row_major>& dst,
		  host_dense_matrix<float,row_major>&   img,
		  host_dense_matrix<float,row_major>&   filter,
		  int numFilters) {
	int imgSize = sqrt(img.w());
	int numImages = img.h();
	int filterSize = sqrt(filter.w()/numFilters);
	int dstSize = sqrt(dst.w()/numFilters);

	conv2CPU(img.ptr(), filter.ptr(), dst.ptr(), imgSize, filterSize, numImages, numFilters);
}

// images --> blocks
template<>
void grid_to_matrix(dev_dense_matrix<float,row_major>& mat,
		  dev_dense_matrix<float,row_major>&   grid,
		  int poolSize) {
	int numImages = grid.h();
	int imgPixels = grid.w();
	int regionsPerImage = imgPixels / (poolSize * poolSize);
	int imgSize = sqrt(grid.w());

	// some preliminary checks
	cuvAssert(imgSize*imgSize == grid.w());
	cuvAssert(mat.h() == numImages*regionsPerImage);
	cuvAssert(mat.w() == poolSize*poolSize);

	// make nvMatrices with this data
	NVMatrix nv_mat(mat.ptr(), mat.h(), mat.w(), false);
	NVMatrix nv_grid(grid.ptr(), grid.h(), grid.w(), false);
	fill(mat.vec(),0);

	gridToMatrix(&nv_grid, &nv_mat, poolSize, true);

	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void matrix_to_grid(dev_dense_matrix<float,row_major>& grid,
		  dev_dense_matrix<float,row_major>&   mat,
		  int poolSize) {
	int numImages = grid.h();
	int imgPixels = grid.w();
	int regionsPerImage = imgPixels / (poolSize * poolSize);
	int imgSize = sqrt(grid.w());

	// some preliminary checks
	cuvAssert(imgSize*imgSize == grid.w());
	cuvAssert(mat.h() == numImages*regionsPerImage);
	cuvAssert(mat.w() == poolSize*poolSize);

	// make nvMatrices with this data
	NVMatrix nv_mat(mat.ptr(), mat.h(), mat.w(), false);
	NVMatrix nv_grid(grid.ptr(), grid.h(), grid.w(), false);
	fill(grid.vec(),0);

	// transform and calculate maximum
	matrixToGrid(&nv_mat, &nv_grid, poolSize, true);

	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void sample_multinomial(dev_dense_matrix<float,row_major>& grid){
   /*dev_dense_matrix<float,row_major> tmp(grid.h(),grid.w());*/
   /*apply_binary_functor(tmp,grid,BF_COPY);*/

   dev_dense_matrix<float,row_major> rnd(grid.h(),1);
   fill_rnd_uniform(rnd.vec());

   NVMatrix nv_grid(grid.ptr(),grid.h(),grid.w(),false);
   NVMatrix nv_rnd(rnd.ptr(),rnd.h(),rnd.w(),false);
   /*NVMatrix nv_tmp(tmp.ptr(),tmp.h(),tmp.w(),false);*/

   /*sampleMultinomial(&nv_tmp,&nv_rnd,&nv_grid); */
   sampleMultinomial(&nv_grid,&nv_rnd,&nv_grid); 
   cuvSafeCall(hipDeviceSynchronize());
}

template<>
void prob_max_pooling(dev_vector<float>& sums,dev_dense_matrix<float,row_major>& grid, int poolSize, bool sample){
	int numImages = grid.h();
	int imgPixels = grid.w();
	int regionsPerImage = imgPixels / (poolSize * poolSize);

	dev_dense_matrix<float,row_major> mat(numImages*regionsPerImage, poolSize*poolSize);
	grid_to_matrix(mat,grid,poolSize);

	// normalize rows
	reduce_to_col(sums,mat);                    // sums      = sum(mat, axis=1)
	apply_scalar_functor(sums,SF_ADD,1.f);      // sums     += 1
	apply_scalar_functor(sums,SF_INV);          // sums      = 1/sums
	matrix_times_col(mat,sums);                 // mat[:,i] *= sums

	if(sample){
		sample_multinomial(mat);
		reduce_to_col(sums,mat);                // now is 0 or 1
	}else{
		/*apply_scalar_functor(sums,SF_SMAX);             // sums      = (sums-1)/sums*/
		reduce_to_col(sums,mat);                
	}
	matrix_to_grid(grid,mat,poolSize);
}

template<>
void prob_max_pooling(dev_dense_matrix<float,row_major>& grid, int poolSize, bool sample){
	int numImages = grid.h();
	int imgPixels = grid.w();
	int regionsPerImage = imgPixels / (poolSize * poolSize);

	dev_vector<float> sums(numImages*regionsPerImage);
	prob_max_pooling(sums, grid, poolSize,sample);
}


/* Convolve N patterns, each consisting of F images/maps with F filters and add
 * them up. Resulting in N target images
 *
 * img		contains F input pattern in each row
 * filters	contains one filter in each row, number of filters must
 * 			be multiples of 16.
 * dst		holds the target image of the convolution. one row for each
 *			input image. width = dstSize^2
 */

template<>
	void convolve3(dev_dense_matrix<float,row_major>& dst,
			  dev_dense_matrix<float,row_major>&   img,
			  dev_dense_matrix<float,row_major>&   filter) {

	int numFilters = filter.h();
	int smallSize = sqrt(img.w()/numFilters);
	int filterSize = sqrt(filter.w());
	int bigSize = sqrt(dst.w());
	int numImages = img.h();

	// make NVMatrices with this data
	NVMatrix nv_dst(dst.ptr(), dst.h(), dst.w(), false);
	NVMatrix nv_img(img.ptr(), img.h(), img.w(), false);
	NVMatrix nv_filter(filter.ptr(), filter.h(), filter.w(), false);

	// execute convolution
	convolve3_bw(&nv_img, &nv_filter, &nv_dst);
	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void convolve3(host_dense_matrix<float,row_major>& dst,
		  host_dense_matrix<float,row_major>&   img,
		  host_dense_matrix<float,row_major>&   filter) {
	// TODO
	printf("convolve3 NYI on host!\n");
}

#include <iostream>
using namespace std;

__global__
void reorder_kernel(float*dst, float* src, int len) {
	int tx = threadIdx.x, ix = blockIdx.x;
	int pt = blockIdx.y;

	if(tx >= len)
		return;
	while(tx < len) {
		dst[tx + pt * len + gridDim.y * len * ix] = src[tx + ix * len + pt * len * gridDim.x];
		tx += blockDim.x;
	}
}

/* sort the images in a matrix in a different order
 * input:  A1 B1 C1 D1
 *         A2 B2 C2 D2
 *         A3 B3 C3 D3
 * , where A1 is an image with blockLength pixels
 * output: A1
 *         A2
 *         A3
 *         B1
 *         B2
 *         ..
 */
template<>
void reorder(dev_dense_matrix<float,row_major>& M,
		  int blockLength) {
	int patternCount = M.h();
	int imgCount = M.w()/blockLength;

	float* temp;
	cuvSafeCall(hipMalloc( (void**) &temp, sizeof(float) * M.n() ));
	float* img_ptr = M.ptr();

	dim3 grid(imgCount, patternCount);
	dim3 threads(min(blockLength, 512));
	reorder_kernel<<<grid,threads>>>(temp, M.ptr(), blockLength);

	cuvSafeCall(hipDeviceSynchronize());

	cuvSafeCall(hipMemcpy(M.ptr(), temp, sizeof(float) * M.n(),hipMemcpyDeviceToDevice));
	M.resize(patternCount*imgCount, blockLength);
	cuvSafeCall(hipFree(temp));

	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void reorder(host_dense_matrix<float,row_major>& M,
		  int blockLength) {
	int patternCount = M.h();
	int imgCount = M.w()/blockLength;

	float* temp = (float*) malloc(sizeof(float) * M.n());
	float* tmp_ptr = temp;
	float* img_ptr = M.ptr();

	for(int p = 0; p < patternCount; p++) {
		for(int m = 0; m < imgCount; m++) {
			memcpy(	&tmp_ptr[blockLength * patternCount * m],
					img_ptr, sizeof(float)*blockLength);
			img_ptr += blockLength;
		}
		tmp_ptr += blockLength;
	}

	memcpy(M.ptr(), temp, sizeof(float) * M.n());
	M.resize(patternCount*imgCount, blockLength);
	free(temp);
}

__global__
void supersample_kernel(float*dst, float* src, int* indices, int len, int factor, int smallLen) {
	int tx = threadIdx.x; // ty = threadIdx.y;
	int bx = blockIdx.x, by = blockIdx.y;

	dst += by * len*len + tx * len * factor + bx * factor;
	indices += by * smallLen * smallLen + tx * smallLen + bx;
	src += by * smallLen * smallLen + tx * smallLen + bx;

	int idx = indices[0]; // coalesced???
	int row = idx % factor;
	int col = idx / factor;

	dst[row*len + col] = *src;
}

/*
 * Supersampling takes a n x (m*m) matrix img with n images of size (m x m)
 * and a factor s. Output is a n x (m*s*m*s) matrix dst with n enlarged images
 * of size (m*s x m*s)
 */
template<>
void supersample(dev_dense_matrix<float,row_major>& dst,
		dev_dense_matrix<float,row_major>& img,
		int factor,
		dev_dense_matrix<int,row_major>* indices) {
	int numImages = img.h();
	int imgPixels = img.w();
	int dstPixels = imgPixels * (factor * factor);
	int imgSize = sqrt(img.w());
	int dstSize = imgSize * factor;

	cuvAssert(dstSize / factor == imgSize);

	NVMatrix nv_img(img.ptr(), numImages, imgPixels, false);
	NVMatrix nv_dst(dst.ptr(), numImages, dstPixels, false);

	if(indices == NULL) {
		supersample(&nv_img, &nv_dst, factor);
	} else {
		assert(imgSize < 512);
		fill(dst, 0.0f);

		dim3 grid(imgSize, img.h());
		dim3 threads(min(imgSize, 512));
		supersample_kernel<<<grid,threads>>>(dst.ptr(), img.ptr(), indices->ptr(), dstSize, factor, imgSize);
	}

	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void supersample(host_dense_matrix<float,row_major>& dst,
		host_dense_matrix<float,row_major>& img,
		int factor,
		host_dense_matrix<int,row_major>* indices) {
	int numImages = img.h();
	int imgSize = sqrt(img.w());
	int dstSize = imgSize * factor;

	cuvAssert(dstSize / factor == imgSize);

	float* image = img.ptr();
	float* target = dst.ptr();

	if(indices != NULL) {
		for(int i = 0; i < numImages; i++) {
			for(int r = 0; r < imgSize; r++)
				for(int c = 0; c < imgSize; c++) {
					int idx = (indices->vec())[r*imgSize+c + i*imgSize*imgSize];
					int row = idx % factor;
					int col = idx / factor;
					target[(r*factor+row)*dstSize + c*factor+col] = image[r*imgSize + c];
				}
			target += dst.w();
			image += img.w();
		}
	} else {
		for(int i = 0; i < numImages; i++) {
			for(int r = 0; r < dstSize; r++)
				for(int c = 0; c < dstSize; c++) {
					target[0] = image[(r/factor)*imgSize+c/factor];
					target++;
				}
			image += img.w();
		}
	}

}

template<>
void super_to_max(dev_dense_matrix<float,row_major>& bigError,
		dev_dense_matrix<float,row_major>& smallError,
		dev_dense_matrix<float,row_major>& bigImg,
		dev_dense_matrix<float,row_major>& smallImg,
		int factor) {
	printf("Warning! superToMax() NYI!\n");
}

template<>
void super_to_max(host_dense_matrix<float,row_major>& bigError,
		host_dense_matrix<float,row_major>& smallError,
		host_dense_matrix<float,row_major>& bigImg,
		host_dense_matrix<float,row_major>& smallImg,
		int factor) {
	int numImages = smallImg.h();
	int imgSize = sqrt(smallImg.w());
	int dstSize = imgSize * factor;

	cuvAssert(dstSize / factor == imgSize);

	fill(bigError.vec(), 0.0f);

	float* be_ptr = bigError.ptr();
	float* se_ptr = smallError.ptr();
	float* bi_ptr = bigImg.ptr();
	float* si_ptr = smallImg.ptr();

	for(int i = 0; i < numImages; i++) {
		for(int r = 0; r < dstSize; r++)
			for(int c = 0; c < dstSize; c++) {
				float val = si_ptr[(r/factor)*imgSize+c/factor];
				if(val == bi_ptr[0])
					be_ptr[0] = se_ptr[(r/factor)*imgSize+c/factor];
				bi_ptr++;
				be_ptr++;
			}
		si_ptr += smallImg.w();
		se_ptr += smallImg.w();
	}
}

template<>
	void copy_into(dev_dense_matrix<float,row_major>& dst,
			  dev_dense_matrix<float,row_major>&   img,
			  int padding) {
	int inputSize = sqrt(img.w());
	int outputSize = sqrt(dst.w());
	cuvAssert(inputSize * inputSize == img.w());
	cuvAssert(outputSize * outputSize == dst.w());
	cuvAssert(inputSize + 2 * padding == outputSize);
	cuvAssert(img.h() == dst.h());

	// make NVMatrices with this data
	NVMatrix nv_dst(dst.ptr(), dst.h(), dst.w(), false);
	NVMatrix nv_img(img.ptr(), img.h(), img.w(), false);

	copyInto(&nv_img, &nv_dst, padding, false);
}

template<>
	void copy_into(host_dense_matrix<float,row_major>& dst,
			  host_dense_matrix<float,row_major>&   img,
			  int padding) {
	int inputSize = sqrt(img.w());
	int outputSize = sqrt(dst.w());
	cuvAssert(inputSize * inputSize == img.w());
	cuvAssert(outputSize * outputSize == dst.w());
	cuvAssert(inputSize + 2 * padding == outputSize);
	cuvAssert(img.h() == dst.h());

	float* img_ptr = img.ptr();
	float* dst_ptr = dst.ptr();
	for(int i=0; i<img.h(); i++) {
		dst_ptr += outputSize * padding;
		for(int j=0; j<inputSize;j++) {
			dst_ptr += padding;
			for(int k=0; k<inputSize;k++) {
				*dst_ptr++ = *img_ptr++;
			}
			dst_ptr += padding;
		}
		dst_ptr += outputSize * padding;
	}
}

template<>
	void max_pooling(host_dense_matrix<float,row_major>& dst,
			host_dense_matrix<float,row_major>& img,
			unsigned int poolSize,
			unsigned int overlap,
			host_dense_matrix<int,row_major>* indices) {
	cuvAssert(poolSize > overlap);
	int numImages = dst.h();
	cuvAssert(numImages == img.h());
	int imgSize = sqrt(img.w());
	cuvAssert(imgSize * imgSize == img.w());
	int stepSize = poolSize - overlap;
	int dstSize = (imgSize - poolSize)/stepSize + 1;
	cuvAssert(dstSize * dstSize == dst.w());
	cuvAssert((dstSize-1)*stepSize + poolSize == imgSize);

	float* img_ptr = img.ptr();
	float* dst_ptr = dst.ptr();

	for(int p=0; p<numImages; p++) {
		for(int r=0; r<dstSize; r++)
			for(int c=0; c<dstSize; c++) {
				int imax = 0;
				float cmax = -FLT_MAX;
				// loop through pool
				for(int i=0; i<poolSize; i++)
					for(int j=0; j<poolSize; j++) {
						int idx = c*stepSize+j + (r*stepSize+i)*imgSize;
						float val = img_ptr[idx];
						if(cmax < val) {
							cmax = val;
							imax = j*poolSize+i; // transpose due to dev local_maximum() function
						}
					}
				*dst_ptr++ = cmax;
				if(indices != NULL)
					indices->set(p, r*dstSize+c, imax);
			}

		img_ptr += imgSize * imgSize;
	}
}

// naive, but flexible implementation
// better distinguish between different cases and load image into shared memory
template<bool INDEX>
__global__
void max_pooling_kernel(float* dst, float* img, int* indices, int imgSize, int dstSize, int poolSize, int stepSize) {
	int tx = threadIdx.x; // ty = threadIdx.y;
	int bx = blockIdx.x, by = blockIdx.y;

	int p = tx + by * 256;
	if(p >= dstSize * dstSize)
		return;

	img += bx * imgSize * imgSize;

	float cmax = -FLT_MAX;
	int imax = 0;
	int column = p % dstSize;
	int row = p / dstSize;

	// loop through pool
	for(int i=0; i<poolSize; i++)
		for(int j=0; j<poolSize; j++) {
			int idx = column*stepSize+j + (row*stepSize+i)*imgSize;
			float val = img[idx];
			if(cmax < val) {
				cmax = val;
				if(INDEX)
					imax = j*poolSize+i; // transpose due to dev local_maximum() function
			}
		}

	// write result
	dst += bx * dstSize * dstSize + p;
	//	indices
	if(INDEX) {
		indices += bx * dstSize * dstSize + p;
		*indices = imax;
	}
	*dst = cmax;
}

/* This implementation only achieves a speedup of 5-10x, and is even
 * worse if the pools do not overlap. Better use local_maximum() in this
 * case.
 */

template<>
	void max_pooling(dev_dense_matrix<float,row_major>& dst,
			dev_dense_matrix<float,row_major>& img,
			unsigned int poolSize,
			unsigned int overlap,
			dev_dense_matrix<int,row_major>* indices) {

	cuvAssert(poolSize > overlap);
	int numImages = dst.h();
	cuvAssert(numImages == img.h());
	int imgSize = sqrt(img.w());
	cuvAssert(imgSize * imgSize == img.w());
	int stepSize = poolSize - overlap;
	int dstSize = (imgSize - poolSize)/stepSize + 1;
	cuvAssert(dstSize * dstSize == dst.w());
	cuvAssert((dstSize-1)*stepSize + poolSize == imgSize);

	int numThreads = 256;
	int numBlocksX = numImages;
	int numBlocksY = ceil((float) (dstSize * dstSize)/numThreads);

	dim3 grid(numBlocksX, numBlocksY);
	dim3 threads(numThreads);
	if(indices==NULL)
		max_pooling_kernel<false><<<grid,threads>>>(dst.ptr(), img.ptr(), NULL, imgSize, dstSize, poolSize, stepSize);
	else
		max_pooling_kernel<true><<<grid,threads>>>(dst.ptr(), img.ptr(), indices->ptr(), imgSize, dstSize, poolSize, stepSize);

	cuvSafeCall(hipDeviceSynchronize());
}


}

