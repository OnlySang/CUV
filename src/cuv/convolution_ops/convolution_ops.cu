#include "hip/hip_runtime.h"
//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*





/** 
 * @file convolution_ops.cu
 * @brief Operations used for convolution and max-pooling
 * @ingroup convolution
 * @date 2010-03-21
 */
#include <float.h>

#include <cuv/convolution_ops/convolution_ops.hpp>

#include <cuv/convert/convert.hpp>
#include <cuv/matrix_ops/matrix_ops.hpp>
#include <cuv/tensor_ops/tensor_ops.hpp>
#include <cuv/random/random.hpp>
#include <3rd_party/CudaConv/conv_util.cuh>
#include <3rd_party/CudaConv/nvmatrix.cuh>
#include <3rd_party/CudaConv/conv.cuh>
#include <3rd_party/CudaConv/conv2.cuh>
#include <3rd_party/CudaConv/conv3.cuh>
#include <3rd_party/CudaConv/conv_util.cuh>
#include <3rd_party/CudaConv/convCPU.h>
#include <iostream>
using namespace std;
#ifdef __CDT_PARSER__
#define __shared__
#define __global__
#endif

namespace cuv{

template<>
	void convolve(tensor<float,dev_memory_space,row_major>& dst,
			  tensor<float,dev_memory_space,row_major>&   img,
			  tensor<float,dev_memory_space,row_major>&   filter,
			  int numGroups){

	// some preliminary checks to ensure compatibility
	int numFilters = filter.shape()[0] / numGroups;
	cuvAssert(filter.shape()[0] == numFilters*numGroups);
	int numImages = img.shape()[0] / numGroups;
	cuvAssert(img.shape()[0] == numImages*numGroups);
	int filterSize = sqrt(filter.shape()[1]);
	int imgSize = sqrt(img.shape()[1]);
	int dstSize = sqrt(dst.shape()[1]/numImages);
//	printf("imgSize = %i, dstSize = %i, filterSize = %i\n", imgSize, dstSize, filterSize);
//	printf("(%ix%i) x (%ix%i) = (%ix%i)\n", img.shape()[0], img.shape()[1], filter.shape()[0], filter.shape()[1], dst.shape()[0], dst.shape()[1]);
	cuvAssert(numFilters%2 == 0);
	cuvAssert(filterSize*filterSize == filter.shape()[1]);
	cuvAssert(imgSize*imgSize == img.shape()[1]);
	cuvAssert(dstSize == imgSize - filterSize + 1);

	// make NVMatrices with this data
	NVMatrix nv_dst(dst.ptr(), dst.shape()[0], dst.shape()[1], false);
	NVMatrix nv_img(img.ptr(), img.shape()[0], img.shape()[1], false);
	NVMatrix nv_filter(filter.ptr(), filter.shape()[0], filter.shape()[1], false);

	// execute convolution
	convolve(&nv_img, &nv_filter, &nv_dst, numGroups, false);
	cuvSafeCall(hipDeviceSynchronize());
	}

template<>
void convolve(tensor<float,host_memory_space,row_major>& dst,
		  tensor<float,host_memory_space,row_major>&   img,
		  tensor<float,host_memory_space,row_major>&   filter,
		  int numGroups) {

	int numImages = img.shape()[0] / numGroups;
	int numFilters = filter.shape()[0] / numGroups;

	int filterSize = sqrt(filter.shape()[1]);
	int imgSize = sqrt(img.shape()[1]);
	int dstSize = sqrt(dst.shape()[1]/numImages);

	int dstPixels = dstSize * dstSize;

	float* targets = dst.ptr();

	for(int g=0; g<numGroups; g++) {
		float* filters = filter.ptr() + g*numGroups*filterSize;
		for(int f=0; f<numFilters; f++) {
			float* images = img.ptr();
			for(int i=0; i<numImages; i++) {
				for(int r=0; r<dstSize; r++)
					for(int c=0; c<dstSize; c++) {
						float sum = 0.0f;
						for(int y=0; y<filterSize; y++) {
							float subsum = 0.0f;
							for(int x=0; x<filterSize; x++)
								subsum += images[(r+y)*imgSize + (c+x)] * filters[y * filterSize + x];
							sum += subsum;
						}
						targets[i*dstPixels + r*dstSize + c] += sum;
					}
				images += img.shape()[1];
			}
			targets += dst.shape()[1];
			filters += filter.shape()[1];
		}
	}
}


template<>
	void convolve2(tensor<float,dev_memory_space,row_major>& dst,
			  tensor<float,dev_memory_space,row_major>&   img,
			  tensor<float,dev_memory_space,row_major>&   filter,
			  int numFilters,
			  int numGroups) {
	int imgSize = sqrt(img.shape()[1]);
	int numImages = img.shape()[0] / numGroups;
	int filterSize = sqrt(filter.shape()[1]/numImages);
	int dstSize = sqrt(dst.shape()[1]/numFilters);

	// some preliminary checks to ensure compatibility
	cuvAssert(filter.shape()[0] == numFilters*numGroups);
	cuvAssert(numFilters%2 == 0);
	cuvAssert(numImages*filterSize*filterSize == filter.shape()[1]);
	cuvAssert(imgSize*imgSize == img.shape()[1]);
	if (!(dstSize == (imgSize - filterSize + 1)))
		std::cout << "destSize should be " << imgSize - filterSize + 1 << " but is " << dstSize;
	cuvAssert(dstSize == imgSize - filterSize + 1);

	// make NVMatrices with this data
	NVMatrix nv_dst(dst.ptr(), dst.shape()[0], dst.shape()[1], false);
	NVMatrix nv_img(img.ptr(), img.shape()[0], img.shape()[1], false);
	NVMatrix nv_filter(filter.ptr(), filter.shape()[0], filter.shape()[1], false);

	// execute convolution
    convolve2(&nv_img, &nv_filter, &nv_dst, filterSize, numGroups, false);
	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void convolve2(tensor<float,host_memory_space,row_major>& dst,
		  tensor<float,host_memory_space,row_major>&   img,
		  tensor<float,host_memory_space,row_major>&   filter,
		  int numFilters,
		  int numGroups) {
	int imgSize = sqrt(img.shape()[1]);
	int numImages = img.shape()[0];
	int filterSize = sqrt(filter.shape()[1]/numImages);
	int dstSize = sqrt(dst.shape()[1]/numFilters);

	conv2CPU(img.ptr(), filter.ptr(), dst.ptr(), imgSize, filterSize, numImages, numFilters, numGroups);
}

// images --> blocks
template<>
void grid_to_matrix(tensor<float,dev_memory_space,row_major>& mat,
		  tensor<float,dev_memory_space,row_major>&   grid,
		  int poolSize) {
	int numImages = grid.shape()[0];
	int imgPixels = grid.shape()[1];
	int regionsPerImage = imgPixels / (poolSize * poolSize);
	int imgSize = sqrt(grid.shape()[1]);

	// some preliminary checks
	cuvAssert(imgSize*imgSize == grid.shape()[1]);
	cuvAssert(mat.shape()[0] == numImages*regionsPerImage);
	cuvAssert(mat.shape()[1] == poolSize*poolSize);

	// make nvMatrices with this data
	NVMatrix nv_mat(mat.ptr(), mat.shape()[0], mat.shape()[1], false);
	NVMatrix nv_grid(grid.ptr(), grid.shape()[0], grid.shape()[1], false);
	fill(mat,0);

	gridToMatrix(&nv_grid, &nv_mat, poolSize, true);

	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void matrix_to_grid(tensor<float,dev_memory_space,row_major>& grid,
		  tensor<float,dev_memory_space,row_major>&   mat,
		  int poolSize) {
	int numImages = grid.shape()[0];
	int imgPixels = grid.shape()[1];
	int regionsPerImage = imgPixels / (poolSize * poolSize);
	int imgSize = sqrt(grid.shape()[1]);

	// some preliminary checks
	cuvAssert(imgSize*imgSize == grid.shape()[1]);
	cuvAssert(mat.shape()[0] == numImages*regionsPerImage);
	cuvAssert(mat.shape()[1] == poolSize*poolSize);

	// make nvMatrices with this data
	NVMatrix nv_mat(mat.ptr(), mat.shape()[0], mat.shape()[1], false);
	NVMatrix nv_grid(grid.ptr(), grid.shape()[0], grid.shape()[1], false);
	fill(grid,0);

	// transform and calculate maximum
	matrixToGrid(&nv_mat, &nv_grid, poolSize, true);

	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void sample_multinomial(tensor<float,dev_memory_space,row_major>& grid){
   /*dense_matrix<float,dev_memory_space,row_major> tmp(grid.shape()[0],grid.shape()[1]);*/
   /*apply_binary_functor(tmp,grid,BF_COPY);*/

   tensor<float,dev_memory_space,row_major> rnd(grid.shape()[0]);
   fill_rnd_uniform(rnd);

   NVMatrix nv_grid(grid.ptr(),grid.shape()[0],grid.shape()[1],false);
   NVMatrix nv_rnd(rnd.ptr(),rnd.shape()[0],rnd.shape()[1],false);
   /*NVMatrix nv_tmp(tmp.ptr(),tmp.shape()[0],tmp.shape()[1],false);*/

   /*sampleMultinomial(&nv_tmp,&nv_rnd,&nv_grid); */
   sampleMultinomial(&nv_grid,&nv_rnd,&nv_grid); 
   cuvSafeCall(hipDeviceSynchronize());
}

template<>
void prob_max_pooling(tensor<float,dev_memory_space>& sums,tensor<float,dev_memory_space,row_major>& grid, int poolSize, bool sample){
	int numImages = grid.shape()[0];
	int imgPixels = grid.shape()[1];
	int regionsPerImage = imgPixels / (poolSize * poolSize);

	tensor<float,dev_memory_space,row_major> mat(extents[numImages*regionsPerImage][poolSize*poolSize]);
	grid_to_matrix(mat,grid,poolSize);

	// normalize rows
	reduce_to_col(sums,mat);                    // sums      = sum(mat, axis=1)
	apply_scalar_functor(sums,SF_ADD,1.f);      // sums     += 1
	apply_scalar_functor(sums,SF_INV);          // sums      = 1/sums
	matrix_times_col(mat,sums);                 // mat[:,i] *= sums

	if(sample){
		sample_multinomial(mat);
		reduce_to_col(sums,mat);                // now is 0 or 1
	}else{
		/*apply_scalar_functor(sums,SF_SMAX);             // sums      = (sums-1)/sums*/
		reduce_to_col(sums,mat);                
	}
	matrix_to_grid(grid,mat,poolSize);
}

template<>
void prob_max_pooling(tensor<float,dev_memory_space,row_major>& grid, int poolSize, bool sample){
	int numImages = grid.shape()[0];
	int imgPixels = grid.shape()[1];
	int regionsPerImage = imgPixels / (poolSize * poolSize);

	tensor<float,dev_memory_space> sums(numImages*regionsPerImage);
	prob_max_pooling(sums, grid, poolSize,sample);
}


template<>
	void convolve3(tensor<float,dev_memory_space,row_major>& dst,
			  tensor<float,dev_memory_space,row_major>&   img,
			  tensor<float,dev_memory_space,row_major>&   filter,
			  int numGroups) {

	int numFilters = filter.shape()[0];
	int smallSize = sqrt(img.shape()[1]/numFilters);
	int filterSize = sqrt(filter.shape()[1]);
	int bigSize = sqrt(dst.shape()[1]);
	int numImages = img.shape()[0];

	// make NVMatrices with this data
	NVMatrix nv_dst(dst.ptr(), dst.shape()[0], dst.shape()[1], false);
	NVMatrix nv_img(img.ptr(), img.shape()[0], img.shape()[1], false);
	NVMatrix nv_filter(filter.ptr(), filter.shape()[0], filter.shape()[1], false);

	// execute convolution
	convolve3(&nv_img, &nv_filter, &nv_dst, numGroups, false);
	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void convolve3(tensor<float,host_memory_space,row_major>& dst,
		  tensor<float,host_memory_space,row_major>&   img,
		  tensor<float,host_memory_space,row_major>&   filter,
		  int numGroups) {
	// TODO
	printf("convolve3 NYI on host!\n");
}

__global__
void reorder_kernel(float*dst, float* src, int len) {
	int tx = threadIdx.x, ix = blockIdx.x;
	int pt = blockIdx.y;

	if(tx >= len)
		return;
	while(tx < len) {
		dst[tx + pt * len + gridDim.y * len * ix] = src[tx + ix * len + pt * len * gridDim.x];
		tx += blockDim.x;
	}
}

template<class V>
void reorder_impl(tensor<V,dev_memory_space,row_major>& dst,
				  tensor<V,dev_memory_space,row_major>& src,
		  		  int blockLength) {
	int patternCount = src.shape()[0];
	int imgCount = src.shape()[1]/blockLength;

	dim3 grid(imgCount, patternCount);
	dim3 threads(min(blockLength, 512));
	reorder_kernel<<<grid,threads>>>(dst.ptr(), src.ptr(), blockLength);

	cuvSafeCall(hipDeviceSynchronize());

	dst.reshape(extents[patternCount*imgCount][blockLength]);

	cuvSafeCall(hipDeviceSynchronize());
}

template<class V>
void reorder_impl(tensor<V,host_memory_space,row_major>& dst,
		tensor<V,host_memory_space,row_major>& src,
		int blockLength) {
	int patternCount = src.shape()[0];
	int imgCount = src.shape()[1]/blockLength;

	float* dst_ptr = dst.ptr();
	float* src_ptr = src.ptr();

	for(int p = 0; p < patternCount; p++) {
		for(int m = 0; m < imgCount; m++) {
			memcpy(	&dst_ptr[blockLength * patternCount * m],
					src_ptr, sizeof(float)*blockLength);
			src_ptr += blockLength;
		}
		dst_ptr += blockLength;
	}

	dst.reshape(extents[patternCount*imgCount][blockLength]);
}

template<class V, class M, class T>
void reorder(tensor<V,M,T>& m,
		int blockLength) {
	// create temporary destination matrix
	tensor<V,M,T> tmp(m.shape());

	// perform reorder
	reorder_impl(tmp, m, blockLength);

	// change pointer to temp matrix / copy
        m = tmp;
}

template<class V, class M, class T>
void reorder(tensor<V,M,T>& dst,
		tensor<V,M,T>& src,
		int blockLength) {
	reorder_impl(dst, src, blockLength);
}

#define REORDER_INSTANTIATE(V) \
	template void reorder( tensor<V,host_memory_space,row_major>&, int); \
	template void reorder( tensor<V,host_memory_space,row_major>&, tensor<V,host_memory_space,row_major>&, int); \
	template void reorder( tensor<V,dev_memory_space,row_major>&, int); \
	template void reorder( tensor<V,dev_memory_space,row_major>&, tensor<V,dev_memory_space,row_major>&, int);

REORDER_INSTANTIATE(float);

template<>
	void subsample(tensor<float,dev_memory_space,row_major>& dst,
			  tensor<float,dev_memory_space,row_major>&   img,
			  int factor,
			  bool avoidBankConflicts) {
	// make NVMatrices with this data
	NVMatrix nv_dst(dst.ptr(), dst.shape()[0], dst.shape()[1], false);
	NVMatrix nv_img(img.ptr(), img.shape()[0], img.shape()[1], false);

	if (dst.shape()[1]*dst.shape()[0] != img.shape()[1]* img.shape()[0] / (factor*factor)){
		std::cout << dst.shape()[1] << "*" << dst.shape()[0] << "==" << img.shape()[1] << "*" << img.shape()[0] << "/" << factor << "*" << factor << "==" << dst.shape()[1]*dst.shape()[0] << "!=" << img.shape()[1]* img.shape()[0] / (factor*factor);
	}

	cuvAssert(dst.shape()[1]*dst.shape()[0] == img.shape()[1]* img.shape()[0] / (factor*factor));
	// execute convolution
    subsample(&nv_img, &nv_dst, factor, avoidBankConflicts);
	cuvSafeCall(hipDeviceSynchronize());
}

template<>
	void subsample(tensor<float,host_memory_space,row_major>& dst,
			  tensor<float,host_memory_space,row_major>&   img,
			  int factor,
			  bool avoidBankConflicts) {
	int imgSize = sqrt(img.shape()[1]);
	int numImg = img.shape()[0];

	// execute convolution
    subsampleCPU(img.ptr(), dst.ptr(), imgSize, factor, numImg);
	cuvSafeCall(hipDeviceSynchronize());
}


__global__
void supersample_kernel(float*dst, float* src, int* indices, int len, int factor, int smallLen) {
	int tx = threadIdx.x; // ty = threadIdx.y;
	int bx = blockIdx.x, by = blockIdx.y;

	dst += by * len*len + tx * len * factor + bx * factor;
	indices += by * smallLen * smallLen + tx * smallLen + bx;
	src += by * smallLen * smallLen + tx * smallLen + bx;

	int idx = indices[0]; // coalesced???
	int row = idx % factor;
	int col = idx / factor;

	dst[row*len + col] = *src;
}

template<>
void supersample(tensor<float,dev_memory_space,row_major>& dst,
		tensor<float,dev_memory_space,row_major>& img,
		int factor,
		tensor<int,dev_memory_space,row_major>* indices) {
	int numImages = img.shape()[0];
	int imgPixels = img.shape()[1];
	int dstPixels = imgPixels * (factor * factor);
	int imgSize = sqrt(img.shape()[1]);
	int dstSize = imgSize * factor;
	
	cuvAssert(dstSize / factor == imgSize);
	
	cuvAssert(img.shape()[1]  *factor * factor == dst.shape()[1]);
	cuvAssert(img.shape()[0]== dst.shape()[0]);

	NVMatrix nv_img(img.ptr(), numImages, imgPixels, false);
	NVMatrix nv_dst(dst.ptr(), numImages, dstPixels, false);

	if(indices == NULL) {
		supersample(&nv_img, &nv_dst, factor);
	} else {
		assert(imgSize < 512);
		fill(dst, 0.0f);

		dim3 grid(imgSize, img.shape()[0]);
		dim3 threads(min(imgSize, 512));
		supersample_kernel<<<grid,threads>>>(dst.ptr(), img.ptr(), indices->ptr(), dstSize, factor, imgSize);
	}

	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void supersample(tensor<float,host_memory_space,row_major>& dst,
		tensor<float,host_memory_space,row_major>& img,
		int factor,
		tensor<int,host_memory_space,row_major>* indices) {
	int numImages = img.shape()[0];
	int imgSize = sqrt(img.shape()[1]);
	int dstSize = imgSize * factor;

	cuvAssert(img.shape()[1]  *factor * factor == dst.shape()[1]);
	cuvAssert(img.shape()[0]== dst.shape()[0]);
	cuvAssert(dstSize / factor == imgSize);

	float* image = img.ptr();
	float* target = dst.ptr();

	if(indices != NULL) {
		for(int i = 0; i < numImages; i++) {
			for(int r = 0; r < imgSize; r++)
				for(int c = 0; c < imgSize; c++) {
					int idx = (*indices)[r*imgSize+c + i*imgSize*imgSize];
					int row = idx % factor;
					int col = idx / factor;
					target[(r*factor+row)*dstSize + c*factor+col] = image[r*imgSize + c];
				}
			target += dst.shape()[1];
			image += img.shape()[1];
		}
	} else {
		for(int i = 0; i < numImages; i++) {
			for(int r = 0; r < dstSize; r++)
				for(int c = 0; c < dstSize; c++) {
					target[0] = image[(r/factor)*imgSize+c/factor];
					target++;
				}
			image += img.shape()[1];
		}
	}

}

// arbitrary value can be anything <= 64 KB
#define CONST_SIZE 8192
__device__ __constant__ float c_filter[CONST_SIZE];

template<bool FILTER>
__global__
void super_to_max_kernel(float*dst, float* src, int* indices, int imgSize, int dstSize, int poolSize, int stepSize, int patchSize, int numPatches, int batch) {
	int tx = threadIdx.x; // ty = threadIdx.y;
	int bx = blockIdx.x;
	
	int patch = tx + batch * 256;

	if(patch >= numPatches * numPatches)
		return;

	int c = patch % (numPatches);
	int r = patch / (numPatches);

	dst += bx * dstSize * dstSize + c * patchSize * stepSize + r * patchSize * stepSize * dstSize;
	src += bx * imgSize * imgSize + c * patchSize + r * patchSize * imgSize;
	indices += bx * imgSize * imgSize + c * patchSize + r * patchSize * imgSize;

	for(int i=0; i<patchSize; i++) {
		for(int j=0; j<patchSize; j++) {
			if(c*patchSize+j < imgSize && r*patchSize+i < imgSize) {
				int idx = indices[0];
				int row = idx % poolSize;
				int col = idx / poolSize;
				float val = src[0];
				if(FILTER)
					val *= (float) c_filter[row*poolSize+col];
				dst[col + row*dstSize] += val;
			}

			dst += stepSize;
			src++;
			indices++;
			syncthreads();
		}
		dst += dstSize * stepSize - patchSize * stepSize;
		src += imgSize - patchSize;
		indices += imgSize - patchSize;
	}
}

template<>
void super_to_max(tensor<float,dev_memory_space,row_major>& dst,
		tensor<float,dev_memory_space,row_major>& img,
		int poolSize,
		int overlap,
		tensor<int,dev_memory_space,row_major>* indices,
		tensor<float,dev_memory_space,row_major>* filter) {
	cuvAssert(indices->shape()[1] == img.shape()[1]);
	cuvAssert(indices->shape()[0] == img.shape()[0]);
	cuvAssert(poolSize > overlap);
	int numImages = dst.shape()[0];
	cuvAssert(numImages == img.shape()[0]);
	int imgSize = sqrt(img.shape()[1]);
	if(imgSize * imgSize != img.shape()[1]){
			cout << std::endl<<"Error: imgSize x imgSize (" <<imgSize<<")²="<< imgSize*imgSize<<" should be img.w = "<<img.shape()[1]<<std::endl;
		}
	cuvAssert(imgSize * imgSize == img.shape()[1]);
	int stepSize = poolSize - overlap;
	int dstSize = (imgSize - 1) * stepSize + poolSize;
	if(dstSize * dstSize != dst.shape()[1]){
				cout << std::endl<<"Error: dstSize x dstSize (" <<dstSize<<")²="<< dstSize*dstSize<<" should be dst.w = "<<dst.shape()[1]<<std::endl;
			}
	cuvAssert(dstSize * dstSize == dst.shape()[1]);
	cuvAssert((dstSize-poolSize)/stepSize + 1 == imgSize);

	// we have to split the small image into disjoint "patches", in order to
	// avoid that data from the same patch is written to identical positions
	int patchSize = (int) ceil(((float) poolSize)/stepSize);
	int numPatches = ceil((float) imgSize / patchSize);

	if(indices == NULL) {
		printf("super_to_max() NYI without indices\n");
		return;
	}

	if(filter!=NULL) {
		cuvAssert(filter->shape()[1] == poolSize);
		cuvAssert(filter->shape()[0] == poolSize);
		cuvAssert(sizeof(float) * filter->size() <= CONST_SIZE);
		cuvSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_filter), filter->ptr(), sizeof(float) * filter->size(), 0, hipMemcpyDeviceToDevice) );
	}

	fill(dst, 0.0f);

	int numThreads = 256;
	int numBlocks = numImages;
	int numBatches = ceil((float) (numPatches * numPatches)/numThreads); // can't spread this to multiple blocks due to overlapping borders. loop instead

	for(int b = 0; b < numBatches; b++) {
		dim3 grid(numBlocks);
		dim3 threads(numThreads);
		if(filter==NULL)
			super_to_max_kernel<false><<<grid,threads>>>(dst.ptr(), img.ptr(), indices->ptr(), imgSize, dstSize, poolSize, stepSize, patchSize, numPatches, b);
		else
			super_to_max_kernel<true><<<grid,threads>>>(dst.ptr(), img.ptr(), indices->ptr(), imgSize, dstSize, poolSize, stepSize, patchSize, numPatches, b);
		cuvSafeCall(hipDeviceSynchronize());
	}
}

template<>
void super_to_max(tensor<float,host_memory_space,row_major>& dst,
		tensor<float,host_memory_space,row_major>& img,
		int poolSize,
		int overlap,
		tensor<int,host_memory_space,row_major>* indices,
		tensor<float,host_memory_space,row_major>* filter) {
	cuvAssert(poolSize > overlap);
	int numImages = dst.shape()[0];
	cuvAssert(numImages == img.shape()[0]);
	int imgSize = sqrt(img.shape()[1]);
	cuvAssert(imgSize * imgSize == img.shape()[1]);
	int stepSize = poolSize - overlap;
	int dstSize = (imgSize - 1) * stepSize + poolSize;
	cuvAssert(dstSize * dstSize == dst.shape()[1]);
	cuvAssert((dstSize-poolSize)/stepSize + 1 == imgSize);

	fill(dst, 0.0f);

	float* img_ptr = img.ptr();
	int* idx_ptr = indices->ptr();
	float* dst_ptr = dst.ptr();

	for(int i = 0; i < numImages; i++) {
		for(int r = 0; r < imgSize; r++) {
			for(int c = 0; c < imgSize; c++) {
				int idx = *idx_ptr;
				int row = idx % poolSize;
				int col = idx / poolSize;
				float val = *img_ptr;
				if(filter != NULL)
					val *= (float) (filter->ptr())[row*poolSize+col];
				dst_ptr[col + row * dstSize] += val;
				img_ptr++;
				idx_ptr++;
				dst_ptr += stepSize;
			}
			dst_ptr += overlap + (stepSize - 1) * dstSize;
		}
		dst_ptr += overlap * dstSize;
	}
}

template<>
	void copy_into(tensor<float,dev_memory_space,row_major>& dst,
			  tensor<float,dev_memory_space,row_major>&   img,
			  int padding) {
	int inputSize = sqrt(img.shape()[1]);
	int outputSize = sqrt(dst.shape()[1]);
	cuvAssert(inputSize * inputSize == img.shape()[1]);
	cuvAssert(outputSize * outputSize == dst.shape()[1]);
	cuvAssert(inputSize + 2 * padding == outputSize);
	cuvAssert(img.shape()[0] == dst.shape()[0]);

	// make NVMatrices with this data
	NVMatrix nv_dst(dst.ptr(), dst.shape()[0], dst.shape()[1], false);
	NVMatrix nv_img(img.ptr(), img.shape()[0], img.shape()[1], false);

	copyInto(&nv_img, &nv_dst, padding, false);
}

template<>
	void copy_into(tensor<float,host_memory_space,row_major>& dst,
			  tensor<float,host_memory_space,row_major>&   img,
			  int padding) {
	int inputSize = sqrt(img.shape()[1]);
	int outputSize = sqrt(dst.shape()[1]);
	cuvAssert(inputSize * inputSize == img.shape()[1]);
	cuvAssert(outputSize * outputSize == dst.shape()[1]);
	cuvAssert(inputSize + 2 * padding == outputSize);
	cuvAssert(img.shape()[0] == dst.shape()[0]);

	float* img_ptr = img.ptr();
	float* dst_ptr = dst.ptr();
	for(int i=0; i<img.shape()[0]; i++) {
		dst_ptr += outputSize * padding;
		for(int j=0; j<inputSize;j++) {
			dst_ptr += padding;
			for(int k=0; k<inputSize;k++) {
				*dst_ptr++ = *img_ptr++;
			}
			dst_ptr += padding;
		}
		dst_ptr += outputSize * padding;
	}
}

template<>
	void max_pooling(tensor<float,host_memory_space,row_major>& dst,
			tensor<float,host_memory_space,row_major>& img,
			unsigned int poolSize,
			unsigned int overlap,
			tensor<int,host_memory_space,row_major>* indices,
			tensor<float,host_memory_space,row_major>* filter) {
	if (indices!=NULL) {
		cuvAssert(indices->shape()[1] == dst.shape()[1]);
		cuvAssert(indices->shape()[0] == dst.shape()[0]);
	}

	cuvAssert(poolSize > overlap);
	int numImages = dst.shape()[0];
	cuvAssert(numImages == img.shape()[0]);
	int imgSize = sqrt(img.shape()[1]);
	cuvAssert(imgSize * imgSize == img.shape()[1]);
	int stepSize = poolSize - overlap;
	int dstSize = (imgSize - poolSize)/stepSize + 1;
	cuvAssert(dstSize * dstSize == dst.shape()[1]);
	cuvAssert((dstSize-1)*stepSize + poolSize == imgSize);
	if(filter!=NULL) {
		cuvAssert(filter->shape()[1] == poolSize);
		cuvAssert(filter->shape()[0] == poolSize);
	}

	float* img_ptr = img.ptr();
	float* dst_ptr = dst.ptr();

	for(int p=0; p<numImages; p++) {
		for(int r=0; r<dstSize; r++)
			for(int c=0; c<dstSize; c++) {
				int imax = 0;
				float cmax = -FLT_MAX;
				// loop through pool
				for(int i=0; i<poolSize; i++)
					for(int j=0; j<poolSize; j++) {
						int idx = c*stepSize+j + (r*stepSize+i)*imgSize;
						float val = img_ptr[idx];
						if(filter!=NULL)
							val *= (*filter)(i,j);
						if(cmax < val) {
							cmax = val;
							imax = j*poolSize+i; // transpose due to dev local_maximum() function
						}
					}
				*dst_ptr++ = cmax;
				if(indices != NULL)
					(*indices)(p, r*dstSize+c)=imax;
			}

		img_ptr += imgSize * imgSize;
	}
}

// naive, but flexible implementation
// better distinguish between different cases and load image into shared memory
template<bool INDEX, bool FILTER>
__global__
void max_pooling_kernel(float* dst, float* img, int* indices, int imgSize, int dstSize, int poolSize, int stepSize) {
	int tx = threadIdx.x; // ty = threadIdx.y;
	int bx = blockIdx.x, by = blockIdx.y;

	int p = tx + by * 256;
	if(p >= dstSize * dstSize)
		return;

	img += bx * imgSize * imgSize;

	float cmax = -FLT_MAX;
	int imax = 0;
	int column = p % dstSize;
	int row = p / dstSize;

	// loop through pool
	for(int i=0; i<poolSize; i++)
		for(int j=0; j<poolSize; j++) {
			int idx = column*stepSize+j + (row*stepSize+i)*imgSize;
			float val = img[idx];
			if(FILTER)
				val *= (float) c_filter[i*poolSize+j];
			if(cmax < val) {
				cmax = val;
				if(INDEX)
					imax = j*poolSize+i; // transpose due to dev local_maximum() function
			}
		}

	// write result
	dst += bx * dstSize * dstSize + p;
	//	indices
	if(INDEX) {
		indices += bx * dstSize * dstSize + p;
		*indices = imax;
	}
	*dst = cmax;
}

/* This implementation only achieves a speedup of 5-10x, and is even
 * worse if the pools do not overlap. Better use local_maximum() in this
 * case.
 */

__global__
void first_pooling_zeros_kernel(float* img, int imgSize, int stepSize) {
	int tx = threadIdx.x; // ty = threadIdx.y;
	int bx = blockIdx.x, by = blockIdx.y;

	int p = tx + by * 256;
	if(p >= imgSize * imgSize)
		return;

	img += bx * imgSize * imgSize;

	int column = p % imgSize;
	int row = p / imgSize;

	// write result
	if ((column  % stepSize) || (row % stepSize)){
		img +=  p;
		*img = 0;
		}	
}

template<>
	void first_pooling_zeros(tensor<float,dev_memory_space,row_major>& img,
			unsigned int poolSize
			) {

	int numImages = img.shape()[0];
	int imgSize = sqrt(img.shape()[1]);
	cuvAssert(imgSize * imgSize == img.shape()[1]);
	int stepSize = poolSize;
	int dstSize = (imgSize - poolSize)/stepSize + 1;
	cuvAssert((dstSize-1)*stepSize + poolSize == imgSize);

	int numThreads = 256;
	int numBlocksX = numImages;
	int numBlocksY = ceil((float) (imgSize * imgSize)/numThreads);

	dim3 grid(numBlocksX, numBlocksY);
	dim3 threads(numThreads);
	first_pooling_zeros_kernel<<<grid,threads>>>(img.ptr(), imgSize, stepSize);

	cuvSafeCall(hipDeviceSynchronize());
}

__global__
void first_pooling_kernel(float* dst, float* img, int imgSize, int dstSize, int stepSize) {
	int tx = threadIdx.x; // ty = threadIdx.y;
	int bx = blockIdx.x, by = blockIdx.y;

	int p = tx + by * 256;
	if(p >= dstSize * dstSize)
		return;

	img += bx * imgSize * imgSize;

	int column = p % dstSize;
	int row = p / dstSize;

	// write result
	dst += bx * dstSize * dstSize + p;
	*dst = img[column*stepSize + (row*stepSize)*imgSize];
}

template<>
	void first_pooling(tensor<float,dev_memory_space,row_major>& dst,
			tensor<float,dev_memory_space,row_major>& img,
			unsigned int poolSize
			) {

	int numImages = dst.shape()[0];
	cuvAssert(numImages == img.shape()[0]);
	int imgSize = sqrt(img.shape()[1]);
	cuvAssert(imgSize * imgSize == img.shape()[1]);
	int stepSize = poolSize;
	int dstSize = (imgSize - poolSize)/stepSize + 1;
	cuvAssert(dstSize * dstSize == dst.shape()[1]);
	cuvAssert((dstSize-1)*stepSize + poolSize == imgSize);

	int numThreads = 256;
	int numBlocksX = numImages;
	int numBlocksY = ceil((float) (dstSize * dstSize)/numThreads);

	dim3 grid(numBlocksX, numBlocksY);
	dim3 threads(numThreads);
	first_pooling_kernel<<<grid,threads>>>(dst.ptr(), img.ptr(), imgSize, dstSize, stepSize);

	cuvSafeCall(hipDeviceSynchronize());
}

template<>
	void max_pooling(tensor<float,dev_memory_space,row_major>& dst,
			tensor<float,dev_memory_space,row_major>& img,
			unsigned int poolSize,
			unsigned int overlap,
			tensor<int,dev_memory_space,row_major>* indices,
			tensor<float,dev_memory_space,row_major>* filter) {

	if (indices!=NULL) {
		cuvAssert(indices->shape()[1] == dst.shape()[1]);
		cuvAssert(indices->shape()[0] == dst.shape()[0]);
	}
	cuvAssert(poolSize > overlap);
	int numImages = dst.shape()[0];
	cuvAssert(numImages == img.shape()[0]);
	int imgSize = sqrt(img.shape()[1]);
	cuvAssert(imgSize * imgSize == img.shape()[1]);
	int stepSize = poolSize - overlap;
	int dstSize = (imgSize - poolSize)/stepSize + 1;
	cuvAssert(dstSize * dstSize == dst.shape()[1]);
	cuvAssert((dstSize-1)*stepSize + poolSize == imgSize);

	int numThreads = 256;
	int numBlocksX = numImages;
	int numBlocksY = ceil((float) (dstSize * dstSize)/numThreads);

	if(filter!=NULL) {
		cuvAssert(filter->shape()[1] == poolSize);
		cuvAssert(filter->shape()[0] == poolSize);
		cuvAssert(sizeof(float) * filter->size() <= CONST_SIZE);
		cuvSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_filter), filter->ptr(), sizeof(float) * filter->size(), 0, hipMemcpyDeviceToDevice) );
	}
	cuvSafeCall(hipDeviceSynchronize());

	dim3 grid(numBlocksX, numBlocksY);
	dim3 threads(numThreads);
	if(indices==NULL && filter==NULL)
		max_pooling_kernel<false, false><<<grid,threads>>>(dst.ptr(), img.ptr(), NULL, imgSize, dstSize, poolSize, stepSize);
	else if(indices==NULL && filter!=NULL)
		max_pooling_kernel<false, true><<<grid,threads>>>(dst.ptr(), img.ptr(), NULL, imgSize, dstSize, poolSize, stepSize);
	else if(indices!=NULL && filter==NULL)
		max_pooling_kernel<true, false><<<grid,threads>>>(dst.ptr(), img.ptr(), indices->ptr(), imgSize, dstSize, poolSize, stepSize);
	else if(indices!=NULL && filter!=NULL)
		max_pooling_kernel<true, true><<<grid,threads>>>(dst.ptr(), img.ptr(), indices->ptr(), imgSize, dstSize, poolSize, stepSize);

	cuvSafeCall(hipDeviceSynchronize());
}

/*
 * Block size 16x16.
 */
__global__ void strip_padding_kernel(float* targets, float* images, const int imgSize, const int paddingSize, const int numImages) {
    const int imgIdx = blockIdx.y;

    //check if index is still in matrix
    if (imgIdx < numImages) {
        const int targetSize = imgSize - 2 * paddingSize;
        // move pointer by imgIdx images
        images += imgIdx * imgSize * imgSize;

        // move pointer by imgIdx images
        targets += imgIdx * targetSize * targetSize;

        // what is this pixels index in the source image
        int px = blockIdx.x * blockDim.x + threadIdx.x;

        // pixels coordinates
        int x = px % imgSize;
        int y = px / imgSize;
        if ( x >= paddingSize && x < paddingSize+targetSize &&
        	 y >= paddingSize && y < paddingSize+targetSize){
            // move source pointer to this pixels index in source umage
            images+=px;

        	// move target pointer to target position,
        	targets	+=	(y-paddingSize)*targetSize+(x-paddingSize);

        	// copy contents
        	*targets = *images;
        }
    }
}

/*
 * strip padding removes a border of padding size from each picture_row
 *
 */
template<>
	void strip_padding(tensor<float,dev_memory_space,row_major>& dst,
					   tensor<float,dev_memory_space,row_major>& img,
					   unsigned int padding) {
	int inputSize = sqrt(img.shape()[1]);
	int imgWidth = inputSize;
	int outputSize = sqrt(dst.shape()[1]);
	int numImages = img.shape()[0];
	cuvAssert(inputSize * inputSize == img.shape()[1]);
	cuvAssert(outputSize * outputSize == dst.shape()[1]);
	cuvAssert(inputSize - 2 * padding == outputSize);
	cuvAssert(img.shape()[0] == dst.shape()[0]);
	int numThreads = 256;
	int numBlocksX = ceil((float) (imgWidth * imgWidth)/numThreads);
	int numBlocksY = numImages;
	dim3 grid(numBlocksX, numBlocksY);
	dim3 dimBlock(numThreads,1);
	strip_padding_kernel<<<grid,dimBlock>>>(dst.ptr(), img.ptr(), imgWidth, numImages, padding);
	cuvSafeCall(hipDeviceSynchronize());
}

template<>
	void strip_padding(tensor<float,host_memory_space,row_major>& dst,
					   tensor<float,host_memory_space,row_major>& img,
					   unsigned int padding) {
	int inputSize = sqrt(img.shape()[1]);
	int imgWidth = inputSize;
	int outputSize = sqrt(dst.shape()[1]);
	int numImages = img.shape()[0];
	cuvAssert(inputSize * inputSize == img.shape()[1]);
	cuvAssert(outputSize * outputSize == dst.shape()[1]);
	cuvAssert(inputSize - 2 * padding == outputSize);
	cuvAssert(img.shape()[0] == dst.shape()[0]);
	fill(dst, 0.0f);


	int x,y, idx, idx_padded;
	float val;
	int stripped_width = imgWidth - 2 * padding;

	for (int imgIdx = 0; imgIdx < img.shape()[0]; imgIdx++){
		for(int px = 0; px < img.shape()[1]; px++){
			x = px % inputSize;
			y = px / inputSize;
			if ( x >=padding && x < padding+stripped_width &&
				 y >=padding && y < padding+stripped_width)
			{
				idx 		=	y*inputSize+x;
				idx_padded 	=	(y-padding)*stripped_width+(x-padding);
				val = img(imgIdx, idx);
				dst(imgIdx, idx_padded)=val;
			}
		}
	}

}

/*
 * Block size 16x16.
 */
__global__ void row_ncopy_kernel(float* targets, float* row, const int imgSize, const int n) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    //check if index is still in matrix
    if (idx < imgSize) {
    	row += idx;
    	targets += idx;
    	for(int i=0; i < n ;i++){
    	     *targets = *row;
    	     targets += imgSize;
    	}
    }
}

/*
 * copy 1st row n times in 1 one column
 */

template<>
	void row_ncopy(tensor<float,dev_memory_space,row_major>& dst,
				   tensor<float,dev_memory_space>& row,
				   unsigned int n) {
	int inputSize = row.size();
	cuvAssert(n == dst.shape()[0]);
	cuvAssert(n <= 4096);
	fill(dst, 0.0f);

	int numThreads = 256;
	int numBlocksX = ceil((float)inputSize/numThreads);
	int numBlocksY = 1;
	dim3 grid(numBlocksX, numBlocksY);
	dim3 dimBlock(numThreads,1);
	row_ncopy_kernel<<<grid,dimBlock>>>(dst.ptr(), row.ptr(), inputSize, n);
	cuvSafeCall(hipDeviceSynchronize());
}


template<>
	void row_ncopy(tensor<float,host_memory_space,row_major>& erg_h,
				   tensor<float,host_memory_space>& row,
				   unsigned int n) {

	cuvAssert(n == erg_h.shape()[0]);
	cuvAssert(n <= 4096);
	fill(erg_h, 0.0f);

	fill(erg_h, 0.0f);
	for(int idx = 0; idx < erg_h.shape()[1]; idx++ ){
		for (int idy = 0; idy < n; idy++){
			erg_h(idy,idx)=row[idx];
		}
	}
}


__global__ void cols_ncopy_kernel(float* targets, float* cols, const int rowSize, const int n) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y;
    int newRowSize = rowSize * n;

    //check if index is still in matrix
    if (idx < rowSize) {
	int offset_src_adr = idx + rowSize * row;
    	int offset_dst_adr = idx + newRowSize * row;
    	for(int i=0; i < n ;i++){
    	     *(targets + offset_dst_adr + i * rowSize)= *(cols+offset_src_adr);
    	}
    }
}


template<>
void cols_ncopy(	tensor<float, dev_memory_space, row_major>& dst,
			tensor<float, dev_memory_space, row_major>& col,
			unsigned int n){
	int inputSize 	= col.shape()[1]*col.shape()[0];
	int row_size 	= col.shape()[1];
	
	cuvAssert(n <= 4096);
	cuvAssert(dst.shape()[1] == row_size*n)
	fill(dst, 0.0f);
	int numThreads = 512;
	int numBlocksX = ceil((float)row_size/numThreads);
	int numBlocksY = col.shape()[0];
	dim3 grid(numBlocksX, numBlocksY);
	dim3 dimBlock(numThreads,1);
	cols_ncopy_kernel<<<grid,dimBlock>>>(dst.ptr(), col.ptr(), row_size, n);
	cuvSafeCall(hipDeviceSynchronize());

}


template<>
void cols_ncopy(tensor<float, host_memory_space, row_major>& dst,
		tensor<float, host_memory_space, row_major>& col,
		unsigned int n){
	int inputSize 	= col.shape()[1]*col.shape()[0];
	int row_size 	= col.shape()[1];
	cuvAssert(n <= 4096);
	cuvAssert(dst.shape()[1] == row_size*n)
	fill(dst, 0.0f);
	for(int r = 0; r < col.shape()[0]; r++){
		for(int c = 0; c < col.shape()[1]; c++){
			for(int j = 0; j < n; j++){		
				*(dst.ptr() 	+ r * row_size * n  // shift to correct row using new row width
						+ c 		    // shift by column
						+ j * row_size)	    // shift by old row size to the new position 
						= *(col.ptr() + r * row_size + c); 		
			}
		}	
	}
}



template<>
	void filter_rotate(tensor<float,host_memory_space,row_major>& dst,
					    tensor<float,host_memory_space,row_major>& filter,
					    unsigned int fs){
		int f = filter.shape()[1] / fs;
		float* f_h_ptr = filter.ptr();
		int row_offset=0;
		int f_h_w = filter.shape()[1];
		int numCases = filter.shape()[0];

		// iterate on every filter in a row
		for(int filter = 0; filter < f*fs; filter = filter+fs){
			// iterate on every element of the filter
			for(int y = 0; y < fs; y++){
				// every filterrow
				for(int nC = 0; nC <numCases; nC++){
					row_offset = nC*f_h_w;
					*(dst.ptr()+row_offset+filter+y) = *(f_h_ptr+row_offset+(fs-1)+filter-y);
				}

			}
		}
}

/*
 * this is limited to 22 x 22 filter kernels yet
 */
__global__ void filter_rotate_kernel(float* dst, float* src, const int h, const int w, const int fs, const int size) {

	const int col_idx = threadIdx.x;
	const int row_idx = blockIdx.y;

	// load weights in shared memory
	__shared__  float filter[512];

	int px_adr_glob = 0;

	// check if col idx is less than the number of cells in one row and less than the number of cells at all (at bottom of matrix)
	if( (col_idx < w) && (row_idx * w + col_idx <= size)){
		// I. load pixels in a coalesced way

		// global memory adress for pixel
		px_adr_glob = row_idx * w + col_idx;

		//load filter element
		*(filter+col_idx) =  *(src+px_adr_glob);

		// wait until everything is loaded
		__syncthreads();

		// II. now write with hopefully only few bank conflicts
		int filter_start = (col_idx / fs) * fs;
		int filter_element_idx = col_idx % fs;

		*(dst+px_adr_glob) = *(filter + filter_start + (fs-1) - filter_element_idx);
	}

}

template<>
void filter_rotate(	tensor<float,dev_memory_space,row_major>& dst,
					tensor<float,dev_memory_space,row_major>& filter,
					unsigned int fs){
		cuvAssert(dst.shape()[0] == filter.shape()[0])
		cuvAssert(dst.shape()[1] == filter.shape()[1])

		int num_filter = filter.shape()[1] / fs;
		cuvAssert(sqrt(fs) <= 22)

		float* f_h_ptr = filter.ptr();
		int f_h_w = filter.shape()[1];
		int numCases = filter.shape()[0];

		// we put as many filter in a row of width 512 as possible
		int numFiltersPerRow = 512 / fs;
		int numRows = ceil((float)(num_filter*filter.shape()[0]) / numFiltersPerRow);
		//std::cout << "resizing from " << num_filter << "x" << filter.shape()[0] << " to " << numFiltersPerRow << " x " << numRows << std::endl;
		int _h = numRows;
		int _w = numFiltersPerRow*fs;

		int numThreads = 512;
		int numBlocksX = 1;
		int numBlocksY = _h;
		dim3 grid(numBlocksX, numBlocksY);
		dim3 dimBlock(numThreads,1);

//		std::cout << "filter.h =  " << filter.shape()[0] << std::endl;
		filter_rotate_kernel<<<grid,dimBlock>>>(dst.ptr(), filter.ptr(), _h, _w, fs, filter.shape()[0]*filter.shape()[1]);
		cuvSafeCall(hipDeviceSynchronize());

}

//__global__ void add_maps_h_kernel(float* dst, float* img, const int img_w, const int imagesize) {
//
//	int px = threadIdx.x +  blockDim.x * blockIdx.x;
//	int row = blockIdx.y;
//
//	int num_maps = img_w / imagesize;
//
//	__shared__ float summedMaps[512];
//
//	// sum up in fast shared mem
//	for(int i = 0; i < num_maps; i++){
//		summedMaps[px] += *(img + row * img_w		// goto row in matrix
//								+ px				// pixel
//								+ i * imagesize);   // iterate on images
//	}
//
//	// move result to global mem
//	*(dst + row * img_w + px) = *(summedMaps + row * img_w + px);
//}
//
//template<>
//void add_maps_h(	dense_matrix<float,dev_memory_space,row_major>& dst,
//					dense_matrix<float,dev_memory_space,row_major>& mat,
//					unsigned int image_size){
//
//		int num_images = mat.shape()[1] / image_size;
//		cuvAssert(dst.shape()[1] == image_size);
//		cuvAssert(dst.shape()[0] == mat.shape()[0]);
//		cuvAssert(num_images * image_size == mat.shape()[1]);
//
//		int numThreads = 512;
//		int numBlocksX = ceil((float)mat.shape()[1]/numThreads);
//		int numBlocksY = mat.shape()[0];
//		dim3 grid(numBlocksX, numBlocksY);
//		dim3 dimBlock(numThreads,1);
//
//		add_maps_h_kernel<<<grid,dimBlock>>>(dst.ptr(), mat.ptr(), mat.shape()[1], image_size);
//		cuvSafeCall(hipDeviceSynchronize());
//}
//
//template<>
//void add_maps_h(	dense_matrix<float,host_memory_space,row_major>& dst,
//					dense_matrix<float,host_memory_space,row_major>& mat,
//					unsigned int image_size){
//
//		int num_images = mat.shape()[1] / image_size;
//		cuvAssert(dst.shape()[1] == image_size);
//		cuvAssert(dst.shape()[0] == mat.shape()[0]);
//		cuvAssert(num_images * image_size == mat.shape()[1]);
//
//		float* e_ptr = dst.ptr();
//		float* i_ptr = mat.ptr();
//
//		// host solution
//		for (int row = 0; row<mat.shape()[0]; row++){
//			for(int px = 0; px < image_size; px++){
//				for(int img = 0; img < num_images; img++){
//					*(e_ptr + row*dst.shape()[1] + px) += *(i_ptr + row * dst.shape()[1]  // move to right row
//															 + img * image_size // move to img
//															 + px);				// move to pixel in img
//				}
//			}
//		}
//}

__global__ void calc_error_to_blob_kernel(float* img,
										  float* src,
										  float* blob,
										  const int img_w,
										  const int img_h,
										  float sigma,
										  const int num_maps,
										  float temporal_weight,
										  float interval_size,
										  float interval_offset,
										  const int window_radius) {

	int idx = threadIdx.x +  blockDim.x * blockIdx.x;
	int row = blockIdx.y;

	int x = idx % img_w;
	int y = idx / img_w;

	float center_x = *(blob+row*2);
	float center_y = *(blob+row*2+1);

	float a = (float)(x - center_x)/ sigma;
	float b = (float)(y - center_y)/ sigma;

	// destination is calculated by the row the pixel is in (row*imagesize) and the index in the picture (idx)
	// img_w and img_h refers to the dimensions of an image (one row) in the img matrix

	//p(x,α,σ) = 1/sqrt(2πσ²)*exp(-(x-α)²/2σ²)
	if(idx < img_w * img_h){
		if (window_radius > 0){
			if(		window_radius*window_radius > pow(a*sigma,2) + pow(a*sigma,2)
				or 	(x == center_x and y==center_y)){
				float gauss_value = interval_size*expf(-(a*a + b*b)/2.f)-interval_offset;
				float act_val = *(src+idx+row*(img_w*img_h));
				*(img+idx+row*(img_w*img_h)) =(temporal_weight * (gauss_value - act_val));
			}else
				*(img+idx+row*(img_w*img_h)) = 0.0f;
		}else{
			float gauss_value = interval_size*expf(-(a*a + b*b)/2.f)-interval_offset;
			float act_val = *(src+idx+row*(img_w*img_h));
			*(img+idx+row*(img_w*img_h)) =(temporal_weight * (gauss_value - act_val));
		}
	}
}



template<>
void calc_error_to_blob(	tensor<float,dev_memory_space,row_major>& dst,
							tensor<float,dev_memory_space,row_major>& img,
							tensor<float,dev_memory_space,row_major>& blob_mat,
							unsigned int image_w,
							unsigned int image_h,
							float sigma,
							float temporal_weight,
							float interval_size,
							float interval_offset,
							unsigned int window_radius){
	cuvAssert(dst.shape()[0] == img.shape()[0]);
	cuvAssert(dst.shape()[1] == img.shape()[1]);

	int numThreads = 512;
	int numBlocksX = ceil((float)img.shape()[1]/numThreads);
	int numBlocksY = dst.shape()[0];

	int numTeacherMaps = blob_mat.shape()[0];

	dim3 grid(numBlocksX, numBlocksY);
	dim3 dimBlock(numThreads,1);

	calc_error_to_blob_kernel<<<grid,dimBlock>>>(	dst.ptr(),
													img.ptr(),
													blob_mat.ptr(),
													image_w,
													image_h,
													sigma,
													numTeacherMaps,
													temporal_weight,													
													interval_size,
													interval_offset,
													window_radius);
	cuvSafeCall(hipDeviceSynchronize());
};

template<>
void calc_error_to_blob(	tensor<float,host_memory_space,row_major>& dst,
							tensor<float,host_memory_space,row_major>& img,
							tensor<float,host_memory_space,row_major>& blob_mat,
							unsigned int image_w,
							unsigned int image_h,
							float sigma,
							float temporal_weight,
							float interval_size,
							float interval_offset,
							unsigned int window_radius){
	cuvAssert(dst.shape()[0] == img.shape()[0]);
	cuvAssert(dst.shape()[1] == img.shape()[1]);

	float center_x 		= 0;
	float center_y	 	= 0;
	float a 			= 0;
	float b 			= 0;
	float gauss_value 	= 0;
	float act_val 		= 0;
	int idx 			= 0;
	int mapsize = image_w*image_h;

	for(int y=0; y < image_h; y++){
		for(int x=0; x < image_w; x++){
			for(int map = 0; map < blob_mat.shape()[0]; map++){
				center_x = *(blob_mat.ptr()+map*2);
				center_y = *(blob_mat.ptr()+map*2+1);

				a = (float)(x - center_x)/ sigma;
				b = (float)(y - center_y)/ sigma;

				idx = y * image_w + x;

				gauss_value = interval_size*expf(-(a*a + b*b)/2.f)-interval_offset;
				act_val = *(img.ptr()+idx+map*mapsize);
				*(dst.ptr()+idx+map*mapsize) =(temporal_weight * (gauss_value - act_val));
			}
		}
	}

};

__global__ void check_exitatory_inhibitory_kernel(float* filter,
												  const int filter_w,
												  const int filter_h,
												  const int start_filter,
												  const int filter_pixels,
												  const int num_inhibitory,
												  const int num_exitatory) {

	int idx = threadIdx.x +  blockDim.x * blockIdx.x;
	int row = blockIdx.y;

	int inhib_start_col 	= start_filter * filter_pixels;
	int exit_start_col		= inhib_start_col + num_inhibitory*filter_pixels;

	int ptr_adr =  	inhib_start_col		//move to the beginning of the block
						+ idx				//move to column in block
						+ (row*filter_w);	// move pointer by row many rows down

	if(idx < filter_w and row<filter_h)
		if(idx >= exit_start_col - inhib_start_col){ // if idx is in exitatory block
			if(*(filter+ptr_adr) < 0)
				*(filter + ptr_adr) = 0;
		}else{										 // if idx is in inhibitory
			if(*(filter + ptr_adr) > 0)
				*(filter + ptr_adr) = 0;
		}

}

template<>
void check_exitatory_inhibitory(tensor<float,dev_memory_space,row_major>& filter,
								unsigned int start_filter,
								unsigned int filter_pixels,
								unsigned int num_inhibitory,
								unsigned int num_exitatory){

	int row_size = filter.shape()[1];
	int inhib_start_col 	= start_filter * filter_pixels;
	int inhib_end_col 		= inhib_start_col + num_inhibitory * filter_pixels - 1;
	int exit_start_col		= inhib_end_col + 1;
	int exit_end_col		= exit_start_col + num_exitatory * filter_pixels - 1;
//	std::cout << "filter.h: " << filter.shape()[0] << " cols ges: "<< filter.shape()[1] <<"inhib_start: " << inhib_start_col << " inhib_end: " << inhib_end_col << " exhib_start: "<< exit_start_col << " exit_end: "<< exit_end_col<< std::endl;
	int numThreads = 512;
	int numBlocksX = ceil((float)(exit_end_col-inhib_start_col)/numThreads);
 	int numBlocksY = filter.shape()[0];
// 	std::cout << "launching " << numBlocksX << "x" << numBlocksY << "x512 Threads for " << (exit_end_col-inhib_start_col)*filter.shape()[0] <<" elements"<<std::endl;
	dim3 grid(numBlocksX, numBlocksY);
	dim3 dimBlock(numThreads,1);

	check_exitatory_inhibitory_kernel<<<grid,dimBlock>>>( filter.ptr(),
														  filter.shape()[1],
														  filter.shape()[0],
														  start_filter,
														  filter_pixels,
														  num_inhibitory,
														  num_exitatory);
	cuvSafeCall(hipDeviceSynchronize());

};

template<>
void check_exitatory_inhibitory(tensor<float,host_memory_space,row_major>& filter,
								unsigned int start_filter,
								unsigned int filter_pixels,
								unsigned int num_inhibitory,
								unsigned int num_exitatory){

	int row_size = filter.shape()[1];
	int inhib_start_col 	= start_filter * filter_pixels;
	int inhib_end_col 		= inhib_start_col + num_inhibitory * filter_pixels-1;
	int exit_start_col		= inhib_end_col+1;
	int exit_end_col		= exit_start_col + num_exitatory * filter_pixels-1;
	std::cout << "filter.h: " << filter.shape()[0] << " cols ges: "<< filter.shape()[1] <<"inhib_start: " << inhib_start_col << " inhib_end: " << inhib_end_col << " exhib_start: "<< exit_start_col << " exit_end: "<< exit_end_col<< std::endl;

	// horizontal direction
	for(int c = inhib_start_col; c < inhib_end_col; c = c + 1 ){
		// vertical direction
		for(int r = 0; r < filter.shape()[0]; r++){
			if(*(filter.ptr()+c+(r*row_size)) > 0)
				*(filter.ptr()+c+(r*row_size)) = 0;
		}
	}

	for(int c = exit_start_col; c <= exit_end_col; c = c + 1 ){
		// vertical direction
		for(int r = 0; r < filter.shape()[0]; r++){
			if(*(filter.ptr()+c+(r*row_size)) < 0)
				*(filter.ptr()+c+(r*row_size)) = 0;
		}
	}

};

__global__ void init_exitatory_inhibitory_kernel(float* filter,
												  const int filter_w,
												  const int filter_h,
												  const int start_filter,
												  const int filter_pixels,
												  const int num_inhibitory,
												  const int num_exitatory) {

	int idx = threadIdx.x +  blockDim.x * blockIdx.x;
	int row = blockIdx.y;

	int inhib_start_col 	= start_filter * filter_pixels;
	int exit_start_col		= inhib_start_col + num_inhibitory*filter_pixels;

	int ptr_adr =  	inhib_start_col		//move to the beginning of the block
						+ idx				//move to column in block
						+ (row*filter_w);	// move pointer by row many rows down

	if(idx < filter_w and row<filter_h)
		if(idx >= exit_start_col - inhib_start_col){ // if idx is in exitatory block
			if(*(filter+ptr_adr) < 0)
				*(filter + ptr_adr) = -1 * *(filter + ptr_adr);
		}else{										 // if idx is in inhibitory
			if(*(filter + ptr_adr) > 0)
				*(filter + ptr_adr) = -1 * *(filter + ptr_adr);
		}

}

template<>
void init_exitatory_inhibitory(tensor<float,dev_memory_space,row_major>& filter,
								unsigned int start_filter,
								unsigned int filter_pixels,
								unsigned int num_inhibitory,
								unsigned int num_exitatory){

	int row_size = filter.shape()[1];
	int inhib_start_col 	= start_filter * filter_pixels;
	int inhib_end_col 		= inhib_start_col + num_inhibitory * filter_pixels - 1;
	int exit_start_col		= inhib_end_col + 1;
	int exit_end_col		= exit_start_col + num_exitatory * filter_pixels - 1;
//	std::cout << "filter.h: " << filter.shape()[0] << " cols ges: "<< filter.shape()[1] <<"inhib_start: " << inhib_start_col << " inhib_end: " << inhib_end_col << " exhib_start: "<< exit_start_col << " exit_end: "<< exit_end_col<< std::endl;
	int numThreads = 512;
	int numBlocksX = ceil((float)(exit_end_col-inhib_start_col)/numThreads);
 	int numBlocksY = filter.shape()[0];
// 	std::cout << "launching " << numBlocksX << "x" << numBlocksY << "x512 Threads for " << (exit_end_col-inhib_start_col)*filter.shape()[0] <<" elements"<<std::endl;
	dim3 grid(numBlocksX, numBlocksY);
	dim3 dimBlock(numThreads,1);

	check_exitatory_inhibitory_kernel<<<grid,dimBlock>>>( filter.ptr(),
														  filter.shape()[1],
														  filter.shape()[0],
														  start_filter,
														  filter_pixels,
														  num_inhibitory,
														  num_exitatory);
	cuvSafeCall(hipDeviceSynchronize());

};

template<>
void init_exitatory_inhibitory(tensor<float,host_memory_space,row_major>& filter,
								unsigned int start_filter,
								unsigned int filter_pixels,
								unsigned int num_inhibitory,
								unsigned int num_exitatory){

	int row_size = filter.shape()[1];
	int inhib_start_col 	= start_filter * filter_pixels;
	int inhib_end_col 		= inhib_start_col + num_inhibitory * filter_pixels-1;
	int exit_start_col		= inhib_end_col+1;
	int exit_end_col		= exit_start_col + num_exitatory * filter_pixels-1;
	std::cout << "filter.h: " << filter.shape()[0] << " cols ges: "<< filter.shape()[1] <<"inhib_start: " << inhib_start_col << " inhib_end: " << inhib_end_col << " exhib_start: "<< exit_start_col << " exit_end: "<< exit_end_col<< std::endl;

	// horizontal direction
	for(int c = inhib_start_col; c < inhib_end_col; c = c + 1 ){
		// vertical direction
		for(int r = 0; r < filter.shape()[0]; r++){
			if(*(filter.ptr()+c+(r*row_size)) > 0)
				*(filter.ptr()+c+(r*row_size)) = -1 * *(filter.ptr()+c+(r*row_size));
		}
	}

	for(int c = exit_start_col; c <= exit_end_col; c = c + 1 ){
		// vertical direction
		for(int r = 0; r < filter.shape()[0]; r++){
			if(*(filter.ptr()+c+(r*row_size)) < 0)
				*(filter.ptr()+c+(r*row_size)) = -1 * *(filter.ptr()+c+(r*row_size));
		}
	}

};

}
