#include "hip/hip_runtime.h"
//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*





/** 
 * @file convolution_ops.cu
 * @brief Operations used for convolution and max-pooling
 * @ingroup convolution
 * @date 2010-03-21
 */

#include <cuv/basics/tensor.hpp>
#include <cuv/convert/convert.hpp>
#include <cuv/matrix_ops/matrix_ops.hpp>
#include <cuv/tensor_ops/tensor_ops.hpp>
#include <cuv/random/random.hpp>
#include <3rd_party/cudaconv2/include/cudaconv2/conv_util.cuh>
#include <3rd_party/cudaconv2/include/cudaconv2/cudaconv2.cuh>
#include <3rd_party/cudaconv2/include/nvmatrix/nvmatrix.cuh>
/*#include <3rd_party/cudaconv2/include/convCPU.h>*/
#include <cuv/convolution_ops/convolution_ops.hpp>

#define NVView3D(X)  \
        (const_cast<float*>(X.ptr()), X.shape(0)*X.shape(1), X.shape(2), X.shape(2),false)

namespace cuv{ namespace alex_conv{

template<class V,class M, class T>
    void reorder_for_conv(tensor<V,M,T>& dst, const tensor<V,M,T>& src){
        cuvAssert(src.ndim()==3);
        cuvAssert(dst.ndim()==3);
        std::vector<unsigned int> s = src.shape();
        /*tensor<V,M,T> src_view(indices[index_range()][index_range()][index_range()], src);*/
        tensor<V,M,T>& src_view  = const_cast<tensor<V,M,T>&>(src);
        src_view.reshape(extents[s[0]][s[1]*s[2]]);
        dst.reshape(extents[s[1]*s[2]][s[0]]);
        cuv::transpose(dst,src_view);
        src_view.reshape(s);
        dst.reshape(extents[s[1]][s[2]][s[0]]);
    }
template<class V,class M, class T>
    void reorder_from_conv(tensor<V,M,T>& dst, const tensor<V,M,T>& src){
        cuvAssert(src.ndim()==3);
        cuvAssert(dst.ndim()==3);
        tensor<V,M,T> src_view(indices[index_range()][index_range()][index_range()], src);
        src_view.reshape(extents[src.shape(0)*src.shape(1)][src.shape(2)]);
        dst.reshape(extents[dst.shape(0)][dst.shape(1)*dst.shape(2)]);
        cuv::transpose(dst,src_view);
        dst.reshape(extents[src.shape(2)][src.shape(0)][src.shape(1)]);
    }

template<>
    void 
    convolve2d(tensor<float,dev_memory_space>& dst, 
            const tensor<float,dev_memory_space>& img, 
            const tensor<float,dev_memory_space>& filter,
            unsigned int paddingStart, 
            unsigned int moduleStride,
            unsigned int nGroups){
        // check compatibility before converting to NVMatrix format
        /*cuvAssert(dst.ndim()==3);*/
        cuvAssert(img.ndim()==3);
        unsigned int nImgChan = img.shape(0);
        unsigned int nImgPix  = img.shape(1);
        unsigned int nImg     = img.shape(2);

        cuvAssert(filter.ndim()==3);
        unsigned int nFiltChan = filter.shape(0);
        unsigned int nFiltPix  = filter.shape(1);
        unsigned int nFilt     = filter.shape(2);

        cuvAssert(dst.shape(0)==nFilt);
        unsigned int nModules = dst.shape(1);
        unsigned int nModulesX = sqrt(nModules);
        cuvAssert(nModules == nModulesX * nModulesX);
        cuvAssert(dst.shape(2)==nImg);

        // make NVMatrices with this data
        NVMatrix nv_dst    NVView3D(dst);
        NVMatrix nv_img    NVView3D(img);
        NVMatrix nv_filter NVView3D(filter);

        if(nFilt<16){
            // we can use this for output maps, which still must be divisible by four(!)
            // this is still fully connected, however we must resort to "sparse" conv
            // since the non-sparse conv only allows 
            int* colorIndices = new int[nGroups*nFiltChan]; 
            for(unsigned int i=0;i<nGroups*nFiltChan;i++) colorIndices[i]=i;
            convFilterActsSparse(nv_img, nv_filter, nv_dst, colorIndices, nModulesX, paddingStart, moduleStride, nImgChan, nFiltChan, nGroups);
        }{
            convFilterActs(nv_img, nv_filter, nv_dst, nModulesX, paddingStart, moduleStride, nImgChan, nGroups);
        }
    }
template<>
	void d_conv2d_dimg(tensor<float,dev_memory_space,row_major>& dst,
			  const tensor<float,dev_memory_space,row_major>&   delta,
			  const tensor<float,dev_memory_space,row_major>&   filter,
              unsigned int paddingStart, unsigned int moduleStride, unsigned int nGroups){


        cuvAssert(delta.ndim()==3);
        unsigned int nFilt    = delta.shape(0);
        unsigned int nModules = delta.shape(1); 
        unsigned int nImg     = delta.shape(2);

        cuvAssert(filter.ndim()==3);
        unsigned int nFiltChan = filter.shape(0);
        unsigned int nFiltPix  = filter.shape(1);
        /*unsigned int nFilt     = filter.shape(2);*/
        cuvAssert(filter.shape(2) == nFilt);

        cuvAssert(dst.ndim()==3);
        unsigned int nImgChan  = dst.shape(0);
        unsigned int nImgPix   = dst.shape(1);
        cuvAssert(dst.shape(2) == nImg);

        unsigned int imgSize = sqrt(nImgPix);
        cuvAssert(nImgPix == imgSize*imgSize);

        /*void convImgActs(NVMatrix& hidActs, NVMatrix& filters, NVMatrix& targets,*/
        /*    int imgSize, int paddingStart, int moduleStride, int numImgColors, int numGroups);*/

        NVMatrix nv_dst    NVView3D(dst);
        NVMatrix nv_delta  NVView3D(delta);
        NVMatrix nv_filter NVView3D(filter);

        convImgActs(nv_delta, nv_filter, nv_dst,
                imgSize, paddingStart, moduleStride, nImgChan, nGroups);
    }
template<>
	void d_conv2d_dfilt(tensor<float,dev_memory_space,row_major>& dst_,
			  const tensor<float,dev_memory_space,row_major>&   delta,
			  const tensor<float,dev_memory_space,row_major>&   input,
              unsigned int paddingStart,
            unsigned int moduleStride, unsigned int nGroups, unsigned int partialSum){

        cuvAssert(dst_.ndim()==3);
        unsigned int nFiltChan = dst_.shape(0);
        unsigned int nFiltPix  = dst_.shape(1);
        unsigned int nFilt     = dst_.shape(2);



        unsigned int filtSize = sqrt(nFiltPix);
        cuvAssert ( nFiltPix == filtSize*filtSize );


        cuvAssert(delta.ndim()==3);
        cuvAssert(delta.shape(0) == nFilt);
        unsigned int nModules  = delta.shape(1);
        unsigned int nImg      = delta.shape(2);

        unsigned int nModulesX = sqrt(nModules);
        cuvAssert(nModules == nModulesX * nModulesX);

        cuv::tensor<float,dev_memory_space> dst(extents[nModules/partialSum][nFiltChan][nFiltPix][nFilt]);

        cuvAssert(input.ndim()==3);
        unsigned int nImgChan = input.shape(0);
        unsigned int nImgPix  = input.shape(1);
        cuvAssert(input.shape(2) == nImg);

        unsigned int imgSize = sqrt(nImgPix);
        cuvAssert(nImgPix == imgSize*imgSize);


        /*void convWeightActs(NVMatrix& images, NVMatrix& hidActs, NVMatrix& targets,*/
        /*                    int numModulesX, int filterSize, int paddingStart,*/
        /*                    int moduleStride, int numImgColors, int numGroups, int partialSum);*/
        NVMatrix nv_dst   NVView3D(dst);
        NVMatrix nv_delta NVView3D(delta);
        NVMatrix nv_input NVView3D(input);
        convWeightActs(nv_input, nv_delta, nv_dst,
                nModulesX, filtSize, paddingStart,
                moduleStride, nImgChan, nGroups, partialSum);

        dst.reshape(extents[nModules/partialSum][nFiltChan*nFiltPix*nFilt]);
        dst_.reshape(extents[nFiltChan*nFiltPix*nFilt]);
        cuv::reduce_to_row(dst_,dst);
        dst_.reshape(extents[nFiltChan][nFiltPix][nFilt]);
    }


template<>
    void local_pool(tensor<float,dev_memory_space>& target,
            const tensor<float,dev_memory_space>& images,
            int subsX, int startX, int strideX, int outputsX, pool_type pooler){

        cuvAssert(images.ndim()==3);
        unsigned int nFilt   = images.shape(0);
        unsigned int nImgPix = images.shape(1);
        unsigned int nImg    = images.shape(2);

        cuvAssert(target.ndim()==3);
        cuvAssert(target.shape(0) == nFilt);
        unsigned int outputs = target.shape(1);
        cuvAssert(target.shape(2) == nImg);

        unsigned int imgSize = sqrt(nImgPix);
        cuvAssert(imgSize*imgSize == nImgPix);

        unsigned int outSize = sqrt(outputs);
        cuvAssert(outSize*outSize == outputs);

        unsigned int poolSize = imgSize / outSize;
        cuvAssert(poolSize*outSize == imgSize);

        NVMatrix nv_target NVView3D(target);
        NVMatrix nv_images NVView3D(images);
        

        switch(pooler){
            case PT_MAX:
                convLocalPool(nv_images, nv_target, nFilt,
                        subsX, startX, strideX, outputsX, MaxPooler());
                break;
            case PT_AVG:
                convLocalPool(nv_images, nv_target, nFilt,
                        subsX, startX, strideX, outputsX, AvgPooler(poolSize));
                break;
        }
    }
template<>
    void local_max_pool_grad(tensor<float,dev_memory_space>& target, const tensor<float,dev_memory_space>& images, const tensor<float,dev_memory_space>& maxGrads,
            const tensor<float,dev_memory_space>& maxActs, int subsX, int startX, int strideX){


        cuvAssert(target.ndim()==3);
        unsigned int nImgChan  = target.shape(0);
        unsigned int nImgPix   = target.shape(1);
        unsigned int nImg      = target.shape(2);

        cuvAssert(images.ndim()==3);
        cuvAssert(nImgChan == images.shape(0));
        unsigned int nOutPix = images.shape(1);
        cuvAssert(nImg     == images.shape(2));

        unsigned int outputsX = sqrt(nOutPix);
        cuvAssert(outputsX*outputsX==nOutPix);

        NVMatrix nv_target NVView3D(target);
        NVMatrix nv_images NVView3D(images);
        NVMatrix nv_maxGrads NVView3D(maxGrads);
        NVMatrix nv_maxActs NVView3D(maxActs);
        
/*void convLocalMaxUndo(NVMatrix& images, NVMatrix& maxGrads, NVMatrix& maxActs, NVMatrix& target,*/
/*                      int subsX, int startX, int strideX, int outputsX);*/
        convLocalMaxUndo(nv_images,nv_maxGrads, nv_maxActs, nv_target, subsX,startX,strideX,outputsX);
    }

template<>
    void local_avg_pool_grad(tensor<float,dev_memory_space>& target, const tensor<float,dev_memory_space>& avgGrads,
            int subsX, int startX, int strideX){


        cuvAssert(target.ndim()==3);
        unsigned int nImgChan  = target.shape(0);
        unsigned int nImgPix   = target.shape(1);
        unsigned int nImg      = target.shape(2);

        cuvAssert(avgGrads.ndim()==3);
        cuvAssert(nImgChan == avgGrads.shape(0));
        unsigned int nOutPix = avgGrads.shape(1);
        cuvAssert(nImg == avgGrads.shape(2));

        unsigned int outputsX = sqrt(nOutPix);
        cuvAssert(outputsX*outputsX==nOutPix);

        unsigned int imgX = sqrt(nImgPix);
        cuvAssert(imgX*imgX == nImgPix);

        NVMatrix nv_target NVView3D(target);
        NVMatrix nv_avgGrads NVView3D(avgGrads);
        
        convLocalAvgUndo(nv_avgGrads, nv_target, subsX,startX,strideX,outputsX,imgX);
    }

// instantiate
#define  TENS(V,M,T)       tensor<V,M,T>
#define CTENS(V,M,T) const TENS(V,M,T)
#define INST(V,M,T) \
template void reorder_for_conv<V,M,T>(TENS(V,M,T)&, CTENS(V,M,T)&); \
template void reorder_from_conv<V,M,T>(TENS(V,M,T)&, CTENS(V,M,T)&);
INST(float,host_memory_space,row_major);
INST(float,dev_memory_space,row_major);
}}

