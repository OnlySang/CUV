#include "hip/hip_runtime.h"
/*
 * Original source from nvidia cuda SDK 4.0
 */

#include <cuv/basics/tensor.hpp>
#include <cuv/tensor_ops/tensor_ops.hpp>
#include <cuv/tensor_ops/functors.hpp>
#include <cuv/libs/separable_conv/separable_convolution.hpp>

namespace cuv{

	namespace sep_conv{


#define PITCH(PTR,PITCH,Y,X) ((typeof(PTR))((char*)PTR + PITCH*Y) + X)
#define MAX_KERNEL_RADIUS 8
#define      MAX_KERNEL_W (2 * MAX_KERNEL_RADIUS + 1)
		__device__ __constant__ float c_Kernel[MAX_KERNEL_W];

		////////////////////////////////////////////////////////////////////////////////
		// Row convolution filter
		////////////////////////////////////////////////////////////////////////////////
#define   ROWS_BLOCKDIM_X 16
#define   ROWS_BLOCKDIM_Y 4
#define ROWS_RESULT_STEPS 8
#define   ROWS_HALO_STEPS 1

		/*template<int KERNEL_RADIUS, class SrcT, class DstT, class BinFuncConv>*/
		template<int KERNEL_RADIUS, class SrcT, class DstT>
		__global__ void convolutionRowGPU(
				DstT *d_Dst,
				const SrcT *d_Src,
				int imageW,
				int imageH,
				int dpitch,
				int spitch/*,*/
				/*BinFuncConv mult*/
				){
			__shared__ SrcT s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

			//Offset to the left halo edge
			const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
			const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

			d_Src = PITCH(d_Src, spitch, baseY, baseX);
			d_Dst = PITCH(d_Dst, dpitch, baseY, baseX);

			//Load main data
#pragma unroll
			for(int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
				s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X)  ? d_Src[i * ROWS_BLOCKDIM_X] : 0;

			//Load left halo
#pragma unroll
			for(int i = 0; i < ROWS_HALO_STEPS; i++)
				s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X ) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;

			//Load right halo
#pragma unroll
			for(int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
				s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;

			//Compute and store results
			__syncthreads();

#pragma unroll
			for(int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++){
				float sum = 0;

#pragma unroll
				for(int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++){
					/*sum += mult(s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j], c_Kernel[KERNEL_RADIUS - j]);*/
					sum += s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j]* c_Kernel[KERNEL_RADIUS - j];
				}

				d_Dst[i * ROWS_BLOCKDIM_X] = sum;
			}
		}

		////////////////////////////////////////////////////////////////////////////////
		// Column convolution filter
		////////////////////////////////////////////////////////////////////////////////
#define   COLUMNS_BLOCKDIM_X 16
#define   COLUMNS_BLOCKDIM_Y 8
#define COLUMNS_RESULT_STEPS 8
#define   COLUMNS_HALO_STEPS 1

		/*template<int KERNEL_RADIUS, class SrcT, class DstT, class BinFuncConv>*/
		template<int KERNEL_RADIUS, class SrcT, class DstT>
		__global__ void convolutionColumnGPU(
				DstT *d_Dst,
				const SrcT *d_Src,
				int imageW,
				int imageH,
				int dpitch,
				int spitch
				/*BinFuncConv mult*/
				){
			__shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

			//Offset to the upper halo edge
			const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
			const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
			d_Src = PITCH(d_Src, spitch, baseY, baseX);
			d_Dst = PITCH(d_Dst, dpitch, baseY, baseX);

			//Main data
#pragma unroll
			for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
				s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (imageH-baseY > i * COLUMNS_BLOCKDIM_Y) ? *PITCH(d_Src, spitch, i*COLUMNS_BLOCKDIM_Y,0) : 0;

			//Upper halo
#pragma unroll
			for(int i = 0; i < COLUMNS_HALO_STEPS; i++)
				s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? *PITCH(d_Src,spitch,i*COLUMNS_BLOCKDIM_Y,0) : 0;

			//Lower halo
#pragma unroll
			for(int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
				s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? *PITCH(d_Src,spitch,i*COLUMNS_BLOCKDIM_Y,0) : 0;

			//Compute and store results
			__syncthreads();
#pragma unroll
			for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++){
				float sum = 0;
#pragma unroll
				for(int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
					sum += s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j]* c_Kernel[KERNEL_RADIUS - j];
				if(imageH - baseY > i * COLUMNS_BLOCKDIM_Y)
					*PITCH(d_Dst,dpitch,i*COLUMNS_BLOCKDIM_Y,0) = sum;
			}
		}


		int iDivUp(int a, int b){  	  	 
			return (a % b != 0) ? (a / b + 1) : (a / b);  	  	 
		} 
#define V(X) #X << " : "<< (X)<<"  "
		template<int radius, int channels, class DstV, class SrcV, class A>
		void convolve_call_kernel(tensor<DstV,dev_memory_space,row_major,A>& dst,
				     const tensor<SrcV,dev_memory_space,row_major,A>& src, int dir
				     ){

			int dw = src.shape()[1]/channels;
			int dh = src.shape()[0];

			typedef vector_type_traits<DstV> dst_traits;
			typedef vector_type_traits<SrcV> src_traits;
			typedef typename dst_traits::template vector<channels>::type* dst_vec_t;
			typedef typename src_traits::template vector<channels>::type* src_vec_t;

			if(dir==0){
				dim3 blocks(iDivUp(dw , (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X)), iDivUp(dh , ROWS_BLOCKDIM_Y));
				dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);
				 convolutionRowGPU<radius><<<blocks, threads>>>( (dst_vec_t) dst.ptr(), (src_vec_t) src.ptr(), dw, dh,dst.pitch(),src.pitch());
				/*convolutionRowGPU<radius><<<blocks, threads>>>( (dst_vec_t) dst.ptr(), (src_vec_t) src.ptr(), dw, dh,dst.pitch(),src.pitch(),*/
						/*make_bf_vd_vd<channels,1>(bf_multiplies<DstV,SrcV,float>()));*/
			}else if(dir==1){
				dim3 blocks(iDivUp(dw , COLUMNS_BLOCKDIM_X), iDivUp(dh , (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y)));
				dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);
				convolutionColumnGPU<radius><<<blocks, threads>>>( (dst_vec_t) dst.ptr(), (src_vec_t) src.ptr(), dw, dh, dst.pitch(), src.pitch());
				/*convolutionColumnGPU<radius><<<blocks, threads>>>( (dst_vec_t) dst.ptr(), (src_vec_t) src.ptr(), dh, dw, dst.pitch(), src.pitch(),*/
						/*make_bf_vd_vd<channels,1>(bf_multiplies<DstV,SrcV,float>()));*/
			}
			cuvSafeCall(hipDeviceSynchronize());
		}


		template<int Channels,class DstV, class SrcV, class M, class A>
		void radius_dispatch(const unsigned int& radius,
					   interleaved_image<Channels,DstV,M>& dst,
				     const interleaved_image<Channels,SrcV,M>& src,int dir){
			switch(radius){
				case 1: convolve_call_kernel<1,Channels>(dst.tens(),src.tens(),dir); break;
				case 2: convolve_call_kernel<2,Channels>(dst.tens(),src.tens(),dir); break;
				case 3: convolve_call_kernel<3,Channels>(dst.tens(),src.tens(),dir); break;
				case 4: convolve_call_kernel<4,Channels>(dst.tens(),src.tens(),dir); break;
				case 5: convolve_call_kernel<5,Channels>(dst.tens(),src.tens(),dir); break;
				case 6: convolve_call_kernel<6,Channels>(dst.tens(),src.tens(),dir); break;
				case 7: convolve_call_kernel<7,Channels>(dst.tens(),src.tens(),dir); break;
				case 8: convolve_call_kernel<8,Channels>(dst.tens(),src.tens(),dir); break;
				default: cuvAssert(false);
			}
		}
		template<class DstV, class SrcV, class M, class A>
		void radius_dispatch(const unsigned int& radius,tensor<DstV,M,row_major,A>& dst,
				     const tensor<SrcV,M,row_major,A>& src,int dir){
			switch(radius){
				case 1: convolve_call_kernel<1,1>(dst,src,dir); break;
				case 2: convolve_call_kernel<2,1>(dst,src,dir); break;
				case 3: convolve_call_kernel<3,1>(dst,src,dir); break;
				case 4: convolve_call_kernel<4,1>(dst,src,dir); break;
				case 5: convolve_call_kernel<5,1>(dst,src,dir); break;
				case 6: convolve_call_kernel<6,1>(dst,src,dir); break;
				case 7: convolve_call_kernel<7,1>(dst,src,dir); break;
				case 8: convolve_call_kernel<8,1>(dst,src,dir); break;
				default: cuvAssert(false);
			}
		}
		template<class DstV, class SrcV, class M, class A>
		void
		convolve(       tensor<DstV,M,row_major, A>& dst,
			  const tensor<SrcV,M,row_major, A>& src,
			  const unsigned int&   filter_radius,
			  const separable_filter& filt, int axis, 
			  const float& param ){

			typedef tensor<DstV,M,row_major,A> result_type;
			typedef tensor<SrcV,M,row_major,A>    src_type;
			cuvAssert(filter_radius <= MAX_KERNEL_RADIUS);
                        cuvAssert(src.ndim()==2 || src.ndim()==3);

			if(!equal_shape(dst,src)){
				dst = result_type(src.shape());
			}

			if(src.ndim()==3){
				const std::vector<typename src_type::index_type>& s = src.shape();
				for(unsigned int i=0;i<s[0];i++){
					src_type    sview(indices[i][index_range(0,s[1])][index_range(0,s[2])], src);
					result_type dview(indices[i][index_range(0,s[1])][index_range(0,s[2])], dst);
					convolve(dview,sview,filter_radius,filt,axis,param);
				}
				return;
			}

			if(filt == SP_GAUSS){
				const int kernel_w = 2*filter_radius+1;
				cuv::tensor<float, host_memory_space> kernel(kernel_w);
				for(int i = 0; i < kernel_w; i++){
					float dist = (float)(i - (int)filter_radius);
					kernel[i]  = expf(- dist * dist / (2*param*param));
				}
				kernel /= cuv::sum(kernel);
				cuvSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), kernel.ptr(), kernel.memsize()) );
				result_type tmp(extents[src.shape()[0]][src.shape()[1]]);
				radius_dispatch(filter_radius,tmp,src,0);
				radius_dispatch(filter_radius,dst,tmp,1);
			}else if(filt == SP_CENTERED_DERIVATIVE){
				cuvAssert(axis==0 || axis==1);
				cuv::tensor<float, host_memory_space> kernel(3);
				kernel[0]=-0.5;
				kernel[1]= 0;
				kernel[2]= 0.5;
				cuvSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), kernel.ptr(), kernel.memsize()) );
				radius_dispatch(1,dst,src,axis);
			}else if(filt == SP_BOX){
				const int kernel_w = 2*filter_radius+1;
				cuv::tensor<float, host_memory_space> kernel(kernel_w);
				cuv::fill(kernel, 1.f);
				kernel /= (float) kernel_w;
				cuvSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), kernel.ptr(), kernel.memsize()) );
				result_type tmp(extents[src.shape()[0]][src.shape()[1]]);
				radius_dispatch(filter_radius,tmp,src,0);
				radius_dispatch(filter_radius,dst,tmp,1);
			}
		}

		template<int Channels,class DstV, class SrcV, class M>
		void
		convolve(       interleaved_image<Channels,DstV,M>& dst,
			  const interleaved_image<Channels,SrcV,M>& src,
			  const unsigned int&   filter_radius,
			  const separable_filter& filt, int axis, 
			  const float& param ){

			typedef interleaved_image<Channels,DstV,M> result_type;
			typedef interleaved_image<Channels,SrcV,M>    src_type;
			cuvAssert(filter_radius <= MAX_KERNEL_RADIUS);

			if(filt == SP_GAUSS){
				const int kernel_w = 2*filter_radius+1;
				cuv::tensor<float, host_memory_space> kernel(kernel_w);
				for(int i = 0; i < kernel_w; i++){
					float dist = (float)(i - (int)filter_radius);
					kernel[i]  = expf(- dist * dist / (2*param*param));
				}
				kernel /= cuv::sum(kernel);
				cuvSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), kernel.ptr(), kernel.memsize()) );
				result_type tmp(src.height(), src.width(), src.channels());

				float4 f4;
				float f;
				f4 = make_uf_vd_vd<4>(uf_abs<float,float>())(f4);
				f4 = make_bf_vd_vd<4,1>(bf_plus<float,float,float>())(f4,f);
				f4 = make_bf_vd_vd<4,4>(bf_plus<float,float,float>())(f4,f4);
				/*radius_dispatch(filter_radius,tmp,src,0);*/
				/*radius_dispatch(filter_radius,dst,tmp,1);*/
			}	
		}
		
		// instantiations
#define INST(DSTV, SRCV,M,A) \
		template void \
		convolve<DSTV,SRCV,M>( tensor<DSTV,M,row_major, A>&, \
				const tensor<SRCV,M,row_major, A>&, \
				const unsigned int&,                     \
				const separable_filter&, int axis, \
				const float&);
#define INST_IL(CHANNELS,DSTV, SRCV,M) \
		template void \
		convolve<CHANNELS,DSTV,SRCV,M>( interleaved_image<CHANNELS,DSTV,M>&, \
				const interleaved_image<CHANNELS,SRCV,M>&, \
				const unsigned int&,                     \
				const separable_filter&, int axis, \
				const float&);
		INST(float,float,dev_memory_space,linear_memory_tag);
		INST(float,float,dev_memory_space,memory2d_tag);

		INST_IL(4,float,float,dev_memory_space);
	} // namespace separable convolution
} // namespace cuv
