#include "hip/hip_runtime.h"
/*
 * Original source from nvidia cuda SDK 2.0
 * Modified by S. James Lee (sjames@evl.uic.edi)
 * 2008.12.05
 * Further modified by Hannes Schulz
 */

#include <cuv/basics/tensor.hpp>
#include <cuv/tensor_ops/tensor_ops.hpp>
#include <cuv/libs/separable_conv/separable_convolution.hpp>

namespace cuv{

	namespace sep_conv{

		//24-bit multiplication is faster on G80,
		//but we must be sure to multiply integers
		//only within [-8M, 8M - 1] range
#define IMUL(a, b) __mul24(a, b)

		////////////////////////////////////////////////////////////////////////////////
		// Kernel configuration
		////////////////////////////////////////////////////////////////////////////////
#define MAX_KERNEL_RADIUS 10
#define      MAX_KERNEL_W (2 * MAX_KERNEL_RADIUS + 1)
		__device__ __constant__ float d_Kernel[MAX_KERNEL_W];

		// Assuming ROW_TILE_W, KERNEL_RADIUS_ALIGNED and dataW 
		// are multiples of coalescing granularity size,
		// all global memory operations are coalesced in convolutionRowGPU()
#define            ROW_TILE_W 128
#define KERNEL_RADIUS_ALIGNED 16

		// Assuming COLUMN_TILE_W and dataW are multiples
		// of coalescing granularity size, all global memory operations 
		// are coalesced in convolutionColumnGPU()
#define COLUMN_TILE_W 16
#define COLUMN_TILE_H 48


		////////////////////////////////////////////////////////////////////////////////
		// Row convolution filter
		////////////////////////////////////////////////////////////////////////////////
		template<int KERNEL_RADIUS, class SrcT, class DstT>
			__global__ void convolutionRowGPU(
					DstT       *d_Result,
					const SrcT *d_Data,
					int dataW,
					int dataH
					){
				//Data cache
				__shared__ SrcT data[KERNEL_RADIUS + ROW_TILE_W + KERNEL_RADIUS];

				//Current tile and apron limits, relative to row start
				const int         tileStart = IMUL(blockIdx.x, ROW_TILE_W);
				const int           tileEnd = tileStart + ROW_TILE_W - 1;
				const int        apronStart = tileStart - KERNEL_RADIUS;
				const int          apronEnd = tileEnd   + KERNEL_RADIUS;

				//Clamp tile and apron limits by image borders
				const int    tileEndClamped = min(tileEnd, dataW - 1);
				const int apronStartClamped = max(apronStart, 0);
				const int   apronEndClamped = min(apronEnd, dataW - 1);

				//Row start index in d_Data[]
				const int          rowStart = IMUL(blockIdx.y, dataW);

				//Aligned apron start. Assuming dataW and ROW_TILE_W are multiples 
				//of half-warp size, rowStart + apronStartAligned is also a 
				//multiple of half-warp size, thus having proper alignment 
				//for coalesced d_Data[] read.
				const int apronStartAligned = tileStart - KERNEL_RADIUS_ALIGNED;

				const int loadPos = apronStartAligned + threadIdx.x;
				//Set the entire data cache contents
				//Load global memory values, if indices are within the image borders,
				//or initialize with zeroes otherwise
				if(loadPos >= apronStart){
					const int smemPos = loadPos - apronStart;

					data[smemPos] = 
						((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ?
						d_Data[rowStart + loadPos] : 0;
				}


				//Ensure the completness of the loading stage
				//because results, emitted by each thread depend on the data,
				//loaded by another threads
				__syncthreads();
				const int writePos = tileStart + threadIdx.x;

				//Assuming dataW and ROW_TILE_W are multiples of half-warp size,
				//rowStart + tileStart is also a multiple of half-warp size,
				//thus having proper alignment for coalesced d_Result[] write.
				if(writePos <= tileEndClamped){
					const int smemPos = writePos - apronStart;
					DstT sum = 0;

					for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
						sum += data[smemPos + k] * d_Kernel[KERNEL_RADIUS - k];

					d_Result[rowStart + writePos] = sum;
				}
			}



		////////////////////////////////////////////////////////////////////////////////
		// Column convolution filter
		////////////////////////////////////////////////////////////////////////////////
		template<int KERNEL_RADIUS, class SrcT, class DstT>
			__global__ void convolutionColumnGPU(
					DstT       *d_Result,
					const SrcT *d_Data,
					int dataW,
					int dataH,
					int smemStride,
					int gmemStride
					){
				//Data cache
				__shared__ SrcT data[COLUMN_TILE_W * (KERNEL_RADIUS + COLUMN_TILE_H + KERNEL_RADIUS)];

				//Current tile and apron limits, in rows
				const int         tileStart = IMUL(blockIdx.y, COLUMN_TILE_H);
				const int           tileEnd = tileStart + COLUMN_TILE_H - 1;
				const int        apronStart = tileStart - KERNEL_RADIUS;
				const int          apronEnd = tileEnd   + KERNEL_RADIUS;

				//Clamp tile and apron limits by image borders
				const int    tileEndClamped = min(tileEnd, dataH - 1);
				const int apronStartClamped = max(apronStart, 0);
				const int   apronEndClamped = min(apronEnd, dataH - 1);

				//Current column index
				const int       columnStart = IMUL(blockIdx.x, COLUMN_TILE_W) + threadIdx.x;

				//Shared and global memory indices for current column
				int smemPos = IMUL(threadIdx.y, COLUMN_TILE_W) + threadIdx.x;
				int gmemPos = IMUL(apronStart + threadIdx.y, dataW) + columnStart;

				//Cycle through the entire data cache
				//Load global memory values, if indices are within the image borders,
				//or initialize with zero otherwise
				for(int y = apronStart + threadIdx.y; y <= apronEnd; y += blockDim.y){
					data[smemPos] = 
						((y >= apronStartClamped) && (y <= apronEndClamped)) ? 
						d_Data[gmemPos] : 0;
					smemPos += smemStride;
					gmemPos += gmemStride;
				}

				//Ensure the completness of the loading stage
				//because results, emitted by each thread depend on the data, 
				//loaded by another threads
				__syncthreads();

				//Shared and global memory indices for current column
				smemPos = IMUL(threadIdx.y + KERNEL_RADIUS, COLUMN_TILE_W) + threadIdx.x;
				gmemPos = IMUL(tileStart + threadIdx.y , dataW) + columnStart;

				//Cycle through the tile body, clamped by image borders
				//Calculate and output the results
				for(int y = tileStart + threadIdx.y; y <= tileEndClamped; y += blockDim.y){
					DstT sum = 0;

					for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
						sum += 
							data[smemPos + IMUL(k, COLUMN_TILE_W)] *
							d_Kernel[KERNEL_RADIUS - k];

					d_Result[gmemPos] = sum;
					smemPos += smemStride;
					gmemPos += gmemStride;
				}
			}

		int iDivUp(int a, int b){
			return (a % b != 0) ? (a / b + 1) : (a / b);
		}


#define V(X) #X << " : "<< (X)<<"  "
		template<int radius, class DstV, class SrcV>
		void convolve(tensor<DstV,dev_memory_space,row_major>& dst,
				     const tensor<SrcV,dev_memory_space,row_major>& src, int dir=2){

			int dw = src.shape()[1];
			int dh = src.shape()[0];
			dim3 blockGridRows(iDivUp(dw, ROW_TILE_W), dh);
			dim3 threadBlockRows(KERNEL_RADIUS_ALIGNED + ROW_TILE_W + radius);	// 16 128 8
			dim3 blockGridColumns(iDivUp(dw, COLUMN_TILE_W), iDivUp(dh, COLUMN_TILE_H));
			dim3 threadBlockColumns(COLUMN_TILE_W, 8);
			
			if(dir==2){
				tensor<DstV,dev_memory_space,row_major> intermed(dst.shape());
				convolutionRowGPU<radius><<<blockGridRows, threadBlockRows>>>( intermed.ptr(), src.ptr(), src.shape()[1], src.shape()[0]);
				convolutionColumnGPU<radius><<<blockGridColumns, threadBlockColumns>>>( dst.ptr(), intermed.ptr(), intermed.shape()[1], intermed.shape()[0], COLUMN_TILE_W * threadBlockColumns.y, intermed.shape()[1] * threadBlockColumns.y);
			}
			else if(dir==0){
				convolutionRowGPU<radius><<<blockGridRows, threadBlockRows>>>( dst.ptr(), src.ptr(), src.shape()[1], src.shape()[0]);
			}else if(dir==1){
				convolutionColumnGPU<radius><<<blockGridColumns, threadBlockColumns>>>( dst.ptr(), src.ptr(), src.shape()[1], src.shape()[0], COLUMN_TILE_W * threadBlockColumns.y, src.shape()[1] * threadBlockColumns.y);
			}
			cuvSafeCall(hipDeviceSynchronize());
			safeThreadSync();
		}


		template<class DstV, class SrcV, class M>
		void radius_dispatch(const unsigned int& radius,tensor<DstV,M,row_major>& dst,
				     const tensor<SrcV,M,row_major>& src,int dir=2){
			switch(radius){
				case 1: convolve<1>(dst,src,dir); break;
				case 2: convolve<2>(dst,src,dir); break;
				case 3: convolve<3>(dst,src,dir); break;
				case 4: convolve<4>(dst,src,dir); break;
				case 5: convolve<5>(dst,src,dir); break;
				case 6: convolve<6>(dst,src,dir); break;
				case 7: convolve<7>(dst,src,dir); break;
				case 8: convolve<8>(dst,src,dir); break;
				default: cuvAssert(false);
			}
		}
		template<class DstV, class SrcV, class M>
		boost::ptr_vector<tensor<DstV,M,row_major> >
		convolve( const tensor<SrcV,M,row_major>& src,
			  const unsigned int& radius,
			  const separable_filter& filt ){
                        cuvAssert(src.ndim()==2);

			typedef tensor<DstV,M,row_major> result_type;
			cuvAssert(radius <= MAX_KERNEL_RADIUS);
			boost::ptr_vector<result_type> res;

			if(filt      == SP_GAUSS){
				const int kernel_w = 2*radius+1;
				cuv::tensor<float, host_memory_space> kernel(kernel_w);
				for(int i = 0; i < kernel_w; i++){
					float dist = (float)(i - radius) / (float)radius;
					kernel[i]=expf(- dist * dist / 2);
				}
				kernel /= cuv::sum(kernel);
				cuvSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), kernel.ptr(), kernel.memsize()) );
				res.push_back(new result_type(src.shape()));
				radius_dispatch(radius,res.back(),src,2);
			}else if(filt == SP_SOBEL){
				boost::ptr_vector<result_type> intermed;
				intermed = convolve<DstV>(src,radius,SP_GAUSS);

				const int kernel_w = 3;
				cuv::tensor<float, host_memory_space> kernel(kernel_w);
				kernel[0]=-0.5;
				kernel[1]= 0;
				kernel[2]= 0.5;
				cuvSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), kernel.ptr(), kernel.memsize()) );

				res.push_back(new result_type(src.shape()));
				radius_dispatch(1,res.back(),intermed.front(),0);

				res.push_back(new result_type(src.shape()));
				radius_dispatch(1,res.back(),intermed.front(),1);
			}
			return res;
		}
		
		// instantiations
#define INST(DSTV, SRCV,M, I) \
		template boost::ptr_vector<tensor<DSTV,M,row_major> > \
		convolve<DSTV,SRCV,M>( const tensor<SRCV,M,row_major>&, \
				                      const unsigned int&,                     \
				                      const separable_filter&);
		INST(float,float,dev_memory_space,unsigned int);
	} // namespace separable convolution
} // namespace cuv
