#include "hip/hip_runtime.h"
//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*

#include <cuv/basics/tensor.hpp>
#include <cuv/matrix_ops/matrix_ops.hpp>
#include <iostream>
#include <stdlib.h>
#include <dlfcn.h>
#include <vector>
#include "integral_image.hpp"

#define NUM_BANKS 16  
#define LOG_NUM_BANKS 4  
#define CONFLICT_FREE_OFFSET(x) ((x) >> NUM_BANKS + (x) >> (2 * LOG_NUM_BANKS))
#define PITCH(PTR,PITCH,Y,X) ((typeof(PTR))((char*)(PTR) + (PITCH)*(Y)) + (X))

namespace cuv
{
namespace integral_img{
	template<int BLOCK_SIZE,class DstT, class SrcT, class I>
		__global__ void scan_kernel(DstT *output, const SrcT *input, I width, I dpitch, I spitch) {
			__shared__ DstT sum1, sum2;

			I blockCol = blockIdx.x;
			I tdx      = threadIdx.x;
			if(tdx==0)
				sum1 = sum2 = 0;

			const SrcT* src = PITCH(input, spitch,blockCol,0);
			      DstT* dst = PITCH(output,dpitch,blockCol,0);

			      for(I i = 0; i< width; i+=BLOCK_SIZE) {
				      __syncthreads();
				      __shared__ DstT temp[BLOCK_SIZE];
				      I offset =1;

				      temp[2*tdx]   = (i+2*tdx  )<width ? src[i+2*tdx]   : 0;
				      temp[2*tdx+1] = (i+2*tdx+1)<width ? src[i+2*tdx+1] : 0;

				      for(I outerd = BLOCK_SIZE/2; outerd > 0; outerd /= 2) {
					      __syncthreads();
					      if(tdx < outerd) {
						      I ai      = offset*(2*tdx+1)-1;
						      I bi      = offset*(2*tdx+2)-1;
						      temp[bi] += temp[ai];
					      }
					      offset   *= 2;

				      }
				      if(tdx == 0) {
					      sum2               = temp[BLOCK_SIZE-1];
					      temp[BLOCK_SIZE-1] = 0;
				      }
				      for(I innerd = 1; innerd < BLOCK_SIZE; innerd *= 2) {
					      offset >>= 1;
					      __syncthreads();
					      if(tdx < innerd) {
						      I ai      = offset*(2*tdx+1)-1;
						      I bi      = offset*(2*tdx+2)-1;
						      DstT t    = temp[ai];
						      temp[ai]  = temp[bi];
						      temp[bi] += t;
					      }
				      }
				      __syncthreads();

				      if(i+2*tdx  <width) dst[i+2*tdx]   = temp[2*tdx]  +sum1;
				      if(i+2*tdx+1<width) dst[i+2*tdx+1] = temp[2*tdx+1]+sum1;
				      if(tdx == 0) 
					      sum1 += sum2;
			      }
		}

	template<class V,class W, class L>
		void scan(cuv::tensor<V, dev_memory_space, L>& dst, const cuv::tensor<W, dev_memory_space, L>& src) {
			scan_kernel<256,V><<<src.shape()[0], 128>>>(dst.ptr(), src.ptr(), src.shape(1), (unsigned int)dst.stride(0), (unsigned int)src.stride(0));
			cuvSafeCall(hipDeviceSynchronize());
		}

	template<class V,class W, class L>
		void scan(cuv::tensor<V, host_memory_space, L>& dst, const cuv::tensor<W, host_memory_space, L>& src)
		{
			const W* src_ptr = src.ptr();
			V* dst_ptr = dst.ptr();
			for(int i = 0; i<src.shape()[0]; i++) {
				*dst_ptr = 0;
				dst_ptr++;
				for(int j =0; j< src.shape()[1]-1; j++) {
					*dst_ptr = *(dst_ptr-1) + *(src_ptr);
					dst_ptr++;
					src_ptr++;
				}
				src_ptr++;
			}
		}

	template<class V,class W, class T, class M>
		void integral_image(cuv::tensor<V, T, M>& dst, const cuv::tensor<W, T, M>& src)
		{
			cuvAssert(src.ndim()==2);
			cuvAssert(src.shape()[0]==dst.shape()[1]);
			cuvAssert(src.shape()[1]==dst.shape()[0]);
			tensor<V,T,M> temp (src.shape(),pitched_memory_tag());
			tensor<V,T,M> temp1(dst.shape(),pitched_memory_tag());

			scan(temp, src);
			transpose(temp1, temp);
			scan(dst, temp1);
		}
#define TENS(V,M,L) \
        cuv::tensor<V,M,L>
#define INSTANTIATE_INTIMG(V,W,M,L) \
	template void integral_image(TENS(V, M, L)&, const TENS(W, M, L)&);\
	template void scan(TENS(V          , M, L)&, const TENS(W, M, L)&);

	INSTANTIATE_INTIMG(float, float        , host_memory_space, row_major);
	INSTANTIATE_INTIMG(float, unsigned char, host_memory_space, row_major);
	INSTANTIATE_INTIMG(float, float        , dev_memory_space , row_major);
	INSTANTIATE_INTIMG(float, unsigned char, dev_memory_space , row_major);


} // namespace integral image
} // namespace cuv



