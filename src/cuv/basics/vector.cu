//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*





#include <iostream>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <cuv/tools/cuv_general.hpp>
#include <cuv/basics/vector.hpp>

namespace cuv{

template <class value_type, class index_type>
struct allocator<value_type,index_type,dev_memory_space>{
	void alloc( value_type** ptr, index_type size) const{
		cuvSafeCall(hipMalloc(ptr, sizeof(value_type)*size));
	}
	void dealloc( value_type** ptr) const {
		cuvSafeCall(hipFree((void*)*ptr));
		*ptr = NULL;
	}
};

template <class value_type, class index_type>
struct allocator<const value_type,index_type,dev_memory_space>{
	void alloc(const value_type** ptr, index_type size) const{
		cuvAssert(false);
	}
	void dealloc(const value_type** ptr)const {
		cuvAssert(false);
	}
};

template <class value_type, class index_type>
void entry_set(value_type* ptr, index_type idx, value_type val, dev_memory_space) {
	thrust::device_ptr<value_type> dev_ptr(ptr);
	dev_ptr[idx]=val;
}

template <class value_type, class index_type>
value_type entry_get(const value_type* ptr, index_type idx, dev_memory_space) {
	const thrust::device_ptr<const value_type> dev_ptr(ptr);
	return (value_type) *(dev_ptr+idx);
}


#define VECTOR_INST(T,I) \
template struct allocator<T, I, dev_memory_space>; \
template struct allocator<const T, I, dev_memory_space>; \
template void entry_set(T*, I, T, dev_memory_space); \
template T entry_get(const T*, I, dev_memory_space); \

VECTOR_INST(float, unsigned int);
VECTOR_INST(unsigned char, unsigned int);
VECTOR_INST(signed char, unsigned int);
VECTOR_INST(int, unsigned int);
VECTOR_INST(unsigned int, unsigned int);


}; // cuv
