#include "hip/hip_runtime.h"
#include <cuv/basics/tensor.hpp>
#include <cuv/basics/cuda_array.hpp>

texture<float,         2, hipReadModeElementType> cuda_array_tex_float; 
texture<unsigned char, 2, hipReadModeElementType> cuda_array_tex_uchar; 

template<class T> struct texref{ };
template<> struct texref<float>{
	typedef texture<float, 2, hipReadModeElementType> type;
	static type& get(){ return cuda_array_tex_float; }; 
};
template<> struct texref<unsigned char>{
	typedef texture<unsigned char, 2, hipReadModeElementType> type;
	static type& get(){ return cuda_array_tex_uchar; }; 
};

namespace cuv{

template<class T> struct single_to_4{};
template<>        struct single_to_4<float>        {typedef float4 type;};
template<>        struct single_to_4<unsigned char>{typedef uchar4 type;};

template<class V,class S, class I>
void cuda_array<V,S,I>::alloc(){
	cuvAssert(m_ptr==NULL);
	typedef typename single_to_4<V>::type V4;
	hipChannelFormatDesc channelDesc  = hipCreateChannelDesc<V>();
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<V4>();
	switch(m_dim){
		case 1:
			hipMallocArray(&m_ptr, &channelDesc, m_width, m_height);
			break;
		case 4:
			hipMallocArray(&m_ptr, &channelDesc4, m_width, m_height);
			break;
		default:
			cuvAssert(false);
	}
	checkCudaError("hipMallocArray");
}
template<class V,class S, class I>
void cuda_array<V,S,I>::dealloc(){
	if(m_ptr!=NULL){
		hipFreeArray(m_ptr);
		m_ptr = NULL;
	}
}

/*
 *template<class V,class S, class I>
 *void cuda_array<V,S,I>::bind()const{
 *    cuvAssert(m_ptr!=NULL);
 *    typedef typename texref<V>::type textype;
 *    textype& tex = texref<V>::get();
 *    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<V>();
 *    tex.normalized = false;
 *    tex.filterMode = hipFilterModePoint;
 *    tex.addressMode[0] = hipAddressModeClamp;
 *    tex.addressMode[1] = hipAddressModeClamp;
 *    hipBindTextureToArray(tex, m_ptr, channelDesc);
 *    checkCudaError("hipBindTextureToArray");
 *}
 */

/*
 *template<class V,class S, class I>
 *void cuda_array<V,S,I>::unbind()const{
 *    cuvAssert(m_ptr!=NULL);
 *    typedef typename texref<V>::type textype;
 *    textype& tex = texref<V>::get();
 *    hipUnbindTexture(tex);
 *    checkCudaError("hipUnbindTexture");
 *}
 */


#define CA cuda_array<V,S,I>
template<class V,class S, class I>
void cuda_array<V,S,I>::assign(const tensor<V, host_memory_space, row_major>& src){
        cuvAssert(src.shape().size()==2);
	cuvAssert(src.ptr()!=NULL);
	cuvAssert(src.shape()[1]/m_dim == m_width);
	cuvAssert(src.shape()[0]       == m_height);
	hipMemcpyToArray(ptr(), 0, 0, src.ptr(), src.memsize(), hipMemcpyHostToDevice);
	checkCudaError("hipMemcpyToArray");
}
template<class V,class S, class I>
void cuda_array<V,S,I>::assign(const tensor<V,dev_memory_space,row_major>& src){
        cuvAssert(src.shape().size()==2);
	cuvAssert(src.ptr()!=NULL);
	cuvAssert(src.shape()[1]/m_dim  == m_width);
	cuvAssert(src.shape()[0]        == m_height);
	hipMemcpyToArray(ptr(), 0, 0, src.ptr(), src.memsize(), hipMemcpyDeviceToDevice);
	checkCudaError("hipMemcpyToArray");
}


template<class V,class S, class I>
V
cuda_array<V,S,I>::operator()(const I& i, const I& j)const{
	cuvAssert(false); // only works with (broken) bind active!
	return 0;
}

// explicit instantiations
#define INST(V,M,I) \
	template void cuda_array<V,M,I>::alloc();   \
	template void cuda_array<V,M,I>::dealloc();   \
	template void cuda_array<V,M,I>::assign(const tensor<V,host_memory_space,row_major>&);   \
	template void cuda_array<V,M,I>::assign(const tensor<V,dev_memory_space,row_major>&);    \
	template V cuda_array<V,M,I>::operator()(const I&, const I&)const;   

INST(float,dev_memory_space,unsigned int);
INST(unsigned char,dev_memory_space,unsigned int);

}
