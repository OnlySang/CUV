//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*





#include <iostream>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <cuv/tools/cuv_general.hpp>
#include <cuv/basics/linear_memory.hpp>

#define checkSuccess(X) \
	cuvAssert((X)==hipSuccess)

namespace cuv{

template <class value_type, class index_type>
struct allocator<value_type,index_type,dev_memory_space>{
	void alloc2d( value_type** ptr, index_type& pitch, index_type height, index_type width ) const{
		size_t p;
		checkSuccess(hipMallocPitch(ptr, &p, sizeof(value_type)*width, height));
		pitch = p;
	}
	void alloc( value_type** ptr, index_type size) const{
		checkSuccess(hipMalloc(ptr, sizeof(value_type)*size));
	}
	void dealloc( value_type** ptr) const {
		checkSuccess(hipFree((void*)*ptr));
		*ptr = NULL;
	}
	void alloc(const value_type** ptr, index_type size) const{
	       cuvAssert(false);
	}
	void dealloc(const value_type** ptr)const {
	       cuvAssert(false);
	}
	void copy(value_type* dst, const value_type*src,index_type size, host_memory_space){
		checkSuccess(hipMemcpy( dst, src, size*sizeof( value_type ), hipMemcpyHostToDevice ));
	}
	void copy(value_type* dst, const value_type*src,index_type size, dev_memory_space){
		checkSuccess(hipMemcpy( dst, src, size*sizeof( value_type ), hipMemcpyDeviceToDevice ));
	}
	void copy2d(value_type* dst, const value_type*src,index_type dpitch, index_type spitch, index_type h, index_type w, host_memory_space){
		checkSuccess(hipMemcpy2D(dst,dpitch,src,spitch,w*sizeof(value_type),h,hipMemcpyHostToDevice));
	}
	void copy2d(value_type* dst, const value_type*src,index_type dpitch, index_type spitch, index_type h, index_type w, dev_memory_space){
		checkSuccess(hipMemcpy2D(dst,dpitch,src,spitch,w*sizeof(value_type),h,hipMemcpyDeviceToDevice));
	}
};

template<class V,class I>
void
allocator<V,I,host_memory_space>::alloc2d(V** ptr, I& pitch, I height, I width)const{
	pitch = width*sizeof(V);
	*ptr  = new V[height*width];
}
template<class V,class I>
void
allocator<V,I,host_memory_space>::copy(V*dst, const V*src,I size,dev_memory_space){
	checkSuccess(hipMemcpy( dst, src, size*sizeof( V ), hipMemcpyDeviceToHost ));
}

template<class V,class I>
void
allocator<V,I,host_memory_space>::copy2d(V* dst, const V*src,I dpitch, I spitch, I h, I w, dev_memory_space){
	checkSuccess(hipMemcpy2D(dst,dpitch,src,spitch,w*sizeof(V),h,hipMemcpyDeviceToHost));
}

template<class V,class I>
void
allocator<V,I,host_memory_space>::copy2d(V* dst, const V*src,I dpitch, I spitch, I h, I w, host_memory_space){
	checkSuccess(hipMemcpy2D(dst,dpitch,src,spitch,w*sizeof(V),h,hipMemcpyHostToHost));
}

template <class value_type, class index_type>
void entry_set(value_type* ptr, index_type idx, value_type val, dev_memory_space) {
	thrust::device_ptr<value_type> dev_ptr(ptr);
	dev_ptr[idx]=val;
}

template <class value_type, class index_type>
value_type entry_get(const value_type* ptr, index_type idx, dev_memory_space) {
	const thrust::device_ptr<const value_type> dev_ptr(ptr);
	return (value_type) *(dev_ptr+idx);
}

#define VECTOR_INST(T,I) \
template struct allocator<T, I, dev_memory_space>; \
template struct allocator<T, I, host_memory_space>; \
template void entry_set(T*, I, T, dev_memory_space); \
template T entry_get(const T*, I, dev_memory_space); \

VECTOR_INST(float, unsigned int);
VECTOR_INST(unsigned char, unsigned int);
VECTOR_INST(signed char, unsigned int);
VECTOR_INST(int, unsigned int);
VECTOR_INST(unsigned int, unsigned int);


}; // cuv
