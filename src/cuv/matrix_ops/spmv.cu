#include "hip/hip_runtime.h"
//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*





#include <iostream>
#include <boost/any.hpp>
#include <cuv/basics/dia_matrix.hpp>
#include <cuv/matrix_ops/matrix_ops.hpp>
#include <cuv/tools/texture.h>
#include <boost/preprocessor/arithmetic/inc.hpp>
#include <boost/preprocessor/cat.hpp>
#include <boost/preprocessor/repetition/repeat.hpp>
#include <boost/preprocessor/seq/for_each_product.hpp>
#include <boost/preprocessor/seq/to_tuple.hpp>


using namespace std;

// stuff from NVIDIA SDK
#define DIVIDE_INTO(x,y) ((x + y - 1)/y)
#define small_grid_thread_id(void) ((__umul24(blockDim.x, blockIdx.x) + threadIdx.x))
#define large_grid_thread_id(void) ((__umul24(blockDim.x,blockIdx.x + __umul24(blockIdx.y,gridDim.x)) + threadIdx.x))
#define large_grid_thread_num(void) ((__umul24(blockDim.x,gridDim.x + __umul24(blockDim.y,gridDim.y))))

#define MAX_NUM_IMGS_AT_ONCE 14
#define SEQ_ROW_FACT         1
#define SPMM_BLOCK_SIZE      256


namespace cuv{
	namespace spmv_impl{
		/*
		 *  For a given number of blocks, return a 2D grid large enough to contain them
		 *  FROM NVIDIA SDK
		 */
		dim3 make_large_grid(const unsigned int num_blocks){
			if (num_blocks <= 65535){
				return dim3(num_blocks);
			} else {
				unsigned int side = (unsigned int) ceil(sqrt((double)num_blocks));
				return dim3(side,side);
			}
		}

		dim3 make_large_grid(const unsigned int num_threads, const unsigned int blocksize){
			const unsigned int num_blocks = DIVIDE_INTO(num_threads, blocksize);
			if (num_blocks <= 65535){
				//fits in a 1D grid
				return dim3(num_blocks);
			} else {
				//2D grid is required
				const unsigned int side = (unsigned int) ceil(sqrt((double)num_blocks));
				return dim3(side,side);
			}
		}

// this file is generated using a perl-script from spmv_kernel.cuh
#include "spmv_dia_kernel_inst.cuh"

		template <typename value_type, typename index_type>
			void spmv_dia_device(const dia_matrix<value_type,dev_memory_space,index_type>& A, 
					const tensor<value_type,dev_memory_space>& v, 
					tensor<value_type,dev_memory_space>& dst, 
					char transA,
					const value_type& factAv,
					const value_type& factC)
			{
				const unsigned int toff = bind_x(v.ptr(), v.size());
				spmm_device_dia_dispatch(A,v,dst,transA,factAv,factC,toff);
				cuvSafeCall(hipDeviceSynchronize());
				unbind_x(v.ptr());
			}


		/*template <bool transA, typename value_type, typename index_type>*/
		/*    void spmv_dia_tex_device(const dia_matrix<value_type,dev_memory_space,index_type>& A, */
		/*            const tensor<value_type,dev_memory_space>& v, */
		/*            tensor<value_type,dev_memory_space>& dst)*/
		/*    {*/
		/*        const unsigned int BLOCK_SIZE = 256;*/
		/*        const dim3 grid = make_large_grid(A.shape()[0],BLOCK_SIZE);*/

		/*        cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals*/

		/*        bind_x(v.ptr());*/

		/*        if(!transA){*/
		/*            const unsigned int BLOCK_SIZE = 256;*/
		/*            const dim3 grid = make_large_grid(A.shape()[0],BLOCK_SIZE);*/
		/*            cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals*/
		/*            spmv_dia_kernel<value_type, index_type, BLOCK_SIZE, true> <<<grid, BLOCK_SIZE>>> (A.shape()[0], A.shape()[1],  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec().ptr(), v.ptr(), dst.ptr());*/
		/*        }else{*/
		/*            const unsigned int BLOCK_SIZE = 256;*/
		/*            const dim3 grid = make_large_grid(A.shape()[1],BLOCK_SIZE);*/
		/*            cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals*/
		/*            spmv_dia_kernel_trans<value_type, index_type, BLOCK_SIZE, true> <<<grid, BLOCK_SIZE>>> (A.shape()[0], A.shape()[1],  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec().ptr(), v.ptr(), dst.ptr());*/
		/*        }*/

		/*        unbind_x(v.ptr());*/
		/*    }*/
		template<class value_type>
			void spmv(tensor<value_type,dev_memory_space>& dst, const dia_matrix<value_type,dev_memory_space>& A, const tensor<value_type,dev_memory_space>& v, char transA, const float& factAv, const float& factC){
				// TODO: find a good assert
				/*if(transA=='t'){*/
					/*cuvAssert(A.shape()[1] == dst.size());*/
				/*}else{*/
					/*cuvAssert(A.shape()[0] == dst.size());*/
				/*}*/
				spmv_dia_device(A,v,dst,transA,factAv,factC);
			}


		/****************************************************************
		 *  Host Code
		 ****************************************************************/
		template<class value_type, class index_type>
			void spmv(tensor<value_type,host_memory_space>& dst, const dia_matrix<value_type,host_memory_space,index_type>& A, const tensor<value_type,host_memory_space>& v, char transA, const float& factAv, const float& factC){
				const tensor<int,host_memory_space>& offsets = A.get_offsets();
				const int num_diags             = A.num_dia();
				const int A_h                   = A.h();
				const int A_w                   = A.w();
				const int A_stride              = A.stride();
				index_type max_dst = ((transA=='t') ? A_w : A_h);
				if(factC==0.f)
					for(int i=0;i<max_dst;i++) dst[i]=0;
				else
					for(int i=0;i<max_dst;i++) dst[i]=dst[i] * factC;
				const int rf = A.row_fact();
				if(transA == 't'){
					cuvAssert(A_h == v.size());
					cuvAssert(A_w == dst.size());
					for(index_type i = 0; i < num_diags; i++){
						const int k = offsets[i];  //diagonal offset

						const index_type i_start =  1 * std::max((int)0, k);
						const index_type j_start = rf * std::max((int)0,-k); // the matrix is now _wider_ than high --> stretch columns!

						//number of elements to process
						const index_type N = std::min((A_h - j_start)/rf, A_w - i_start);

						const value_type * d_ = A.vec().ptr() + i*A_stride + j_start;
						const value_type * x_ = v.ptr() + j_start;
						value_type * y_ = dst.ptr() + i_start;

						for(index_type n = 0; n < N; n++,y_++){
							for(int k=0;k<rf;k++,x_++,d_++)
								*y_ += factAv * *d_ * *x_;
						}
					}
				}else{
					cuvAssert(A_w == v.size());
					cuvAssert(A_h == dst.size());
					for(index_type i = 0; i < num_diags; i++){
						const int k = offsets[i];  //diagonal offset

						const index_type i_start = rf*std::max((int)0,-k);
						const index_type j_start =  1*std::max((int)0, k);

						//number of elements to process
						const index_type N = std::min(A_h - i_start, rf*(A_w - j_start));

						const value_type * d_ = A.vec().ptr() + i*A_stride + i_start;
						const value_type * x_ = v.ptr() + j_start;
						value_type * y_ = dst.ptr() + i_start;

						for(index_type n = 0; n < N; n++){
							*y_++ += factAv * *d_++ * x_[n/rf];
						}
					}
				}
			}
	}

	template<>
		void prod(tensor<float,host_memory_space,column_major>& dst,
				  const dia_matrix<float,host_memory_space>&                  A,
				  const tensor<float,host_memory_space,column_major>&   B,
				  char transA,
				  char transB,
				  const float& factAB,
				  const float& factC){
                        cuvAssert(dst.shape().size()==2);
                        cuvAssert(B.shape().size()==2);
			cuvAssert(transB == 'n');
			cuvAssert(dst.shape()[1] == B.shape()[1]);
			for(int i=0;i<dst.shape()[1];i++){
				tensor<float,host_memory_space> dst_v(indices[index_range(0,dst.shape()[0])], dst.ptr()+i*dst.shape()[0]);
				tensor<float,host_memory_space> src_v(indices[index_range(0,B.shape()[0])],   B.ptr()+i*B.shape()[0]);
				spmv(dst_v,A,src_v,transA,factAB,factC);
			}
		}
	template<>
		void prod(tensor<float,dev_memory_space,column_major>& dst,
				  const dia_matrix<float,dev_memory_space>&                  A,
				  const tensor<float,dev_memory_space,column_major>&   B,
				  char transA,
				  char transB,
				  const float& factAB,
				  const float& factC){
                        cuvAssert(dst.shape().size()==2);
                        cuvAssert(B.shape().size()==2);
			cuvAssert(transB == 'n');
			cuvAssert(dst.shape()[1] == B.shape()[1]);
			cuvAssert(dst.ptr());
			cuvAssert(A.vec_ptr());
			cuvAssert(B.ptr());
			if(transA=='t'){
				cuvAssert(A.w() == dst.shape()[0]);
			}else{
				cuvAssert(A.h() == dst.shape()[0]);
			}
			const int num_at_same_time = min(MAX_NUM_IMGS_AT_ONCE, B.shape()[1]);
			for(int i=0; i<dst.shape()[1]; i += num_at_same_time){
				tensor<float,dev_memory_space> dst_v(indices[index_range(0,dst.shape()[0] * min(dst.shape()[1]-i,num_at_same_time))], dst.ptr()+i*dst.shape()[0]);
				tensor<float,dev_memory_space> src_v(indices[index_range(0,B.shape()[0]* min(B.shape()[1]-i,  num_at_same_time))], B.ptr()+i*B.shape()[0]);
				spmv(dst_v,A,src_v,transA,factAB,factC);
			}
		}
      template<class __value_type, class __memory_space_type>
              void spmv(tensor<__value_type, __memory_space_type>& dst, const dia_matrix<__value_type, __memory_space_type>& A, const tensor<__value_type, __memory_space_type>& v,const  char transA, const float& factAv, const float& factC){
			spmv_impl::spmv(dst,A,v,transA,factAv,factC);
		}
	template void spmv<float,host_memory_space>(tensor<float,host_memory_space>&dst, const dia_matrix<float,host_memory_space>& A, const tensor<float,host_memory_space>& v, char, const float&, const float&);
	template void spmv<float,dev_memory_space>(tensor<float,dev_memory_space>&dst, const dia_matrix<float,dev_memory_space>& A, const tensor<float,dev_memory_space>& v, char, const float&, const float&);
}
