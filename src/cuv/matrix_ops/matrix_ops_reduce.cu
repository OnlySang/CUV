#include "hip/hip_runtime.h"
//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*

#include <stdio.h>
#include <stdexcept>

#include <cuv/tools/cuv_general.hpp>
#include <cuv/tools/meta_programming.hpp>
#include <cuv/tensor_ops/functors.hpp>
#include <cuv/tensor_ops/tensor_ops.hpp>
#include <cuv/matrix_ops/matrix_ops.hpp>

template<int BLOCK_DIM, class T, class V, class RF>
__global__
void reduce_to_col_kernel(const T* matrix, V* vector, const unsigned int nCols, const unsigned int nRows,
		const T factNew, const T factOld, RF rf, const T init_value) {
	// reduce to column for column major matrices, reduce to row for row major matrices

	typedef cuv::reduce_functor_traits<typename RF::result_value_functor_type> functor_traits;
	typedef typename cuv::unconst<T>::type unconst_value_type;

	extern __shared__ unsigned char ptr[]; // need this intermediate variable for nvcc :-(
	unconst_value_type* values = (unconst_value_type*) ptr;
	unsigned int* indices = (unsigned int*)(values + BLOCK_DIM*BLOCK_DIM);
	const unsigned int tx = threadIdx.x;
	const unsigned int bx = blockIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int by = blockIdx.y;

	const int row_idx = by * gridDim.x * blockDim.x +   	// offset according to y index in grid
						bx * blockDim.x +  					// offset according to block index
						tx;									// offset in block

	if (row_idx >= nRows)
		return;
	const unsigned int off = blockDim.y;

	unconst_value_type sum = init_value;
	unsigned int arg_index = 0; // for storing indeces of maxima/minima for arg functors

	for (unsigned int my = ty; my < nCols; my += off) {
		T f = matrix[my * nRows + row_idx ];
		rf.rv(sum,arg_index,f,my);
		//sum=rf(sum,f);
	}

	values[ty*BLOCK_DIM+tx] = sum;
	if(functor_traits::returns_index)
		indices[ty*BLOCK_DIM+tx] = arg_index;

	__syncthreads();

	for (unsigned int offset = blockDim.y / 2; offset > 0; offset >>=1) {
		if (ty < offset) {
			const unsigned int v = ty+offset;
			rf.rr(
					  values [ty*BLOCK_DIM+tx],
					  indices[ty*BLOCK_DIM+tx],
					  values [v *BLOCK_DIM+tx],
					  indices[v *BLOCK_DIM+tx]);
		}
		__syncthreads();
	}
	
	if (ty == 0) {
		if (functor_traits::returns_index)
			vector[row_idx] = indices[tx];
		else
			if(factOld != 0.f){
				vector[row_idx] = vector[row_idx] * factOld + values[tx] * factNew;
			}else{
				vector[row_idx] = values[tx] * factNew;
			}
	}
}

template<int BLOCK_DIM, class T, class V, class RF>
__global__
void reduce_to_row_kernel(const T* matrix, V* vector, const unsigned int nCols, const unsigned int nRows,
		const T factNew, const T factOld, RF rf, const T init_value) {
	// reduce to row for column major matrices, reduce to column for row major matrices
	typedef cuv::reduce_functor_traits<typename RF::result_value_functor_type> functor_traits;
	typedef typename cuv::unconst<T>::type unconst_value_type;

	extern __shared__ float sptr[]; // need this intermediate variable for nvcc :-(
	unconst_value_type* values = (unconst_value_type*) sptr;
	unsigned int* indices                  = (unsigned int*)(values + BLOCK_DIM*BLOCK_DIM);
	const unsigned int tx = threadIdx.x; // blockIdx.x is always 0
	const unsigned int by = blockIdx.y; //threadIdx.y is always 0, blockDim.y is always 1!
	const unsigned int off = blockDim.x;
	
	values[tx] = init_value;
	if(functor_traits::returns_index)
		indices[tx] = 0;

	for (unsigned int my = tx; my < nRows; my += off) {
		const T f = matrix[by * nRows + my];
		rf.rv(values[tx],indices[tx],f,my);
	}
	__syncthreads();

	for (unsigned int offset = BLOCK_DIM*BLOCK_DIM/2; offset > 0; offset>>=1) {
		const unsigned int v = tx+offset;
		if (tx < offset)
			rf.rr(values[tx],indices[tx],values[v],indices[v]);
		__syncthreads();
	}
	__syncthreads();
	if (tx == 0) {
		if (functor_traits::returns_index)
			vector[by] = indices[0];
		else{
			if(factOld != 0){
				vector[by] = vector[by]
					* factOld + values[0] * factNew;
			}else{
				vector[by] = values[0] * factNew;
			}
		}
	}
}

template<unsigned int BLOCK_DIM, class I, class T>
__global__
void argmax_row_kernel(I* vector, const T* matrix, unsigned int nCols, unsigned int nRows) {
	__shared__ I shIdx[BLOCK_DIM]; // index of the maximum
	__shared__ T shVal[BLOCK_DIM]; // value

	const unsigned int tx = threadIdx.x;
	const unsigned int by = blockIdx.x + gridDim.x*blockIdx.y;
	if (by >= nCols)
	   return;
	const unsigned int off = blockDim.x;

	unsigned int idx = by * nRows + tx;
	shVal[tx] = (tx<nRows) ? matrix[idx] : (T) INT_MIN;
	shIdx[tx] = (tx<nRows) ? tx          : 0;

	for (unsigned int my = tx + off; my < nRows; my += off) {
	   idx += off;
	   T f = matrix[idx];

	   if (f > shVal[tx]) {
		  shVal[tx] = f;
		  shIdx[tx] = my;
	   }
	}
	__syncthreads();

	for (unsigned int offset = BLOCK_DIM/2 ; offset > 0; offset/=2) {
	   if (tx < offset) {
		   const unsigned int v = tx+offset;
		   if (shVal[tx] < shVal[v]) {
			   shVal[tx] = shVal[v];
			   shIdx[tx] = shIdx[v];
		   }
	   }
	}
	__syncthreads();

	if (tx == 0)
	   vector[by] = shIdx[0];
}

namespace cuv {

namespace reduce_impl {
	template<int dim, class __memory_space_type>
	       struct reduce{};

	template<>
	struct reduce<1, dev_memory_space>{
                template<class __value_type, class __value_type2, class __memory_layout_type, class RF, class S>
	       	void operator()(tensor<__value_type,dev_memory_space> &v,const  dense_matrix<__value_type2,dev_memory_space,__memory_layout_type> &m,const  S & factNew,const  S & factOld, RF rf)const{
                    cuvAssert(m.ptr() != NULL);
                    cuvAssert(m.h() == v.size());
                    static const int BLOCK_DIM = 16;
                    const int blocks_needed = ceil((float)m.h()/(BLOCK_DIM));
                    int grid_x =0, grid_y=0;

                    // how to handle grid dimension constraint
                    if (blocks_needed <= 65535){
                            grid_x = blocks_needed;
                            grid_y = 1;
                    }else{
                            // try to avoid large noop blocks by adjusting x and y dimension to nearly equal size
                            grid_x = ceil(sqrt(blocks_needed));
                            grid_y = ceil((float)blocks_needed/grid_x);
                    }
                    dim3 grid(grid_x, grid_y);
                    dim3 threads(BLOCK_DIM,BLOCK_DIM);
                    typedef __value_type matval_t;
                    typedef typename tensor<__value_type,dev_memory_space>::value_type vecval_t;
                    unsigned int mem = sizeof(matval_t) * BLOCK_DIM*BLOCK_DIM ;

                    typedef reduce_functor_traits<typename RF::result_value_functor_type> traits_type;
                    if(traits_type::returns_index)
                            mem += sizeof(vecval_t)*BLOCK_DIM*BLOCK_DIM;
                    reduce_to_col_kernel<BLOCK_DIM><<<grid,threads,mem>>>(m.ptr(),v.ptr(),m.w(),m.h(),__value_type2(factNew),__value_type2(factOld),rf,__value_type2(traits_type::init_value()));
                    cuvSafeCall(hipDeviceSynchronize());
	}};

	template<>
	struct reduce<0, dev_memory_space>{
                template<class __value_type, class __value_type2, class __memory_layout_type, class RF, class S>
	       	void operator()(tensor<__value_type,dev_memory_space> &v,const  dense_matrix<__value_type2,dev_memory_space,__memory_layout_type> &m,const S & factNew,const  S & factOld, RF rf)const{
		cuvAssert(m.ptr() != NULL);
		cuvAssert(m.w() == v.size());
		static const int BLOCK_DIM = 16;
		dim3 grid(1, m.w());
		dim3 threads(BLOCK_DIM*BLOCK_DIM,1);

		typedef __value_type matval_t;
		typedef typename tensor<__value_type,dev_memory_space>::value_type vecval_t;
		unsigned int mem = sizeof(matval_t) * threads.x*threads.y;
		typedef reduce_functor_traits<typename RF::result_value_functor_type> traits_type;
		if(traits_type::returns_index)
			mem += sizeof(vecval_t)*threads.x*threads.y;

                reduce_to_row_kernel<BLOCK_DIM><<<grid,threads,mem>>>(m.ptr(),v.ptr(),m.w(),m.h(),__value_type2(factNew),__value_type2(factOld),rf,__value_type2(traits_type::init_value()));
		cuvSafeCall(hipDeviceSynchronize());
	}};

	template<int dim>
	struct reduce<dim, host_memory_space>{
                template<class __value_type, class __value_type2, class __memory_layout_type, class RF, class S>
	       	void operator()(tensor<__value_type,host_memory_space> &v,const  dense_matrix<__value_type2,host_memory_space,__memory_layout_type> &m,const S & factNew,const S & factOld, RF rf)const{
		typedef __value_type2 V;
		typedef typename tensor<__value_type,host_memory_space>::value_type V2;
		typedef typename dense_matrix<__value_type,host_memory_space,__memory_layout_type>::index_type I;
		typedef typename unconst<V>::type unconstV;
		typedef cuv::reduce_functor_traits<typename RF::result_value_functor_type> functor_traits;

		cuvAssert(m.ptr() != NULL);
		// assert that vector has correct length
		if (dim==0) cuvAssert(v.size()==m.w());
		if (dim==1) cuvAssert(v.size()==m.h());

		const __value_type2 * A_ptr                         = m.ptr();

		// indices: only needed when arg-max/arg-min etc used
		tensor<I,host_memory_space>* indices = NULL;
		I* indices_ptr                         = NULL;
		if(functor_traits::returns_index){
			indices         =  new tensor<I,host_memory_space>(v.size());
			indices_ptr     =  indices->ptr();
			memset(indices_ptr,indices->memsize(), 0);
		}
		I*const indices_begin = indices_ptr;
		I*const indices_end   = indices_ptr + v.size();

		// values: the new values that are to be combined with v using fact
		tensor<unconstV,host_memory_space> values(v.size());
		unconstV* values_ptr                   = values.ptr();
		V*const values_end                     = values_ptr + values.size();
		while(values_ptr != values_end) 
			*values_ptr++ =functor_traits::init_value(); 
		values_ptr = values.ptr();      // reset pointers to begining of vector

		if (dim==0){
			// apply reduce functor along columns
			for(;values_ptr!=values_end; values_ptr++, indices_ptr++) {
				for(unsigned int j=0; j<m.h(); j++, A_ptr++)
					rf.rv(*values_ptr,*indices_ptr,*A_ptr,j);
			}
		}
		else if(dim==1){
			// apply reduce functor along rows
			for(I i=0;i<m.w();i++) {
				values_ptr  = values.ptr();
				indices_ptr = indices_begin;
				for(; values_ptr!=values_end;A_ptr++,values_ptr++,indices_ptr++) 
					rf.rv(*values_ptr,*indices_ptr,*A_ptr,i);
			}
		}else{
			cuvAssert(false);
		}

		// reset pointers to begining of vectors
		values_ptr  = values.ptr();
		indices_ptr = indices_begin;

		// put result into v via v_ptr.
		V2* v_ptr   = v.ptr();
		if (!functor_traits::returns_index){ 
			if (factOld!=0){
				while(values_ptr!=values_end) 
					*v_ptr   = factOld * *v_ptr++  + factNew * *values_ptr++;
			}else
				while(values_ptr!=values_end) 
					*v_ptr++ = factNew * *values_ptr++;
		}
		else{
			while(indices_ptr!=indices_end) 
				*v_ptr++ = *indices_ptr++;
			delete indices;
		}
	}};

        template<int dimension, class __value_type, class __value_type2, class __memory_space_type, class __memory_layout_type, class S>
	void reduce_switch(tensor<__value_type,__memory_space_type>&v, const dense_matrix<__value_type2,__memory_space_type,__memory_layout_type>& m, reduce_functor rf, const S& factNew, const S& factOld) {
		typedef __value_type2 const_mat_val;
		typedef typename dense_matrix<__value_type2,__memory_space_type,__memory_layout_type>::index_type mat_ind;
		typedef typename dense_matrix<__value_type2,__memory_space_type,__memory_layout_type>::memory_space_type mat_mem;
		typedef typename tensor<__value_type,__memory_space_type>::value_type vec_val;
		typedef typename tensor<__value_type,__memory_space_type>::index_type vec_ind;
		typedef typename unconst<const_mat_val>::type mat_val;
		switch(rf) {
			case RF_ADD:
			reduce_impl::reduce<dimension,mat_mem>()(v,m,factNew,factOld,make_reduce_functor(bf_plus<vec_val,vec_val,mat_val>(),bf_plus<vec_val,vec_val,vec_val>()));
			break;
			case RF_ADD_SQUARED:
			reduce_impl::reduce<dimension,mat_mem>()(v,m,factNew,factOld,make_reduce_functor(bf_add_square<vec_val,vec_val,mat_val>(),bf_plus<vec_val,vec_val,vec_val>()));
			break;
			case RF_MIN:
			reduce_impl::reduce<dimension,mat_mem>()(v,m,factNew,factOld,make_reduce_functor(bf_min<mat_val,mat_val,mat_val>()));
			break;
			case RF_MAX:
			reduce_impl::reduce<dimension,mat_mem>()(v,m,factNew,factOld,make_reduce_functor(bf_max<mat_val,mat_val,mat_val>()));
			break;
			case RF_ARGMAX:
			reduce_impl::reduce<dimension,mat_mem>()(v,m,factNew,factOld,make_arg_reduce_functor(reduce_argmax<mat_val,mat_ind>()));
			break;
			case RF_ARGMIN:
			reduce_impl::reduce<dimension,mat_mem>()(v,m,factNew,factOld,make_arg_reduce_functor(reduce_argmin<mat_val,mat_ind>()));
			break;
			case RF_MULT:
			reduce_impl::reduce<dimension,mat_mem>()(v,m,factNew,factOld,make_reduce_functor(bf_add_log<mat_val,mat_val,mat_val>(), bf_plus<vec_val,vec_val,mat_val>()));
			apply_scalar_functor(v,SF_EXP);
			break;
			case RF_LOGADDEXP:
			reduce_impl::reduce<dimension,mat_mem>()(v,m,factNew,factOld,make_reduce_functor(bf_logaddexp<mat_val>()));
			break;
			case RF_ADDEXP:
			reduce_impl::reduce<dimension,mat_mem>()(v,m,factNew,factOld,make_reduce_functor(bf_logaddexp<mat_val>()));
			apply_scalar_functor(v,SF_EXP);
			break;
			default:
			throw std::runtime_error("supplied reduce_functor is not implemented");
		}
	}


}//namespace reduce_imp

// TODO: make sure this is actually called with a matrix type!
//
template<class __value_type, class __value_type2, class __memory_space_type, class __memory_layout_type>
void reduce_to_col(tensor<__value_type,__memory_space_type>&v, const dense_matrix<__value_type2,__memory_space_type,__memory_layout_type>& m, reduce_functor rf, const __value_type2& factNew, const __value_type2& factOld) {
	if (IsSame<typename dense_matrix<__value_type,__memory_space_type,__memory_layout_type>::memory_layout,row_major>::Result::value){
		//matrix is row major
		//create column major view and call reduce_to_row for column major
		// downstream from here everything is column major
		const dense_matrix<const __value_type2,typename dense_matrix<__value_type,__memory_space_type,__memory_layout_type>::memory_space_type,column_major,typename dense_matrix<__value_type,__memory_space_type,__memory_layout_type>::index_type> cm_view(m.w(),m.h(),m.ptr());
		reduce_impl::reduce_switch<0>(v,cm_view,rf,factNew,factOld); // 0 means zeroth dimension is summed out - meaning summing over the columns in a column major matrix.
	}
	else {
		reduce_impl::reduce_switch<1>(v,m,rf,factNew,factOld); // 1 means first dimension (we start counting at zero) is summed out - meaning summing over the rows in a column major matrix.
	}
}

template<class __value_type, class __value_type2, class __memory_space_type, class __memory_layout_type>
void reduce_to_row(tensor<__value_type,__memory_space_type>&v, const dense_matrix<__value_type2,__memory_space_type,__memory_layout_type>& m,reduce_functor rf, const __value_type2& factNew, const __value_type2& factOld) {
	if (IsSame<typename dense_matrix<__value_type,__memory_space_type,__memory_layout_type>::memory_layout,row_major>::Result::value){
		//matrix is row major
		//create column major view and call reduce_to_row for column major
		// downstream from here everything is column major
		const dense_matrix<const __value_type2,__memory_space_type,column_major> cm_view(m.w(),m.h(),m.ptr());
		reduce_impl::reduce_switch<1>(v,cm_view,rf,factNew,factOld); // 1 means first (we start counting at zero) dimension is summed out - meaning summing over the rows in a column major matrix.
	}
	else {
		reduce_impl::reduce_switch<0>(v,m,rf,factNew,factOld); // 0 means zeroth dimension is summed out - meaning summing over the columns in a column major matrix.
	}
	
}

/*namespace argmax_to_XXX_impl{*/
	/*template<class V, class V2, class I>*/
	/*void argmax_to_row(tensor<V2,dev_memory_space>&v, const dense_matrix<V, dev_memory_space,column_major, I>& m) {*/
		/*cuvAssert(m.ptr() != NULL);*/
		/*cuvAssert(m.w() == v.size());*/
		/*const unsigned int u = min(m.w(), MAX_GRID_SIZE);*/
		/*dim3 grid(u, ceil(m.w()/(float)u));*/
		/*static const unsigned int BLOCK_DIM = 256;*/
		/*argmax_row_kernel<BLOCK_DIM><<<grid,BLOCK_DIM>>>(v.ptr(),m.ptr(),m.w(),m.h());*/
		/*cuvSafeCall(hipDeviceSynchronize());*/
	/*}*/

	/*template<class V, class V2, class I>*/
	/*void argmax_to_column(tensor<V2,dev_memory_space>&v, const dense_matrix<V,row_major,dev_memory_space,I>& m) {*/
		/*cuvAssert(m.ptr() != NULL);*/
		/*cuvAssert(m.h() == v.size());*/
		/*const unsigned int u = min(m.h(), MAX_GRID_SIZE);*/
		/*dim3 grid(u, ceil(m.h()/(float)u));*/
		/*static const unsigned int BLOCK_DIM = 256;*/
		/*argmax_row_kernel<BLOCK_DIM><<<grid,BLOCK_DIM>>>(v.ptr(),m.ptr(),m.h(),m.w());*/
		/*cuvSafeCall(hipDeviceSynchronize());*/
	/*}*/

	/*template<class V, class V2, class I>*/
	/*void argmax_to_row(tensor<V2,host_memory_space>&v, const dense_matrix<V, host_memory_space,column_major,I>& m) {*/
		/*cuvAssert(m.ptr() != NULL);*/
		/*cuvAssert(m.w() == v.size());*/
		/*const V* ptr = m.ptr();*/
		/*V2* res = v.ptr();*/
		/*for(int i=0; i<m.w();i++) {*/
			/*int idx = 0;*/
			/*V val = *ptr;*/
			/*for(int j=0; j<m.h();j++) {*/
				/*if(*ptr > val) {*/
					/*val = *ptr;*/
					/*idx = j;*/
				/*}*/
				/*ptr++;*/
			/*}*/
			/**res++ = idx;*/
		/*}*/
	/*}*/

	/*template<class V, class V2, class I>*/
	/*void argmax_to_column(tensor<V2,host_memory_space>&v, const dense_matrix<V,row_major,host_memory_space,I>& m) {*/
		/*cuvAssert(m.ptr() != NULL);*/
		/*cuvAssert(m.h() == v.size());*/
		/*const V* ptr = m.ptr();*/
		/*V2* res = v.ptr();*/
		/*for(int i=0; i<m.h();i++) {*/
			/*int idx = 0;*/
			/*V val = *ptr;*/
			/*for(int j=0; j<m.w();j++) {*/
				/*if(*ptr > val) {*/
					/*val = *ptr;*/
					/*idx = j;*/
				/*}*/
				/*ptr++;*/
			/*}*/
			/**res++ = idx;*/
		/*}*/
	/*}*/

/*}// namespace argmax_to_xxx*/
/*template<class V, class M>*/
/*void argmax_to_column(V&v, const M& m) {*/
	/*argmax_to_XXX_impl::argmax_to_column(v,m);*/
/*}*/

/*template<class V, class M>*/
/*void argmax_to_row(V&v, const M& m) {*/
	/*argmax_to_XXX_impl::argmax_to_row(v,m);*/
/*}*/

/*#define INSTANTIATE_ARGMAX_TO_ROW(V,M,I) \*/
  /*template void argmax_to_row(tensor<int,dev_memory_space>&,const dense_matrix<V,dev_memory_space,M,I>&);   \*/
  /*template void argmax_to_row(tensor<int,host_memory_space>&,const dense_matrix<V,host_memory_space,M,I>&);  \*/
  /*template void argmax_to_row(tensor<float,dev_memory_space>&,const dense_matrix<V,dev_memory_space,M,I>&);   \*/
  /*template void argmax_to_row(tensor<float,host_memory_space>&,const dense_matrix<V,host_memory_space,M,I>&);  */
/*#define INSTANTIATE_ARGMAX_TO_COL(V,M,I) \*/
  /*template void argmax_to_column(tensor<int,dev_memory_space>&,const dense_matrix<V,dev_memory_space,M,I>&);   \*/
  /*template void argmax_to_column(tensor<int,host_memory_space>&,const dense_matrix<V,host_memory_space,M,I>&); \*/
  /*template void argmax_to_column(tensor<float,dev_memory_space>&,const dense_matrix<V,dev_memory_space,M,I>&);   \*/
  /*template void argmax_to_column(tensor<float,host_memory_space>&,const dense_matrix<V,host_memory_space,M,I>&);   */

#define INSTANTIATE_RED(V,V2,M) \
  template void reduce_to_row(tensor<V2,dev_memory_space>&, const dense_matrix<V,dev_memory_space,M>&, reduce_functor,  const V&,const V&); \
  template void reduce_to_col(tensor<V2,dev_memory_space>&, const dense_matrix<V,dev_memory_space,M>&, reduce_functor, const V&,const V&); \
  template void reduce_to_row(tensor<V2,host_memory_space>&, const dense_matrix<V,host_memory_space,M>&, reduce_functor,  const V&,const V&); \
  template void reduce_to_col(tensor<V2,host_memory_space>&, const dense_matrix<V,host_memory_space,M>&,reduce_functor,  const V&,const V&);


/*INSTANTIATE_ARGMAX_TO_COL(float,row_major,unsigned int);*/
/*INSTANTIATE_ARGMAX_TO_COL(int,row_major,unsigned int);*/

/*INSTANTIATE_ARGMAX_TO_ROW(float,column_major,unsigned int);*/
/*INSTANTIATE_ARGMAX_TO_ROW(int,column_major,unsigned int);*/

INSTANTIATE_RED(float,float,column_major);
/*INSTANTIATE_RED(int,float,column_major);*/
/*INSTANTIATE_RED(unsigned int,float,column_major);*/
/*INSTANTIATE_RED(unsigned char,float,column_major);*/
/*INSTANTIATE_RED(float,int,column_major);*/
INSTANTIATE_RED(float,unsigned int,column_major);
/*INSTANTIATE_RED(float,unsigned char,column_major);*/
INSTANTIATE_RED(unsigned char,unsigned char,column_major);
/*INSTANTIATE_RED(unsigned char,unsigned int,column_major);*/

INSTANTIATE_RED(float,float,row_major);
/*INSTANTIATE_RED(int,float,row_major);*/
/*INSTANTIATE_RED(unsigned int,float,row_major);*/
/*INSTANTIATE_RED(unsigned char,float,row_major);*/
/*INSTANTIATE_RED(float,int,row_major);*/
INSTANTIATE_RED(float,unsigned int,row_major);
/*INSTANTIATE_RED(float,unsigned char,row_major);*/
INSTANTIATE_RED(unsigned char,unsigned char,row_major);
/*INSTANTIATE_RED(unsigned char,unsigned int,row_major);*/
};//namespace cuv

