#include "hip/hip_runtime.h"
#include <iostream>
#include "../../basics/dense_matrix.hpp"
#include "rbm.hpp"

namespace cuv{
namespace libs{
namespace rbm{

	namespace detail{

		/* ****************************
		   column_major
	     * ****************************/
		template<class V, class I>
		void set_binary_sequence(cuv::dense_matrix<V,column_major,host_memory_space,I>& m, const int& start){
			const int len = m.h();
			V* ptr  = m.ptr();
			for(unsigned int i=start; i<m.w()+start; i++){
				for(unsigned int j=0;j<len;j++){
					*ptr++ = (i & (1 << (len-1-j))) ? 1 : 0;
				}
			}
		}
		template<class value_type,class index_type>
		__global__
		void set_binary_sequence_kernel(value_type* dst, index_type h, index_type w, int start){
			// for column-major matrices of size (h x w)
			const index_type x = blockIdx.y * blockDim.y + threadIdx.y;
			const index_type y = blockIdx.x * blockDim.x + threadIdx.x;
			if(y>=h) return;
			if(x>=w) return;
			dst[x*h+y] = ((start+x) & (1 << (h-1-y))) ? 1 : 0;
		}
		template<class V, class I>
		void set_binary_sequence(cuv::dense_matrix<V,column_major,dev_memory_space,I>& m, const int& start){
			dim3 threads(16,16);
			dim3 grid(ceil(m.h()/float(threads.x)), ceil(m.w()/float(threads.y)));
			set_binary_sequence_kernel<<<grid,threads>>>(m.ptr(),m.h(),m.w(),start);
			cuvSafeCall(hipDeviceSynchronize());
		}

		template<class V, class I>
		void sigm_temperature(cuv::dense_matrix<V,column_major,host_memory_space,I>& m, const cuv::vector<V,host_memory_space,I>& temp){
			cuvAssert(m.w() == temp.size())
			V* mptr = m.ptr();
			for(unsigned int col=0;col<m.w();col++){
				const V T = temp[col];
				const V* end  = mptr + m.h();
				while(mptr < end){
					*mptr = 1.0/(1.0+exp(-*mptr / T));
					mptr++; 
				}
			}
		}
		template<class value_type,class index_type>
		__global__
		void sigm_temperature_kernel(value_type* dst, const value_type* src, const value_type* temp, index_type h, index_type w){
			// for column-major matrices of size (h x w)
			const index_type x = blockIdx.y * blockDim.y + threadIdx.y;
			const index_type y = blockIdx.x * blockDim.x + threadIdx.x;
			const value_type T = temp[x];
			if(y>=h) return;
			if(x>=w) return;
			dst[x*h+y] = (value_type) (1.0/(1.0 + expf(-src[x*h+y] / T)));
		}
		template<class V, class I>
		void sigm_temperature(cuv::dense_matrix<V,column_major,dev_memory_space,I>& m, const cuv::vector<V,dev_memory_space,I>& temp){
			dim3 threads(16,16);
			dim3 grid(ceil(m.h()/float(threads.x)), ceil(m.w()/float(threads.y)));
			sigm_temperature_kernel<<<grid,threads>>>(m.ptr(),m.ptr(),temp.ptr(),m.h(),m.w());
			cuvSafeCall(hipDeviceSynchronize());
		}

		/* ****************************
		   row_major
	     * ****************************/
		template<class V, class I>
		void set_binary_sequence(cuv::dense_matrix<V,row_major,host_memory_space,I>& m, const int& start){
			// TODO: make column-major view, then call again
		}
		template<class V, class I>
		void set_binary_sequence(cuv::dense_matrix<V,row_major,dev_memory_space,I>& m, const int& start){
			// TODO: make column-major view, then call again
		}

		template<class V, class I>
		void sigm_temperature(cuv::dense_matrix<V,row_major,host_memory_space,I>& m, const cuv::vector<V,host_memory_space,I>& temp){
			// TODO: make column-major view, then call again
		}
		template<class V, class I>
		void sigm_temperature(cuv::dense_matrix<V,row_major,dev_memory_space,I>& m, const cuv::vector<V,dev_memory_space,I>& temp){
			// TODO: make column-major view, then call again
		}



		/******************************
		  local connectivity kernel
		 ******************************/
		template<class T>
		__global__ void local_connectivity_kernel(T* mat ,int h,int w, int pix_v, int pix_h, int num_map_hid, int patchsize, int px, int py, int maxdist_from_main_dia) {
			const int i = threadIdx.x + blockIdx.x * blockDim.x; // i changes with the visible unit
			const int j = threadIdx.y + blockIdx.y * blockDim.y; // j changes with the hidden unit
			if ((i>=pix_v) || (j>=pix_h)) return;

			const int  map_hid  = (j * pix_v)/pix_h;  // map_hid is now in the same coordinate frame as map_vis (scaled up in most cases...)
			const int& map_vis  = i;
			const int  v_y     = map_vis % py;
			const int  h_y     = map_hid % px;
			const int  v_x     = map_vis / py;
			const int  h_x     = map_hid / px;
			const int  num_map_vis = h / (px*py);

			const bool outpatch = (    abs(v_y-h_y)   > patchsize || abs(v_x-h_x)   > patchsize); // we are not in the patch
			for(int hidx = 0; hidx<num_map_vis; hidx++) // loop over visible maps
				for(int idx = 0; idx<num_map_hid; idx++)// loop over hidden maps
					if(outpatch || abs(idx-hidx)>maxdist_from_main_dia) 
						mat[(idx*pix_h+j)*h + hidx*pix_v+i]=(T)0; // reset this value

			for(int hidx = i; hidx<h; hidx  += pix_v)
			   for(int b=num_map_hid*pix_h; b<w; b += blockDim.y){
				   int col = (b+threadIdx.y);
				   if(col<w)
					   mat[col*h+hidx]=(T)0;
			   }
		}

		template<class V, class I>
		void set_local_connectivity_in_dense_matrix(cuv::dense_matrix<V,column_major,dev_memory_space,I>& m, float factor, int patchsize, int px, int py, int maxdist_from_main_dia){
			cuvAssert(m.ptr());
			/*int num_maps_lo = (m.h()) / (px*py);*/
			int pix_v = px*py;
			int pix_h = ceil(pix_v * factor);
			static const int bs = 16;
			dim3 blocks(ceil(pix_v/(float)bs),ceil(pix_h/(float)bs));
			dim3 threads(bs,bs);
			int num_maps = (m.w()) / pix_h;
			local_connectivity_kernel<<<blocks,threads>>>(m.ptr(),m.h(),m.w(),pix_v,pix_h,num_maps, patchsize,px,py,maxdist_from_main_dia);
			cuvSafeCall(hipDeviceSynchronize());
		}
	}

template<class __matrix_type>
void set_binary_sequence(__matrix_type& m, const int& start){
	detail::set_binary_sequence(m,start);
}
template<class __matrix_type,class __vector_type>
void sigm_temperature(__matrix_type& m, const __vector_type& temp){
	detail::sigm_temperature(m,temp);
}
template<class __matrix_type>
void set_local_connectivity_in_dense_matrix(__matrix_type& m, float factor, int patchsize, int px, int py, int maxdist_from_main_dia){
	detail::set_local_connectivity_in_dense_matrix(m,factor, patchsize, px, py, maxdist_from_main_dia);
}


#define INST(V,L,M,I) \
  template void set_binary_sequence(cuv::dense_matrix<V,L,M,I>& m, const int&); \
  template void sigm_temperature(cuv::dense_matrix<V,L,M,I>& m, const cuv::vector<V,M,I>&); \

INST(float,column_major,host_memory_space,unsigned int);
INST(float,column_major,dev_memory_space,unsigned int);

template
void set_local_connectivity_in_dense_matrix(cuv::dense_matrix<float,column_major,dev_memory_space>& m, float factor, int patchsize, int px, int py, int);

}
}
}
