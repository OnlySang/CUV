#include "hip/hip_runtime.h"
#include <iostream>
#include "../../basics/dense_matrix.hpp"
#include "kernels.hpp"
#define V(X) #X <<": "<<(X) <<"   "

using namespace std;

////////////////////////////////////////////////////////////////////////////////
//! for every row in A, every column in B, calculate sum of squared differences
//! wA is A's width and wB is B's width
////////////////////////////////////////////////////////////////////////////////

template <int BLOCK_DIM, class __value_type, class __distance_type>
__global__ 
void
pairwise_distance_kernel( __distance_type* C,const  __value_type* A, const __value_type* B, int wA, int hB)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = __mul24(__mul24(wA , BLOCK_DIM) , by);
    int aEnd   = aBegin + wA - 1;
    int aStep  = BLOCK_DIM;

    int bBegin = __mul24(__mul24(wA , BLOCK_DIM) , bx);
    int bStep  = BLOCK_DIM;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    __distance_type Csub = 0;


	int waty = __mul24(wA,ty)+tx;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b  = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) {

        __shared__ __distance_type AS[BLOCK_DIM][BLOCK_DIM];
        __shared__ __distance_type BS[BLOCK_DIM][BLOCK_DIM+1];

        AS[ty][tx] = (__distance_type)A[a + waty];
		BS[ty][tx] = (__distance_type)B[b + waty];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // compute squared difference.
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_DIM; ++k){
			__distance_type f = AS[ty][k]-BS[tx][k];
            Csub += f*f;
		}

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = __mul24(__mul24(hB , BLOCK_DIM) , by) + __mul24(BLOCK_DIM , bx);
    C[c + __mul24(hB , ty) + tx] = Csub;
}

namespace cuv{
namespace libs{	
	namespace kernels{
	template <class __matrix_type>
	void 
	pairwise_distance(__matrix_type& result, const __matrix_type& A, const __matrix_type& B){
		/*cuvAssert(A.w() == B.w());*/
		/*cuvAssert(A.h() == result.h());*/
		/*cuvAssert(B.h() == result.w());*/

		const int BLOCK_DIM = 32;
		dim3 threads(BLOCK_DIM, BLOCK_DIM);
		dim3 grid(B.h() / threads.x, A.h() / threads.y);

		/*cuvAssert(B.w()%threads.x == 0);*/
		/*cuvAssert(A.h()%threads.y == 0);*/
		/*cuvAssert(B.h()%threads.x == 0);*/
		/*cuvAssert(A.w()%threads.y == 0);*/

		cuvAssert(grid.x > 0);
		cuvAssert(grid.y > 0);
		pairwise_distance_kernel<BLOCK_DIM><<< grid,threads >>>(result.ptr(),A.ptr(),B.ptr(),A.w(),B.h());
		hipDeviceSynchronize();
		checkCudaError("kernel sqDiff invocation");
	}
typedef dense_matrix<float, row_major, dev_memory_space, unsigned int> dm_rmf;
template void pairwise_distance<dm_rmf>(dm_rmf&, const dm_rmf &, const dm_rmf&);

}}}
