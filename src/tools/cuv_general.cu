#include <string>
#include <stdexcept>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>
#include "cuv_general.hpp"

namespace cuv{
	using namespace std;
	void checkCudaError(const char *msg)
	{
		hipError_t err = hipGetLastError();
		if( hipSuccess != err) 
		{
			cout << "checkCudaError: " << msg << ": " << hipGetErrorString(err) <<endl;
			abort();
			throw std::runtime_error(std::string(msg) + hipGetErrorString(err) );
		}                         
	}
	void initCUDA(int dev){
		cutilSafeCall(hipSetDevice(dev));
	}
	void exitCUDA(){
		hipDeviceReset();
	}

}
