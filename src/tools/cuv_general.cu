#include <string>
#include <stdexcept>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>

#include "cuv_general.hpp"
#include "exception_helper.hpp"

namespace cuv{
	using namespace std;
	void cuvAssertFailed(const char *msg){
			/*cout << "cuvAssert failed: " << msg <<endl;*/
			/*abort();*/
		ExceptionTracer et;
			throw std::runtime_error(std::string(msg));
	}
	void checkCudaError(const char *msg)
	{
		hipError_t err = hipGetLastError();
		if( hipSuccess != err) 
		{
			/*cout << "checkCudaError: " << msg << ": " << hipGetErrorString(err) <<endl;*/
			/*abort();*/
			ExceptionTracer et;
			throw std::runtime_error(std::string(msg) + hipGetErrorString(err) );
		}                         
	}
	void initCUDA(int dev){
		cutilSafeCall(hipSetDevice(dev));
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);
		bool canHostmap = prop.canMapHostMemory;
		if(canHostmap){
			cutilSafeCall(hipSetDeviceFlags(hipDeviceMapHost));
		}
	}
	void exitCUDA(){
		hipDeviceReset();
	}

}
