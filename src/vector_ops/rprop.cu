#include "hip/hip_runtime.h"
#include "rprop.hpp"

//#define sgn(a) (copysign(1.f,a))
#define sgn(a) ((a==0) ? 0 : copysign(1.f,a))

#define ETA_P 1.2f
#define ETA_M 0.5f
#define DELTA_MAX 50.0f
#define DELTA_MIN (1.0E-6)


template<class T, class O>
__global__ void rprop_kernel(T*W, T* dW, O* dW_old, T* rate, int n) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int off = blockDim.x * gridDim.x;
	for (unsigned int i = idx; i < n; i += off){
		O sn = (O)sgn(dW[i]);
		O s  = dW_old[i] * sn;
		T dwn, rn=rate[i];

		if ( s > 0) {
			rn = min( ETA_P * rn, DELTA_MAX );
			dwn = sn * rn;
		}
		else if ( s < 0) {
			rn = max( ETA_M * rn, DELTA_MIN);
			dwn = 0;
		}   
		else {
			dwn = sn * rn;
		}   
		__syncthreads();
		rate[i]   = rn;
		dW_old[i] = (O)sgn(dwn);
		W[i]     += dwn;
	}
} 


template<class T>
__global__ void learn_step_weight_decay_kernel(T* A, T* dA, T alpha, T beta, int n) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int off = blockDim.x * gridDim.x;
	for (unsigned int i = idx; i < n; i += off){
		A[i] = alpha*dA[i] + beta*A[i];
	}
}



namespace cuv{

	template<class V, class O, class I>
	void
	rprop_impl(dev_vector<V,I>& W, dev_vector<V,I>& dW, dev_vector<O,I>& dW_old, dev_vector<V,I>& rate){
		int num_threads = 512;
		int num_blocks  = min(512,(int)ceil((float)dW.size() / num_threads));
		rprop_kernel<<< num_threads, num_blocks>>>(W.ptr(), dW.ptr(), dW_old.ptr(), rate.ptr(), dW.size());
		cuvSafeCall(hipDeviceSynchronize());
	}

	template<class V, class O, class I>
	void
	rprop_impl(host_vector<V,I>& W, host_vector<V,I>& dW, host_vector<O,I>& dW_old, host_vector<V,I>& rate){
		for (unsigned int i = 0; i < dW.size(); i++){
			O sn = (O)sgn(dW[i]);
			O s  = dW_old[i] * sn;
			V dwn,rn=(V)0, r = rate[i];

			if (s > 0) {
				rn = min( ETA_P * r , DELTA_MAX );
				dwn = sn * rn;
			}
			else if (s < 0) {
				rn = max( ETA_M * r, DELTA_MIN);
				dwn = 0;
			}   
			else {
				dwn = sn * r;
			}
			/*__synchthreads();*/
			rate.set(i,rn);
			dW_old.set(i,(O)sgn(dwn));
			W.set(i, W[i]  + dwn);
		}
	}

	template<class __vector_type, class __old_vector_type>
	void rprop(__vector_type& W, __vector_type& dW, __old_vector_type& dW_old, __vector_type& rate){
		cuvAssert(dW.ptr());
		cuvAssert(dW_old.ptr());
		cuvAssert(rate.ptr());
		cuvAssert(dW.size() == dW_old.size());
		cuvAssert(dW.size() ==  rate.size());
		rprop_impl(W,dW,dW_old,rate);
	}



	template<class V, class I>
	void learn_step_weight_decay_impl(dev_vector<V,I>& W,dev_vector<V,I>& dW, const float& alpha, const float& beta){
		int num_threads = 512;
		int num_blocks  = min(512,(int)ceil((float)dW.size() / num_threads));
		learn_step_weight_decay_kernel<<< num_threads, num_blocks>>>(W.ptr(), dW.ptr(), alpha, beta, W.size());
		cuvSafeCall(hipDeviceSynchronize());
	}
	template<class V, class I>
	void learn_step_weight_decay_impl(host_vector<V,I>& W,host_vector<V,I>& dW, const float& alpha, const float& beta){
		for (unsigned int i = 0; i < W.size(); i++){
			W.set(i,alpha*dW[i] + beta*W[i]);
		}
	}
	template<class __vector_type>
	void learn_step_weight_decay(__vector_type& W,__vector_type& dW, const float& learnrate, const float& decay){
		cuvAssert(dW.ptr());
		cuvAssert(W.ptr());
		cuvAssert(W.size() == dW.size());
		learn_step_weight_decay_impl(W,dW,learnrate,1.f-learnrate*decay);
	}

#define RPROP_INSTANTIATE(V,O) \
	template void rprop( host_vector<V>&, host_vector<V>&, host_vector<O>&, host_vector<V>&); \
	template void rprop( dev_vector<V>&,  dev_vector<V>&, dev_vector<O>&, dev_vector<V>&);    
#define LSWD_INSTANTIATE(V) \
	template void learn_step_weight_decay( host_vector<V>&, host_vector<V>&, const float&,const float&); \
	template void learn_step_weight_decay( dev_vector<V>&,  dev_vector<V>&, const float&,const float&);

	RPROP_INSTANTIATE(float,float);
	RPROP_INSTANTIATE(float,signed char);
	LSWD_INSTANTIATE(float);

}
