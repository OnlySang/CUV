#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#include <cuv_general.hpp>

#include <dev_vector.hpp>
#include <host_vector.hpp>

#include "vector_ops.hpp"

#define sgn(a) (copysign(1.f,a))

using namespace cuv;
using namespace std;

template<class T>
struct uf_exp{  __host__ __device__         T operator()(const T& t)const{ return __expf(t);    } };
template<class T>
struct uf_exact_exp{  __device__ __host__   T operator()(const T& t)const{ return exp(t);    } };
template<class T>
struct uf_log{  __device__ __host__         T operator()(const T& t)      const{ return log(t);    } };
template<class T>
struct uf_sign{  __device__ __host__        T operator()(const T& t)      const{ return sgn(t);    } };
template<class T>
struct uf_sigm{  __device__  __host__       T operator()(const T& t)      const{ return ((T)1)/(((T)1)+__expf(-t));    } };
template<class T>
struct uf_exact_sigm{  __device__  __host__ T operator()(const T& t)      const{ return ((T)1)/(((T)1)+exp(-t));    } };
template<class T>
struct uf_dsigm{  __device__ __host__       T operator()(const T& t)      const{ return t * (((T)1)-t); } };
template<class T>
struct uf_tanh{  __device__  __host__       T operator()(const T& t)      const{ return tanh(t); } };
template<class T>
struct uf_dtanh{  __device__  __host__      T operator()(const T& t)      const{ return ((T)1) - (t*t); } };
template<class T>
struct uf_square{  __device__  __host__     T operator()(const T& t)      const{ return t*t; } };
template<class T>
struct uf_sublin{  __device__  __host__     T operator()(const T& t)      const{ return ((T)1)-t; } };
template<class T>
struct uf_energ{  __device__  __host__      T operator()(const T& t)      const{ return -log(t); } };
template<class T>
struct uf_inv{  __device__  __host__        T operator()(const T& t)      const{ return ((T)1)/(t+((T)0.00000001)); } };
template<class T>
struct uf_sqrt{  __device__  __host__       T operator()(const T& t)      const{ return sqrt(t); } };

template<class T, class binary_functor>
struct uf_base_op{
  const T x;
  const binary_functor bf;
  uf_base_op(const T& _x):x(_x),bf(){};
  T operator()(T t){ return bf(t,x); }
};

/*
 * Binary Functors
 */

template<class T, class U>
struct bf_plus{  __device__  __host__       T operator()(const T& t, const U& u)      const{ return  t + (T)u; } };
template<class T, class U>
struct bf_minus{  __device__  __host__       T operator()(const T& t, const U& u)      const{ return  t - (T)u; } };
template<class T, class U>
struct bf_multiplies{  __device__  __host__       T operator()(const T& t, const U& u)      const{ return  t * (T)u; } };
template<class T, class U>
struct bf_divides{  __device__  __host__       T operator()(const T& t, const U& u)      const{ return  t / (T)u; } };

template<class T, class U>
struct bf_axpy{  
	const T a;
	bf_axpy(const T& _a):a(_a){}
	__device__  __host__       T operator()(const T& t, const U& u) const{ return  a*t+(T)u; } 
};
template<class T, class U>
struct bf_xpby{  
	const T b;
	bf_xpby(const T& _b):b(_b){}
	__device__  __host__       T operator()(const T& t, const U& u) const{ return  t+b*(T)u; } 
};
template<class T, class U>
struct bf_axpby{  
	const T a;
	const T b;
	bf_axpby(const T& _a, const T& _b):a(_a),b(_b){}
	__device__  __host__       T operator()(const T& t, const U& u) const{ return  a*t + b*((T)u); } 
};


template<class unary_functor, class value_type, class index_type>
void launch_unary_kernel(
   cuv::dev_vector<value_type, index_type>& dst,
   cuv::dev_vector<value_type, index_type>& src, 
	 unary_functor uf){
	 cuvAssert(dst.ptr());
	 cuvAssert(src.ptr());
	 cuvAssert(dst.size() == src.size());

	 thrust::device_ptr<value_type> dst_ptr(dst.ptr());
	 thrust::device_ptr<value_type> src_ptr(src.ptr());
	 thrust::transform(src_ptr,src_ptr+src.size(),dst_ptr,uf);
	 cuvSafeCall(hipDeviceSynchronize());
}

template<class unary_functor, class value_type, class index_type>
void launch_unary_kernel(
   cuv::host_vector<value_type, index_type>& dst,
   cuv::host_vector<value_type, index_type>& src, 
	 unary_functor uf){
	 cuvAssert(src.ptr());
	 cuvAssert(dst.ptr());
	 cuvAssert(dst.size() == src.size());
	 for(size_t i=0;i<dst.size();i++)
	   dst[i] = uf(src[i]);
}

namespace cuv{
	
/*
 * Nullary Functor
 *
 */

template<class __vector_type>
void
apply_0ary_functor(__vector_type& v, const NullaryFunctor& nf){
	 cuvAssert(v.ptr());
	 typedef typename __vector_type::value_type value_type;

	 thrust::device_ptr<value_type> dst_ptr(v.ptr());
	switch(nf){
		case NF_SEQ:
			thrust::sequence(dst_ptr,dst_ptr+v.size());break;
		default:
			cuvAssert(false);
	}
	cuvSafeCall(hipDeviceSynchronize());
}

template<class __vector_type, class __value_type>
void
apply_0ary_functor(__vector_type& v, const NullaryFunctor& nf, const __value_type& param){
	 cuvAssert(v.ptr());

	 typedef typename __vector_type::value_type value_type;
	 thrust::device_ptr<value_type> dst_ptr(v.ptr());
	 switch(nf){
		 case NF_FILL:
			 thrust::fill(dst_ptr,dst_ptr + v.size(), (value_type)param); break;
		 default:
			 cuvAssert(false);
	 }
	 cuvSafeCall(hipDeviceSynchronize());
}

/*
 * Unary Functor
 *
 */
template<class __vector_type>
struct apply_scalar_functor_impl;

template<class __vector_type>
void
apply_scalar_functor(__vector_type& v, const ScalarFunctor& sf){
  apply_scalar_functor_impl<__vector_type>::apply(v,sf);
}
template<class __vector_type, class __value_type>
void
apply_scalar_functor(__vector_type& v, const ScalarFunctor& sf, const __value_type& param){
  apply_scalar_functor_impl<__vector_type>::apply(v,sf,param);
}

/*
 * Binary Functor
 *
 */
template<class __vector_type1, class __vector_type2>
void
apply_binary_functor(__vector_type1& v, __vector_type2& w, const BinaryFunctor& sf){
	cuvAssert(v.size() == w.size());
	typedef typename __vector_type1::value_type V1;
	typedef typename __vector_type2::value_type V2;
	thrust::device_ptr<V1> v_ptr ( v.ptr() );
	thrust::device_ptr<V2> w_ptr ( w.ptr() );
	switch(sf){
		case BF_ADD:      thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_plus<V1,V2>()); break;
		case BF_SUBTRACT: thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_minus<V1,V2>()); break;
		case BF_MULT:     thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_multiplies<V1,V2>()); break;
		case BF_DIV:      thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_divides<V1,V2>()); break;
		case BF_COPY:     thrust::copy(w_ptr, w_ptr+v.size(), v_ptr); break;
		default: cuvAssert(false);
	}
}

template<class __vector_type1, class __vector_type2, class __value_type>
void
apply_binary_functor(__vector_type1& v, __vector_type2& w, const BinaryFunctor& sf, const __value_type& param){
	cuvAssert(v.size() == w.size());
	typedef typename __vector_type1::value_type V1;
	typedef typename __vector_type2::value_type V2;
	thrust::device_ptr<V1> v_ptr ( v.ptr() );
	thrust::device_ptr<V2> w_ptr ( w.ptr() );
	switch(sf){
		case BF_AXPY:     thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_axpy<V1,V2>(param)); break;
		case BF_XPBY:     thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_xpby<V1,V2>(param)); break;
		default: cuvAssert(false);
	}
}

template<class __vector_type1, class __vector_type2, class __value_type>
void
apply_binary_functor(__vector_type1& v, __vector_type2& w, const BinaryFunctor& sf, const __value_type& param, const __value_type& param2){
	cuvAssert(v.size() == w.size());
	typedef typename __vector_type1::value_type V1;
	typedef typename __vector_type2::value_type V2;
	thrust::device_ptr<V1> v_ptr ( v.ptr() );
	thrust::device_ptr<V2> w_ptr ( w.ptr() );
	switch(sf){
		case BF_AXPBY:     thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_axpby<V1,V2>(param,param2)); break;
		default: cuvAssert(false);
	}
}

template<class __vector_type>
struct apply_scalar_functor_impl{

	template<class __arg_value_type>
	static void
	apply(__vector_type& v, const ScalarFunctor& sf, const __arg_value_type& param){
		typedef typename __vector_type::value_type value_type;
		switch(sf){
			case SF_ADD:       launch_unary_kernel(v,v,uf_base_op<value_type, thrust::plus<value_type> >(param)); break;
			case SF_MULT:      launch_unary_kernel(v,v,uf_base_op<value_type, thrust::multiplies<value_type> >(param)); break;
			case SF_DIV:       launch_unary_kernel(v,v,uf_base_op<value_type, thrust::divides<value_type> >(param)); break;
			case SF_SUBTRACT:  launch_unary_kernel(v,v,uf_base_op<value_type, thrust::minus<value_type> >(param)); break;
		}
	}

	static void
	apply(__vector_type& v, const ScalarFunctor& sf){
		typedef typename __vector_type::value_type value_type;
	  switch(sf){
			case SF_EXP:        launch_unary_kernel(v,v, uf_exp<value_type>()); break;
			case SF_EXACT_EXP:  launch_unary_kernel(v,v, uf_exact_exp<value_type>()); break;
			case SF_LOG:        launch_unary_kernel(v,v, uf_log<value_type>()); break;
			case SF_SIGN:       launch_unary_kernel(v,v, uf_sign<value_type>()); break;
			case SF_SIGM:       launch_unary_kernel(v,v, uf_sigm<value_type>()); break;
			case SF_DSIGM:      launch_unary_kernel(v,v, uf_dsigm<value_type>()); break;
			case SF_TANH:       launch_unary_kernel(v,v, uf_tanh<value_type>()); break;
			case SF_DTANH:      launch_unary_kernel(v,v, uf_dtanh<value_type>()); break;
			case SF_SQUARE:     launch_unary_kernel(v,v, uf_square<value_type>()); break;
			case SF_SUBLIN:     launch_unary_kernel(v,v, uf_sublin<value_type>()); break;
			case SF_ENERG:      launch_unary_kernel(v,v, uf_energ<value_type>()); break;
			case SF_INV:        launch_unary_kernel(v,v, uf_inv<value_type>()); break;
			case SF_SQRT:       launch_unary_kernel(v,v, uf_sqrt<value_type>()); break;
			case SF_NEGATE:     launch_unary_kernel(v,v, thrust::negate<value_type>()); break;
			default:
			 cuvAssert(false);
		}
	}
};


#define SIMPLE_0(X) \
	template void apply_0ary_functor< X >(X&, const NullaryFunctor&); \
	template void apply_0ary_functor< X, float>(X&, const NullaryFunctor&, const float& param); \
	template void apply_0ary_functor< X, int>  (X&, const NullaryFunctor&, const int& param); 

#define SIMPLE_1(X) \
	template void apply_scalar_functor< X >(X&, const ScalarFunctor&); \
	template void apply_scalar_functor< X, float>(X&, const ScalarFunctor&,const float&); \
	template void apply_scalar_functor< X, int>(X&, const ScalarFunctor&,const int&);

#define SIMPLE_2(X,Y) \
	template void apply_binary_functor<X,Y      >(X&, Y&, const BinaryFunctor&); \
	template void apply_binary_functor<X,Y,float>(X&, Y&, const BinaryFunctor&,  const float&); \
	template void apply_binary_functor<X,Y,  int>(X&, Y&, const BinaryFunctor&,  const int&); \
	template void apply_binary_functor<X,Y,float>(X&, Y&, const BinaryFunctor&,  const float&, const float&); \
	template void apply_binary_functor<X,Y,  int>(X&, Y&, const BinaryFunctor&,  const int&, const int&);

	SIMPLE_0(dev_vector<float>); 
	SIMPLE_0(host_vector<float>); 
	SIMPLE_1(dev_vector<float>); 
	SIMPLE_1(host_vector<float>); 
	SIMPLE_2(dev_vector<float>, dev_vector<float>); 
	SIMPLE_2(host_vector<float>, dev_vector<float>);

} // cuv
