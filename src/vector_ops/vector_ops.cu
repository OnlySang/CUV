#include "hip/hip_runtime.h"
//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*





#include <cmath>
#include <iostream>
#include <hipblas.h>

#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform_reduce.h>
#include <thrust/generate.h>
#include <thrust/logical.h>

#include <cuv_general.hpp>
#include <cutil_inline.h>

#include <vector.hpp>

#include "vector_ops.hpp"

#define sgn(a) (copysign(1.f,a))

/*
 * USE_THRUST_LAUNCHER:
 * thrust has an overhead for looking up the correct block/grid-size for threads.
 * this overhead goes away for matrices of about 784*2048 for very simple linear kernels,
 * then they are better on bigcuda1.
 *
 */
#define USE_THRUST_LAUNCHER 0 


using namespace cuv;
using namespace std;

template<class T, class M>
struct memspace_cuv2thrustptr                          { typedef thrust::device_ptr<T> ptr_type; };
template<class T>
struct memspace_cuv2thrustptr<T,cuv::host_memory_space>{ typedef T* ptr_type; };
template<class T>
struct memspace_cuv2thrustptr<T,cuv::dev_memory_space> { typedef thrust::device_ptr<T> ptr_type; };

template<class T>
struct uf_exp{  __host__ __device__         T operator()(const T& t)const{ return expf(t);    } };
template<class T>
struct uf_exact_exp{  __device__ __host__   T operator()(const T& t)const{ return expf(t);    } };
template<class T>
struct uf_log{  __device__ __host__         T operator()(const T& t)      const{ return logf(t);    } };
template<class T>
struct uf_sign{  __device__ __host__        T operator()(const T& t)      const{ return sgn((float)t);    } };
template<class T>

#ifdef __DEVICE_EMULATION__
	struct uf_sigm{  __device__  __host__       T operator()(const T& t)      const{ return ((T)1)/(((T)1)+expf(-t));    } };
#else
	struct uf_sigm{  __device__  __host__       T operator()(const T& t)      const{ return ((T)1)/(((T)1)+expf(-t));    } };
#endif

template<class T>
struct uf_exact_sigm{  __device__  __host__ T operator()(const T& t)      const{ return ((T)1)/(((T)1)+expf(-t));    } };
template<class T>
struct uf_dsigm{  __device__ __host__       T operator()(const T& t)      const{ return t * (((T)1)-t); } };
template<class T>
struct uf_tanh{  __device__  __host__       T operator()(const T& t)      const{ return tanhf(t); } };
template<class T>
struct uf_dtanh{  __device__  __host__      T operator()(const T& t)      const{ return ((T)1) - (t*t); } };
template<class T>
struct uf_square{  __device__  __host__     T operator()(const T& t)      const{ return t*t; } };
template<class T>
struct uf_sublin{  __device__  __host__     T operator()(const T& t)      const{ return ((T)1)-t; } };
template<class T>
struct uf_energ{  __device__  __host__      T operator()(const T& t)      const{ return -logf(t); } };
template<class T>
struct uf_inv{  __device__  __host__        T operator()(const T& t)      const{ return ((T)1)/(t+((T)0.00000001)); } };
template<class T>
struct uf_sqrt{  __device__  __host__       T operator()(const T& t)      const{ return sqrtf(t); } };
template<class T>
struct uf_abs{  __device__  __host__       T operator()(const T& t)      const{ return t < T(0) ? -t : t; } };
template<class T>
struct uf_smax{  __device__  __host__      T operator()(const T& t)      const{ return (((T)1)/t - (T) 1) * t; } };

template<class T>
struct uf_is_nan{                 __device__  __host__   bool operator()(const T& t)             const{ return (t!=t) ; } };
template<>
struct uf_is_nan<int>{            __device__  __host__   bool operator()(const int& t)           const{ return false ; } };
template<>
struct uf_is_nan<unsigned char>{  __device__  __host__   bool operator()(const unsigned char& t) const{ return false ; } };
template<>
struct uf_is_nan<signed char>{    __device__  __host__   bool operator()(const signed char& t)   const{ return false ; } };

template<class T>
struct uf_is_inf{                 __device__  __host__     bool operator()(const T& t)            const{ return (bool)!isfinite(t); } };
template<>                                                                                        
struct uf_is_inf<int>{            __device__  __host__     bool operator()(const int t)           const{ return false; } };
template<>                                                                                        
struct uf_is_inf<signed char>{    __device__  __host__     bool operator()(const signed char t)   const{ return false; } };
template<>
struct uf_is_inf<unsigned char>{  __device__  __host__     bool operator()(const unsigned char t) const{ return false; } };

template<class T>
struct uf_poslin{  __device__  __host__     T operator()(const T& t)      const{ return (t > 0)*t; } };


template<class T>
struct bf_sigm_temp{ __device__  __host__       T operator()(const T& t, const T& temp)           const{ return ((T)1)/(((T)1)+expf(-t / (T)(temp))); } };

template<class T>
struct tf_tanh{  __device__  __host__       T operator()(const T& x, const T& a, const T& b)      const{ return a * tanhf(b * x); } };
template<class T>
struct tf_dtanh{  __device__  __host__      T operator()(const T& x, const T& a, const T& b)      const{ return b/a * (a+x) * (a-x); } };

// rectifying transferfunctions a is param beta
template<class T, class A>
struct tf_rect{  __device__  __host__       T operator()(const T& x, const A& a)      const{
	T ax = a*x;
	if(-ax < -87.33f)
		return (T) 0;
	return log(1.0f+expf(ax))/a;
}};
/*template<class T, class A>*/
/*struct tf_rect{  __device__  __host__       T operator()(const T& x, const A& a)      const{ return (T) log(1.0 + (double)exp((double)a*x))/a; } };*/
template<class T, class A>
struct tf_drect{  __device__  __host__      T operator()(const T& x, const A& a)      const{ return 1-1/(x*expf(a)); } };


template<class T, class binary_functor>
struct uf_base_op{
  const T x;
  const binary_functor bf;
  uf_base_op(const T& _x):x(_x),bf(){};
  __device__ __host__
  T operator()(const T& t){ return bf(t,x); }
};
template<class T, class ternary_functor>
struct uf_base_op3{
  const T x,y;
  const ternary_functor tf;
  uf_base_op3(const T& _x, const T& _y):x(_x),y(_y),tf(){};
  __device__ __host__
  T operator()(const T& t){ return tf(t,x,y); }
};

/*
 * Binary Functors
 */

// functors without parameter
template<class T, class U>
struct bf_plus{  __device__  __host__       T operator()(const T& t, const U& u)      const{ return  t + (T)u; } };
template<class T, class U>
struct bf_minus{  __device__  __host__      T operator()(const T& t, const U& u)      const{ return  t - (T)u; } };
template<class T, class U>
struct bf_multiplies{  __device__  __host__ T operator()(const T& t, const U& u)      const{ return  t * (T)u; } };
template<class T, class U>
struct bf_divides{  __device__  __host__    T operator()(const T& t, const U& u)      const{ return  t / (T)u; } };
template<class T, class U>
struct bf_squared_diff{__device__ __host__  T operator()(const T& t, const U& u)      const{ T ret =  t - (T)u; return ret*ret; } };
template<class T, class U>
struct bf_and{__device__ __host__   T operator()(const T& t, const U& u)      const{ return t && u; } };
template<class T, class U>
struct bf_or{ __device__ __host__   T operator()(const T& t, const U& u)      const{ return t || u; } };
template<class T, class U>
struct bf_min{ __device__ __host__  T operator()(const T& t, const U& u)      const{ return t<u ? t : u; } };
template<class T, class U>
struct bf_max{ __device__ __host__  T operator()(const T& t, const U& u)      const{ return t>u ? t : u; } };

// functors with parameter
template<class T, class U>
struct bf_axpy{  
	const T a;
	bf_axpy(const T& _a):a(_a){}
	__device__  __host__       T operator()(const T& t, const U& u) const{ return  a*t+(T)u; } 
};
template<class T, class U>
struct bf_xpby{  
	const T b;
	bf_xpby(const T& _b):b(_b){}
	__device__  __host__       T operator()(const T& t, const U& u) const{ return  t+b*(T)u; } 
};
template<class T, class U>
struct bf_axpby{  
	const T a;
	const T b;
	bf_axpby(const T& _a, const T& _b):a(_a),b(_b){}
	__device__  __host__       T operator()(const T& t, const U& u) const{ return  a*t + b*((T)u); } 
};


/*
 * launchers for functors
 */

#if ! USE_THRUST_LAUNCHER
template<class unary_functor, class value_type, class index_type>
__global__
void unary_functor_kernel(value_type* dst, value_type* src, index_type n, unary_functor uf){
	const unsigned int idx = __mul24(blockIdx.x , blockDim.x) + threadIdx.x;
	const unsigned int off = __mul24(blockDim.x , gridDim.x);
	for (unsigned int i = idx; i < n; i += off)
		dst[i] = uf(src[i]);
}

template<class binary_functor, class value_type, class value_type2, class index_type>
__global__
void binary_functor_kernel(value_type* dst, value_type* src, value_type2* src2, index_type n, binary_functor bf){
	const unsigned int idx = __mul24(blockIdx.x , blockDim.x) + threadIdx.x;
	const unsigned int off = __mul24(blockDim.x , gridDim.x);
	for (unsigned int i = idx; i < n; i += off)
		dst[i] = bf(src[i],src2[i]);
}

void setLinearGridAndThreads(dim3& blocks, dim3& threads, size_t len, int threads_per_block=512){
	const int padded_len=(int)ceil((float)len/threads_per_block)*threads_per_block;
	blocks = dim3(min(512,padded_len/threads_per_block),1,1);
	threads = dim3(threads_per_block,1,1);
}
#endif

template<class unary_functor, class value_type, class index_type>
void launch_unary_kernel(
   cuv::vector<value_type,dev_memory_space,index_type>& dst,
   cuv::vector<value_type,dev_memory_space,index_type>& src, 
	 unary_functor uf){
	 cuvAssert(dst.ptr());
	 cuvAssert(src.ptr());
	 cuvAssert(dst.size() == src.size());

#if ! USE_THRUST_LAUNCHER
	 dim3 blocks, threads;
	 setLinearGridAndThreads(blocks,threads,dst.size());
	 unary_functor_kernel<<<blocks,threads>>>(dst.ptr(),src.ptr(),dst.size(),uf); //     180 ms
#else
	 thrust::device_ptr<value_type> dst_ptr(dst.ptr());
	 thrust::device_ptr<value_type> src_ptr(src.ptr());
	 thrust::transform(src_ptr,src_ptr+src.size(),dst_ptr,uf);
#endif

	 cuvSafeCall(hipDeviceSynchronize());
}

template<class unary_functor, class value_type, class index_type>
void launch_unary_kernel(
   cuv::vector<value_type,host_memory_space,index_type>& dst,
   cuv::vector<value_type,host_memory_space,index_type>& src, 
	 unary_functor uf){
	 cuvAssert(src.ptr());
	 cuvAssert(dst.ptr());
	 cuvAssert(dst.size() == src.size());
	 value_type* dst_ptr = dst.ptr();
	 value_type* src_ptr = src.ptr();
	 for(size_t i=0;i<dst.size();i++)
	   *dst_ptr++ = uf( *src_ptr++ );
}

template<class binary_functor, class V1, class V2, class index_type>
void launch_binary_kernel(
   cuv::vector<V1,dev_memory_space,index_type>& v,
   cuv::vector<V2,dev_memory_space,index_type>& w, 
	 binary_functor bf){
	 cuvAssert(v.ptr());
	 cuvAssert(w.ptr());
	 cuvAssert(v.size() == w.size());

#if ! USE_THRUST_LAUNCHER
	 dim3 blocks, threads;
	 setLinearGridAndThreads(blocks,threads,v.size());
	 binary_functor_kernel<<<blocks,threads>>>(v.ptr(),v.ptr(),w.ptr(),v.size(),bf); 
#else
	 thrust::device_ptr<V1> v_ptr(v.ptr());
	 thrust::device_ptr<V2> w_ptr(w.ptr());
	 thrust::transform(v_ptr,v_ptr+v.size(),w_ptr,bf);
#endif

	 cuvSafeCall(hipDeviceSynchronize());
}

template<class binary_functor, class V1, class V2, class index_type>
void launch_binary_kernel(
   cuv::vector<V1,host_memory_space,index_type>& dst,
   cuv::vector<V2,host_memory_space,index_type>& src, 
	 binary_functor uf){
	 cuvAssert(src.ptr());
	 cuvAssert(dst.ptr());
	 cuvAssert(dst.size() == src.size());
	 V1* dst_ptr = dst.ptr();
	 V2* src_ptr = src.ptr();
	 for(size_t i=0;i<dst.size();i++)
	   *dst_ptr++ = uf(*dst_ptr,*src_ptr++);
}

namespace cuv{
	
/*
 * Nullary Functor
 *
 */

template<class __vector_type>
void
apply_0ary_functor(__vector_type& v, const NullaryFunctor& nf){
	 cuvAssert(v.ptr());
	 typedef typename __vector_type::value_type value_type;
	 typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	 ptr_type dst_ptr(v.ptr());
	 switch(nf){
		 case NF_SEQ:
			 thrust::sequence(dst_ptr,dst_ptr+v.size());break;
		 default:
			 cuvAssert(false);
	 }
	 cuvSafeCall(hipDeviceSynchronize());
}

template<class __vector_type, class __value_type>
void
apply_0ary_functor(__vector_type& v, const NullaryFunctor& nf, const __value_type& param){
	 cuvAssert(v.ptr());

	 typedef typename __vector_type::value_type value_type;
	 typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	 ptr_type dst_ptr(v.ptr());
	 switch(nf){
		 case NF_FILL:
			 thrust::fill(dst_ptr,dst_ptr + v.size(), (value_type)param); break;
		 default:
			 cuvAssert(false);
	 }
	 cuvSafeCall(hipDeviceSynchronize());
}

/*
 * Unary Functor
 *
 */
template<class __vector_type>
struct apply_scalar_functor_impl;

template<class __vector_type>
void
apply_scalar_functor(__vector_type& v, const ScalarFunctor& sf){
  apply_scalar_functor_impl<__vector_type>::apply(v,sf);
}
template<class __vector_type, class __value_type>
void
apply_scalar_functor(__vector_type& v, const ScalarFunctor& sf, const __value_type& param){
  apply_scalar_functor_impl<__vector_type>::apply(v,sf,param);
}
template<class __vector_type, class __value_type>
void
apply_scalar_functor(__vector_type& v, const ScalarFunctor& sf, const __value_type& param, const __value_type& param2){
  apply_scalar_functor_impl<__vector_type>::apply(v,sf,param,param2);
}

/*
 * Binary Functor
 *
 */
template<class __vector_type1, class __vector_type2>
void
apply_binary_functor(__vector_type1& v, __vector_type2& w, const BinaryFunctor& sf){
	cuvAssert(v.size() == w.size());
	typedef typename __vector_type1::value_type V1;
	typedef typename __vector_type2::value_type V2;
	typedef typename memspace_cuv2thrustptr<V1,typename __vector_type1::memory_space_type>::ptr_type ptr_type1;
	typedef typename memspace_cuv2thrustptr<V2,typename __vector_type2::memory_space_type>::ptr_type ptr_type2;
	ptr_type1 v_ptr(v.ptr());
	ptr_type2 w_ptr(w.ptr());
#if USE_THRUST_LAUNCHER 
	switch(sf){
		case BF_ADD:      thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_plus<V1,V2>()); break;
		case BF_SUBTRACT: thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_minus<V1,V2>()); break;
		case BF_MULT:     thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_multiplies<V1,V2>()); break;
		case BF_DIV:      thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_divides<V1,V2>()); break;
		case BF_MIN:      thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_min<V1,V2>()); break;
		case BF_MAX:      thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_max<V1,V2>()); break;
		case BF_COPY:     thrust::copy(w_ptr, w_ptr+v.size(), v_ptr); break;
		default: cuvAssert(false);
	}
#else
	dim3 blocks, threads;
	setLinearGridAndThreads(blocks,threads,v.size());
	switch(sf){
		case BF_ADD:      launch_binary_kernel(v,w,bf_plus<V1,V2>()); break;
		case BF_SUBTRACT: launch_binary_kernel(v,w,bf_minus<V1,V2>()); break;
		case BF_MULT:     launch_binary_kernel(v,w,bf_multiplies<V1,V2>()); break;
		case BF_DIV:      launch_binary_kernel(v,w,bf_divides<V1,V2>()); break;
		case BF_MIN:      launch_binary_kernel(v,w,bf_min<V1,V2>()); break;
		case BF_MAX:      launch_binary_kernel(v,w,bf_max<V1,V2>()); break;
		case BF_COPY:     thrust::copy(w_ptr, w_ptr+v.size(), v_ptr); break;
		default: cuvAssert(false);
	}
#endif
	cuvSafeCall(hipDeviceSynchronize());
}

template<class __vector_type1, class __vector_type2, class __value_type>
void
apply_binary_functor(__vector_type1& v, __vector_type2& w, const BinaryFunctor& sf, const __value_type& param){
	cuvAssert(v.size() == w.size());
	typedef typename __vector_type1::value_type V1;
	typedef typename __vector_type2::value_type V2;
	typedef typename memspace_cuv2thrustptr<V1,typename __vector_type1::memory_space_type>::ptr_type ptr_type1;
	typedef typename memspace_cuv2thrustptr<V2,typename __vector_type2::memory_space_type>::ptr_type ptr_type2;
	ptr_type1 v_ptr(v.ptr());
	ptr_type2 w_ptr(w.ptr());
#if USE_THRUST_LAUNCHER
	switch(sf){
		case BF_AXPY:     thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_axpy<V1,V2>(param)); break;
		case BF_XPBY:     thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_xpby<V1,V2>(param)); break;
		/*case BF_XPBY:     hipblasSaxpy(v.size(), param, (float*)w.ptr(), 1, (float*)v.ptr(), 1) ; break;*/
		default: cuvAssert(false);
	}
#else
	dim3 blocks, threads;
	setLinearGridAndThreads(blocks,threads,v.size());
	switch(sf){
		case BF_AXPY:     launch_binary_kernel(v,w,bf_axpy<V1,V2>(param)); break;
		case BF_XPBY:     launch_binary_kernel(v,w,bf_xpby<V1,V2>(param)); break;
		default: cuvAssert(false);
	}
#endif
	cuvSafeCall(hipDeviceSynchronize());
}

template<class __vector_type1, class __vector_type2, class __value_type>
void
apply_binary_functor(__vector_type1& v, __vector_type2& w, const BinaryFunctor& sf, const __value_type& param, const __value_type& param2){
	cuvAssert(v.size() == w.size());
	typedef typename __vector_type1::value_type V1;
	typedef typename __vector_type2::value_type V2;
	typedef typename memspace_cuv2thrustptr<V1,typename __vector_type1::memory_space_type>::ptr_type ptr_type1;
	typedef typename memspace_cuv2thrustptr<V2,typename __vector_type2::memory_space_type>::ptr_type ptr_type2;
	ptr_type1 v_ptr(v.ptr());
	ptr_type2 w_ptr(w.ptr());
#if USE_THRUST_LAUNCHER
	switch(sf){
		case BF_AXPBY:     thrust::transform(v_ptr, v_ptr+v.size(), w_ptr,  v_ptr, bf_axpby<V1,V2>(param,param2)); break;
		default: cuvAssert(false);
	}
#else
	dim3 blocks, threads;
	setLinearGridAndThreads(blocks,threads,v.size());
	switch(sf){
		case BF_AXPBY:     launch_binary_kernel(v,w,bf_axpby<V1,V2>(param,param2)); break;
		default: cuvAssert(false);
	}
#endif
	cuvSafeCall(hipDeviceSynchronize());
}

template<class __vector_type>
struct apply_scalar_functor_impl{


	template<class __arg_value_type>
	static void
	apply(__vector_type& v, const ScalarFunctor& sf, const __arg_value_type& param, const __arg_value_type& param2){
		typedef typename __vector_type::value_type value_type;
		switch(sf){
			case SF_TANH:      launch_unary_kernel(v,v,uf_base_op3<value_type, tf_tanh<value_type> >(param,param2)); break;
			case SF_DTANH:     launch_unary_kernel(v,v,uf_base_op3<value_type, tf_dtanh<value_type> >(param,param2)); break;
			default:
				cuvAssert(false);
		}
	}

	template<class __arg_value_type>
	static void
	apply(__vector_type& v, const ScalarFunctor& sf, const __arg_value_type& param){
		typedef typename __vector_type::value_type value_type;
		switch(sf){
			case SF_SIGM:      launch_unary_kernel(v,v,uf_base_op<value_type, bf_sigm_temp<value_type> >(param)); break;
			case SF_ADD:       launch_unary_kernel(v,v,uf_base_op<value_type, thrust::plus<value_type> >(param)); break;
			case SF_MULT:      launch_unary_kernel(v,v,uf_base_op<value_type, thrust::multiplies<value_type> >(param)); break;
			case SF_DIV:       launch_unary_kernel(v,v,uf_base_op<value_type, thrust::divides<value_type> >(param)); break;
			case SF_SUBTRACT:  launch_unary_kernel(v,v,uf_base_op<value_type, thrust::minus<value_type> >(param)); break;
			case SF_MIN:       launch_unary_kernel(v,v,uf_base_op<value_type, bf_min<value_type,__arg_value_type> >(param)); break;
			case SF_MAX:       launch_unary_kernel(v,v,uf_base_op<value_type, bf_max<value_type,__arg_value_type> >(param)); break;
			case SF_RECT:      launch_unary_kernel(v,v,uf_base_op<value_type, tf_rect<value_type,__arg_value_type> >(param)); break;
			case SF_DRECT:     launch_unary_kernel(v,v,uf_base_op<value_type, tf_drect<value_type,__arg_value_type> >(param)); break;
			default:
				cuvAssert(false);
		}
	}

	static void
	apply(__vector_type& v, const ScalarFunctor& sf){
		typedef typename __vector_type::value_type value_type;
	  switch(sf){
			case SF_EXP:        launch_unary_kernel(v,v, uf_exp<value_type>()); break;
			/*case SF_EXACT_EXP:  launch_unary_kernel(v,v, uf_exact_exp<value_type>()); break;*/
			case SF_LOG:        launch_unary_kernel(v,v, uf_log<value_type>()); break;
			case SF_SIGN:       launch_unary_kernel(v,v, uf_sign<value_type>()); break;
			case SF_SIGM:       launch_unary_kernel(v,v, uf_sigm<value_type>()); break;
			case SF_DSIGM:      launch_unary_kernel(v,v, uf_dsigm<value_type>()); break;
			case SF_TANH:       launch_unary_kernel(v,v, uf_tanh<value_type>()); break;
			case SF_DTANH:      launch_unary_kernel(v,v, uf_dtanh<value_type>()); break;
			case SF_SQUARE:     launch_unary_kernel(v,v, uf_square<value_type>()); break;
			case SF_SUBLIN:     launch_unary_kernel(v,v, uf_sublin<value_type>()); break;
			case SF_ENERG:      launch_unary_kernel(v,v, uf_energ<value_type>()); break;
			case SF_INV:        launch_unary_kernel(v,v, uf_inv<value_type>()); break;
			case SF_SQRT:       launch_unary_kernel(v,v, uf_sqrt<value_type>()); break;
			case SF_SMAX:       launch_unary_kernel(v,v, uf_smax<value_type>()); break;
			case SF_NEGATE:     launch_unary_kernel(v,v, thrust::negate<value_type>()); break;
			case SF_POSLIN:     launch_unary_kernel(v,v, uf_poslin<value_type>()); break;
			default:
			 cuvAssert(false);
		}
	}
};

/*
 * Reductions
 */
template<class __vector_type>
bool
has_inf(__vector_type& v){
	typedef typename __vector_type::value_type value_type;
	typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	ptr_type v_ptr(v.ptr());
	bool init=false;
	uf_is_inf<value_type> uo;
	return  thrust::any_of(v_ptr, v_ptr+v.size(), uo);
}
template<class __vector_type>
bool
has_nan(__vector_type& v){
	typedef typename __vector_type::value_type value_type;
	typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	ptr_type v_ptr(v.ptr());
	bool init=false;
	uf_is_nan<value_type> uo;
	return  thrust::any_of(v_ptr, v_ptr+v.size(), uo);
}
template<class __vector_type>
float
norm2(__vector_type& v){
	typedef typename __vector_type::value_type value_type;
	typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	ptr_type v_ptr(v.ptr());
	float init=0;
	return  std::sqrt( thrust::transform_reduce(v_ptr, v_ptr+v.size(), uf_square<float>(), init, bf_plus<float,value_type>()) );
}
template<class __vector_type>
float
norm1(__vector_type& v){
	typedef typename __vector_type::value_type value_type;
	typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	ptr_type v_ptr(v.ptr());
	float init=0;
	uf_abs<float> unary_op;
	bf_plus<float,value_type> binary_op;
	return   thrust::transform_reduce(v_ptr, v_ptr+v.size(), unary_op, init, binary_op);
}
template<class __vector_type>
float
sum(__vector_type& v){
	typedef typename __vector_type::value_type value_type;
	typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	ptr_type v_ptr(v.ptr());
	float init=0.0;
	return   thrust::reduce(v_ptr, v_ptr+v.size(), init, bf_plus<float,value_type>());
}
template<class __vector_type>
float
maximum(__vector_type& v){
	typedef typename __vector_type::value_type value_type;
	typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	ptr_type v_ptr(v.ptr());
	float init=-INT_MAX;
	return   thrust::reduce(v_ptr, v_ptr+v.size(), init, bf_max<float,value_type>());
}
template<class __vector_type>
float
minimum(__vector_type& v){
	typedef typename __vector_type::value_type value_type;
	typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	ptr_type v_ptr(v.ptr());
	float init=INT_MAX;
	return   thrust::reduce(v_ptr, v_ptr+v.size(), init, bf_min<float,value_type>());
}
template<class __vector_type>
float
mean(__vector_type& v){
	return   sum(v) / (float)v.size();
}
template<class __vector_type>
float
var(__vector_type& v){
	typedef typename __vector_type::value_type value_type;
	typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	ptr_type v_ptr(v.ptr());
	float init=0;
	float m = mean(v);
	return   thrust::transform_reduce(v_ptr, v_ptr+v.size(), uf_base_op<float, bf_squared_diff<float,value_type> >(m), init, bf_plus<float,value_type>()) / (float)v.size();
}
template<class __vector_type>
typename __vector_type::index_type
arg_max(__vector_type& v){
	typedef typename __vector_type::value_type value_type;
	typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	ptr_type begin(v.ptr());
	ptr_type elem = thrust::max_element(begin, begin	+v.size());
	return thrust::distance(begin,elem);
}
template<class __vector_type>
typename __vector_type::index_type
arg_min(__vector_type& v){
	typedef typename __vector_type::value_type value_type;
	typedef typename memspace_cuv2thrustptr<value_type,typename __vector_type::memory_space_type>::ptr_type ptr_type;
	ptr_type begin(v.ptr());
	ptr_type elem = thrust::min_element(begin, begin	+v.size());
	return thrust::distance(begin,elem);
}
/*
 * Template instantiations
 */

#define SIMPLE_0(X,Y) \
	template void apply_0ary_functor< vector<X,Y> >( vector<X,Y>&, const NullaryFunctor&);

#define SIMPLE_01(X,P,Z) \
	template void apply_0ary_functor< vector<X,Z>, P>(vector<X,Z>&, const NullaryFunctor&, const P& param);

#define SIMPLE_1(X,Y) \
	template void apply_scalar_functor< vector<X,Y> >(vector<X,Y>&, const ScalarFunctor&);
#define SIMPLE_11(X,P,Z) \
	template void apply_scalar_functor< vector<X,Z>, P>(vector<X,Z>&, const ScalarFunctor&,const P&); \
	template void apply_scalar_functor< vector<X,Z>, P>(vector<X,Z>&, const ScalarFunctor&,const P&, const P&);

#define SIMPLE_2(X,Y,Z) \
	template void apply_binary_functor< vector<X,Z> , vector<Y,Z>  >( vector<X,Z> &,  vector<Y,Z> &, const BinaryFunctor&);
#define SIMPLE_21(X,Y,P,Z) \
	template void apply_binary_functor<vector<X,Z>,vector<Y,Z>,P>(vector<X,Z>&, vector<Y,Z>&, const BinaryFunctor&,  const P&); \
	template void apply_binary_functor<vector<X,Z>,vector<Y,Z>,P>(vector<X,Z>&, vector<Y,Z>&, const BinaryFunctor&,  const P&, const P&);

#define SIMPLE_NORM(X, Y) \
	template bool has_inf<vector<X,Y> >(vector<X,Y>&); \
	template bool has_nan<vector<X,Y> >(vector<X,Y>&); \
	template float minimum<vector<X,Y> >(vector<X,Y>&); \
	template float maximum<vector<X,Y> >(vector<X,Y>&); \
	template float norm1<vector<X,Y> >(vector<X,Y>&); \
	template float norm2<vector<X,Y> >(vector<X,Y>&); \
	template float mean<vector<X,Y> >(vector<X,Y>&);  \
	template float var<vector<X,Y> >(vector<X,Y>&); \
	template typename vector<X,Y>::index_type     arg_max<vector<X,Y> >(vector<X,Y>&); \
	template typename vector<X,Y>::index_type     arg_min<vector<X,Y> >(vector<X,Y>&);


#define SIMPLE_INSTANTIATOR(X) \
	SIMPLE_0( X , dev_memory_space);             \
	SIMPLE_1( X , dev_memory_space);             \
	SIMPLE_2( X, X , dev_memory_space);          \
    SIMPLE_NORM( X , dev_memory_space);			\
	SIMPLE_0( X , host_memory_space);             \
	SIMPLE_1( X , host_memory_space);             \
	SIMPLE_2( X, X , host_memory_space);          \
    SIMPLE_NORM( X , host_memory_space);

#define SIMPLE_INSTANTIATOR1(X, P) \
	SIMPLE_01( X, P , dev_memory_space);             \
	SIMPLE_11( X, P , dev_memory_space);             \
	SIMPLE_21( X, X, P , dev_memory_space);          \
	SIMPLE_01( X, P , host_memory_space);             \
	SIMPLE_11( X, P , host_memory_space);             \
	SIMPLE_21( X, X, P , host_memory_space);          

SIMPLE_INSTANTIATOR( float );
SIMPLE_INSTANTIATOR1( float, float );
SIMPLE_INSTANTIATOR1( float, int );
SIMPLE_INSTANTIATOR( unsigned char );
SIMPLE_INSTANTIATOR1( unsigned char, unsigned char );
SIMPLE_INSTANTIATOR( signed char );
SIMPLE_INSTANTIATOR1( signed char, unsigned char );
SIMPLE_2(float,unsigned char, dev_memory_space);
SIMPLE_2(float,unsigned char, host_memory_space);
SIMPLE_INSTANTIATOR(int );
SIMPLE_INSTANTIATOR1(int, int );
} // cuv
