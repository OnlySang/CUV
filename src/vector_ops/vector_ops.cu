#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>

#include <cuv_general.hpp>

#include <dev_vector.hpp>
#include <host_vector.hpp>

#include "vector_ops.hpp"

#define sgn(a) (copysign(1.f,a))

using namespace cuv;

template<class T>
struct uf_exp{  __host__ __device__ T operator()(const T& t)const{ return __expf(t);    } };
template<class T>
struct uf_exact_exp{  __device__ __host__ T operator()(const T& t)const{ return exp(t);    } };
template<class T>
struct uf_log{  __device__ __host__ T operator()(const T& t)      const{ return log(t);    } };
template<class T>
struct uf_sign{  __device__         T operator()(const T& t)      const{ return sgn(t);    } };
template<class T>
struct uf_sigm{  __device__         T operator()(const T& t)      const{ return ((T)1)/(((T)1)+__expf(-t));    } };
template<class T>
struct uf_exact_sigm{  __device__  __host__ T operator()(const T& t)      const{ return ((T)1)/(((T)1)+exp(-t));    } };
template<class T>
struct uf_dsigm{  __device__ __host__       T operator()(const T& t)      const{ return t * (((T)1)-t); } };
template<class T>
struct uf_tanh{  __device__  __host__       T operator()(const T& t)      const{ return tanh(t); } };
template<class T>
struct uf_dtanh{  __device__  __host__      T operator()(const T& t)      const{ return ((T)1) - (t*t); } };
template<class T>
struct uf_square{  __device__  __host__     T operator()(const T& t)      const{ return t*t; } };
template<class T>
struct uf_sublin{  __device__  __host__     T operator()(const T& t)      const{ return ((T)1)-t; } };
template<class T>
struct uf_energ{  __device__  __host__      T operator()(const T& t)      const{ return -log(t); } };
template<class T>
struct uf_inv{  __device__  __host__        T operator()(const T& t)      const{ return ((T)1)/(t+((T)0.00000001)); } };
template<class T>
struct uf_sqrt{  __device__  __host__       T operator()(const T& t)      const{ return sqrt(t); } };

template<class T, class binary_functor>
struct uf_base_op{
  const T x;
  const binary_functor bf;
  uf_base_op(const T& _x):x(_x),bf(){};
  T operator()(T t){ return bf(t,x); }
};


template<class unary_functor, class value_type, class index_type>
void launch_unary_kernel(
   cuv::dev_vector<value_type, index_type>& dst,
   cuv::dev_vector<value_type, index_type>& src, 
	 unary_functor uf){
	 cuvAssert(dst.ptr());
	 cuvAssert(src.ptr());
	 cuvAssert(dst.size() == src.size());

	 thrust::device_ptr<value_type> dst_ptr(dst.ptr());
	 thrust::device_ptr<value_type> src_ptr(src.ptr());
	 thrust::transform(src_ptr,src_ptr+src.size(),dst_ptr,uf);
	 cuvSafeCall(hipDeviceSynchronize());
}

template<class unary_functor, class value_type, class index_type>
void launch_unary_kernel(
   cuv::host_vector<value_type, index_type>& dst,
   cuv::host_vector<value_type, index_type>& src, 
	 unary_functor uf){
	 cuvAssert(src.ptr());
	 cuvAssert(dst.ptr());
	 cuvAssert(dst.size() == src.size());
	 for(size_t i=0;i<dst.size();i++)
	   dst[i] = uf(src[i]);
}

namespace cuv{

template<class __vector_type>
struct apply_scalar_functor_impl;

template<class __vector_type>
void
apply_scalar_functor(__vector_type& v, const ScalarFunctor& sf){
  apply_scalar_functor_impl<__vector_type>::apply(v,sf);
}
template<class __vector_type, class __value_type>
void
apply_scalar_functor(__vector_type& v, const ScalarFunctor& sf, const __value_type& param){
  apply_scalar_functor_impl<__vector_type>::apply(v,sf,param);
}

template<class __vector_type>
struct apply_scalar_functor_impl{

	template<class __arg_value_type>
	static void
	apply(__vector_type& v, const ScalarFunctor& sf, const __arg_value_type& param){
		typedef typename __vector_type::value_type value_type;
		switch(sf){
			case SF_ADD:       launch_unary_kernel(v,v,uf_base_op<value_type, thrust::plus<value_type> >(param)); break;
			case SF_MULT:      launch_unary_kernel(v,v,uf_base_op<value_type, thrust::multiplies<value_type> >(param)); break;
			case SF_DIV:       launch_unary_kernel(v,v,uf_base_op<value_type, thrust::divides<value_type> >(param)); break;
			case SF_SUBTRACT:  launch_unary_kernel(v,v,uf_base_op<value_type, thrust::minus<value_type> >(param)); break;
		}
	}

	static void
	apply(__vector_type& v, const ScalarFunctor& sf){
		typedef typename __vector_type::value_type value_type;
	  switch(sf){
			case SF_EXP:        launch_unary_kernel(v,v, uf_exp<value_type>()); break;
			case SF_EXACT_EXP:  launch_unary_kernel(v,v, uf_exact_exp<value_type>()); break;
			case SF_LOG:        launch_unary_kernel(v,v, uf_log<value_type>()); break;
			case SF_SIGN:       launch_unary_kernel(v,v, uf_sign<value_type>()); break;
			case SF_SIGM:       launch_unary_kernel(v,v, uf_sigm<value_type>()); break;
			case SF_DSIGM:      launch_unary_kernel(v,v, uf_dsigm<value_type>()); break;
			case SF_TANH:       launch_unary_kernel(v,v, uf_tanh<value_type>()); break;
			case SF_DTANH:      launch_unary_kernel(v,v, uf_dtanh<value_type>()); break;
			case SF_SQUARE:     launch_unary_kernel(v,v, uf_square<value_type>()); break;
			case SF_SUBLIN:     launch_unary_kernel(v,v, uf_sublin<value_type>()); break;
			case SF_ENERG:      launch_unary_kernel(v,v, uf_energ<value_type>()); break;
			case SF_INV:        launch_unary_kernel(v,v, uf_inv<value_type>()); break;
			case SF_SQRT:       launch_unary_kernel(v,v, uf_sqrt<value_type>()); break;
			case SF_NEGATE:     launch_unary_kernel(v,v, thrust::negate<value_type>()); break;
			default:
			 cuvAssert(false);
		}
	}
};

template void apply_scalar_functor<dev_vector<float> >(dev_vector<float>&, const ScalarFunctor&);
template void apply_scalar_functor<dev_vector<float>, float>(dev_vector<float>&, const ScalarFunctor&,const float&);
template void apply_scalar_functor<dev_vector<float>, int>(dev_vector<float>&, const ScalarFunctor&,const int&);
} // cuv
