#include "hip/hip_runtime.h"
#include <dev_dia_matrix.hpp>
#include <host_dia_matrix.hpp>
#include <vector_ops/vector_ops.hpp>
#include <convert.hpp>

namespace cuv{

		namespace convert_impl{
			/*
			 * Vector Conversion
			 */


			// host  --> dev 
			template<class __value_type, class __index_type>
				static void
				convert(        dev_vector <__value_type,   __index_type>& dst, 
						const host_vector<__value_type,  __index_type>& src){
					if(        dst.size() != src.size()){
						dev_vector<__value_type,__index_type> d(src.size());
						dst = d;
					}
					cuvSafeCall(hipMemcpy(dst.ptr(),src.ptr(),dst.memsize(),hipMemcpyHostToDevice));
				}

			// dev  --> host  
			template<class __value_type, class __index_type>
				static void
				convert(        host_vector<__value_type,  __index_type>& dst, 
						const dev_vector<__value_type,  __index_type>& src){
					if( dst.size() != src.size()){
						host_vector<__value_type,__index_type> h(src.size());
						dst = h;
					}
					cuvSafeCall(hipMemcpy(dst.ptr(),src.ptr(),dst.memsize(),hipMemcpyDeviceToHost));
				}

			/*
			 * Matrix Conversion
			 */

			// host (row-major) --> dev (col-major) 
			template<class __value_type, class __index_type>
				static void
				convert(        dev_dense_matrix <__value_type,  column_major,  __index_type>& dst, 
						const host_dense_matrix<__value_type,  row_major, __index_type>& src){
					if(        dst.h() != src.w()
							|| dst.w() != src.h()){

						dev_dense_matrix<__value_type,column_major,__index_type> d(src.w(),src.h());
						dst = d;
					}
					convert(dst.vec(), src.vec());
				}

			// dev (col-major) --> host (row-major) 
			template<class __value_type, class __index_type>
				static void
				convert(        host_dense_matrix<__value_type,  row_major,  __index_type>& dst, 
						const dev_dense_matrix<__value_type,  column_major, __index_type>& src){
					if(        dst.h() != src.w()
							|| dst.w() != src.h()){
						host_dense_matrix<__value_type,row_major,__index_type> h(src.w(),src.h());
						dst = h;
					}
					convert(dst.vec(), src.vec());
				}

			// host (col-major) --> dev (row-major) 
			template<class __value_type, class __index_type>
				static void
				convert(        dev_dense_matrix <__value_type,  row_major,  __index_type>& dst, 
						const host_dense_matrix<__value_type,  column_major, __index_type>& src){
					if(        dst.h() != src.w()
							|| dst.w() != src.h()){

						dev_dense_matrix<__value_type,row_major,__index_type> d(src.w(),src.h());
						dst = d;
					}
					convert(dst.vec(), src.vec());
				}

			// dev (row-major) --> host (col-major) 
			template<class __value_type, class __index_type>
				static void
				convert(        host_dense_matrix<__value_type,  column_major,  __index_type>& dst, 
						const dev_dense_matrix<__value_type,  row_major, __index_type>& src){
					if(        dst.h() != src.w()
							|| dst.w() != src.h()){
						host_dense_matrix<__value_type,column_major,__index_type> h(src.w(),src.h());
						dst = h;
					}
					convert(dst.vec(), src.vec());
				}

			/*
			 * Simple copying
			 *
			 */

			// dev (col-major) --> host (col-major) 
			template<class __value_type, class __index_type>
				static void
				convert(        host_dense_matrix<__value_type,  column_major, __index_type>& dst, 
						const    dev_dense_matrix<__value_type,  column_major, __index_type>& src){
					if(        dst.h() != src.h()
							|| dst.w() != src.w()){
						host_dense_matrix<__value_type,column_major,__index_type> h(src.h(),src.w());
						dst = h;
					}
					convert(dst.vec(), src.vec());
				}

			// dev (col-major) --> host (col-major) 
			template<class __value_type, class __index_type>
				static void
				convert(         dev_dense_matrix<__value_type,  column_major, __index_type>& dst, 
						const   host_dense_matrix<__value_type,  column_major, __index_type>& src){
					if(        dst.h() != src.h()
							|| dst.w() != src.w()){
						dev_dense_matrix<__value_type,column_major,__index_type> h(src.h(),src.w());
						dst = h;
					}
					convert(dst.vec(), src.vec());
				}

			// dev (row-major) --> host (row-major) 
			template<class __value_type, class __index_type>
				static void
				convert(        host_dense_matrix<__value_type,  row_major,  __index_type>& dst, 
						const dev_dense_matrix<__value_type,  row_major, __index_type>& src){
					if(        dst.h() != src.h()
							|| dst.w() != src.w()){
						host_dense_matrix<__value_type,row_major,__index_type> h(src.h(),src.w());
						dst = h;
					}
					convert(dst.vec(), src.vec());
				}

			// host (row-major) --> dev (row-major) 
			template<class __value_type, class __index_type>
				static void
				convert(         dev_dense_matrix<__value_type,  row_major,  __index_type>& dst, 
						const host_dense_matrix<__value_type,  row_major, __index_type>& src){
					if(        dst.h() != src.h()
							|| dst.w() != src.w()){
						dev_dense_matrix<__value_type,row_major,__index_type> h(src.h(),src.w());
						dst = h;
					}
					convert(dst.vec(), src.vec());
				}

			/*
			 * Host Dia -> Host Dense
			 */
			template<class __value_type, class __mem_layout_type, class __index_type>
				static void
				convert(      host_dense_matrix <__value_type,   __mem_layout_type, __index_type>& dst, 
						const host_dia_matrix<__value_type,  __index_type>& src){
					if(        dst.h() != src.h()
							|| dst.w() != src.w()
							){
						host_dense_matrix<__value_type,__mem_layout_type,__index_type> d(src.h(),src.w());
						dst = d;
					}
					fill(dst.vec(),0);
					const host_vector<int>& off = src.get_offsets();
					using namespace std;
					const int rf = src.row_fact();
					for(unsigned int oi=0; oi < off.size(); oi++){
						int o = off[oi];
						__index_type j = 1 *max((int)0, o);
						__index_type i = rf*max((int)0,-o);
						for(;i<src.h() && j<src.w(); j++){
							for(int k=0;k<rf;k++,i++)
								dst.set(i,j, src(i,j));
						}
					}
				}

			/*
			 * Host Dia -> Dev Dia
			 */
			template<class __value_type, class __index_type>
				static void
				convert(      dev_dia_matrix <__value_type, __index_type>& dst, 
						const host_dia_matrix<__value_type, __index_type>& src){
					if(        dst.h() != src.h()
							|| dst.w() != src.w()
							|| dst.row_fact() != src.row_fact()
							|| !dst.vec_ptr()
							){
						dst.dealloc();
						dst = dev_dia_matrix<__value_type,__index_type>(src.h(),src.w(),src.num_dia(),src.stride(),src.row_fact());
					}
					cuv::convert(dst.get_offsets(), src.get_offsets());
					cuv::convert(dst.vec(), src.vec());
					dst.post_update_offsets();
				}

			/*
			 * Dev Dia -> Host Dia
			 */
			template<class __value_type, class __index_type>
				static void
				convert(      host_dia_matrix <__value_type, __index_type>& dst, 
						const dev_dia_matrix<__value_type, __index_type>& src){
					if(        dst.h() != src.h()
							|| dst.w() != src.w()
							|| dst.row_fact() != src.row_fact()
							|| !dst.vec_ptr()
							){
						dst.dealloc();
						dst = host_dia_matrix<__value_type,__index_type>(src.h(),src.w(),src.num_dia(),src.stride(),src.row_fact());
					}
					cuv::convert(dst.get_offsets(), src.get_offsets());
					cuv::convert(dst.vec(), src.vec());
					dst.post_update_offsets();
				}


		};
	template<class Dst, class Src>
		void convert(Dst& dst, const Src& src)
		{
			convert_impl::convert<typename Dst::value_type, typename Dst::index_type>(dst,src); // hmm the compiler should deduce template args, but it fails to do so.
		};

#define CONV_VEC(X) \
	template void convert<dev_vector<X>,          host_vector<X> > \
		(                 dev_vector<X>&,   const host_vector<X>&); \
	template void convert<host_vector<X>,          dev_vector<X> > \
		(                 host_vector<X>&,   const dev_vector<X>&);

#define CONV_INST(X,Y,Z) \
	template void convert<dev_dense_matrix<X,Y>,          host_dense_matrix<X,Z> > \
		(                 dev_dense_matrix<X,Y>&,   const host_dense_matrix<X,Z>&); \
	template void convert<host_dense_matrix<X,Y>,         dev_dense_matrix<X,Z> > \
		(                 host_dense_matrix<X,Y>&,  const dev_dense_matrix<X,Z>&);

CONV_INST(float,column_major,column_major);
CONV_INST(float,column_major,row_major);
CONV_INST(float,row_major,   column_major);
CONV_INST(float,row_major,   row_major);

CONV_INST(unsigned char,column_major,column_major);
CONV_INST(unsigned char,column_major,row_major);
CONV_INST(unsigned char,row_major,   column_major);
CONV_INST(unsigned char,row_major,   row_major);

CONV_INST(signed char,column_major,column_major);
CONV_INST(signed char,column_major,row_major);
CONV_INST(signed char,row_major,   column_major);
CONV_INST(signed char,row_major,   row_major);

CONV_VEC(float);
CONV_VEC(int);
CONV_VEC(unsigned char);
CONV_VEC(signed char);

#define DIA_DENSE_CONV(X,Y,Z) \
	template <>                           \
		void convert(host_dense_matrix<X,Y,Z>& dst, const host_dia_matrix<X,Z>& src)     \
		{                                                                                \
			typedef host_dense_matrix<X,Y,Z> Dst;                                        \
			convert_impl::convert<typename Dst::value_type, typename Dst::memory_layout, typename Dst::index_type>(dst,src);  \
		};   
#define DIA_HOST_DEV_CONV(X,Z) \
	template <>                           \
		void convert(dev_dia_matrix<X,Z>& dst, const host_dia_matrix<X,Z>& src)     \
		{                                                                                \
			typedef dev_dia_matrix<X,Z> Dst;                                        \
			convert_impl::convert<typename Dst::value_type, typename Dst::index_type>(dst,src);  \
		};                                \
	template <>                           \
		void convert(host_dia_matrix<X,Z>& dst, const dev_dia_matrix<X,Z>& src)     \
		{                                                                                \
			typedef host_dia_matrix<X,Z> Dst;                                        \
			convert_impl::convert<typename Dst::value_type, typename Dst::index_type>(dst,src);  \
		}; 
        
DIA_DENSE_CONV(float,column_major,unsigned int)
DIA_DENSE_CONV(float,row_major,unsigned int)
DIA_HOST_DEV_CONV(float,unsigned int)


} // namespace cuv


