#include "hip/hip_runtime.h"
#include <memory>
#include <numeric>
#include "vector_ops/vector_ops.hpp"
#include "diagonals.hpp"

namespace cuv{

	namespace avg_diagonals_impl{

		/***********************************************************
		 * With vector result
		 ***********************************************************/
		template<class T,class I>
			void avg_diagonals(
					cuv::vector<T,dev_memory_space,I>& dst,
					const cuv::dia_matrix<T,dev_memory_space,I>& dia
					){
				for( int i=0;i<dia.num_dia();i++ ){
					std::auto_ptr<const vector<T,dev_memory_space> > diagonal ( dia.get_dia( dia.get_offset( i ) ));
					dst.set( i , mean( *const_cast<vector<T, dev_memory_space>* >(diagonal.get()) ) );
				}
			}
		template<class T,class I>
			void avg_diagonals(
					cuv::vector<T,host_memory_space,I>& dst,
					const cuv::dia_matrix<T,host_memory_space,I>& dia
					){
				cuvAssert( dia.row_fact( )==1 );
				cuvAssert( dia.num_dia( )==dst.size( ));
				typedef I index_type;
				typedef T value_type;
				unsigned int A_stride = dia.stride();
				unsigned int A_h      = dia.h();
				unsigned int A_w      = dia.w();
				const cuv::vector<int,host_memory_space>& offsets = dia.get_offsets();
				for( unsigned int i=0; i<dia.num_dia(); i++ ){
					T sum=0;
					const int k = offsets[i];  //diagonal offset
					const index_type i_start = std::max((int)0,-k);
					const index_type j_start = std::max((int)0, k);

					//number of elements to process
					const index_type N = std::min(A_h - i_start, (A_w - j_start));
					const value_type * d = dia.vec().ptr() + i*A_stride + i_start;
					for( int j=0;j<N;j++)
						sum += *d++;
					dst.set( i , sum/N);
				}
			}

		template<int BLOCK_DIM, class value_type, class index_type>
		__global__ void dia2toeplitz_kernel(
				value_type* tp_data,
				const value_type* dia_data,
				const int* dia_offsets,
				index_type dia_w,
				index_type dia_stride,
				index_type w,
				index_type im,
				index_type om
				){
			index_type dia = blockIdx.x; // the (running) number of the diagonal
			index_type bi  = blockIdx.y; // the block offset (an index along the diagonal to determine in which input/output map we are)
			/*index_type nd  = gridDim.x;  // the total number of diagonals*/

			__shared__ int off;          // the offset of the diagonal (0 for main diagonal, negative to the left, positive to the right)
			__shared__ index_type i_start;          
			__shared__ index_type N;          
			__shared__ index_type startx;          
			if( threadIdx.x == 0 ){
				off = dia_offsets[ dia ];
				int virtual_om  = rintf( off/float( w ));
				int virtual_off = off - virtual_om*w;
				index_type j_start;          
				i_start = max( (int)0, -virtual_off );
				j_start = max( (int)0,  virtual_off );
				N       = min( w-i_start, w-j_start );
				startx  = i_start + bi*w;
			}
			__syncthreads();
			if( startx+N > dia_w )
			  return;


			__shared__ value_type sSums[ BLOCK_DIM ];
			value_type sum = ( value_type ) 0;
			for (int idx = dia*dia_stride + startx + threadIdx.x;
					 idx < dia*dia_stride + startx + N;
					 idx += blockDim.x) {
				value_type f = dia_data[ idx ];
				sum += f;
			}
			sSums[ threadIdx.x ] = sum;
			__syncthreads();

			int offset = blockDim.x / 2;
			while (offset > 0) {
				if (threadIdx.x < offset) 
					sSums[threadIdx.x] += sSums[threadIdx.x + offset];
				offset >>= 1;
			}
			__syncthreads();

			if( threadIdx.x == 0 )
				tp_data[ dia * im + bi ] = sSums[ 0 ]/N;
		}

		/***********************************************************
		 * With toeplitz_matrix result
		 ***********************************************************/
		template<class T,class I>
			void avg_diagonals(
					cuv::toeplitz_matrix<T,dev_memory_space,I>& dst,
					const cuv::dia_matrix<T,dev_memory_space,I>& dia
					){
				cuvAssert( dst.w()==dia.w() );
				cuvAssert( dst.h()==dia.h() );
				I w = dst.w()/dst.output_maps();

				static const int BLOCK_DIM = 256;
				dim3 threads( BLOCK_DIM );
				dim3 grid ( dst.num_dia(), dst.input_maps() );
				dia2toeplitz_kernel<BLOCK_DIM><<<grid,threads>>>(
						dst.vec().ptr(),
						dia.vec().ptr(),
						dia.get_offsets().ptr(),
						dia.w(),
						dia.stride(),
						w,
						dst.input_maps(),
						dst.output_maps() );
				cuvSafeCall( hipDeviceSynchronize() );
			}

		template<class T,class I>
			void avg_diagonals(
					cuv::toeplitz_matrix<T,host_memory_space,I>& dst,
					const cuv::dia_matrix<T,host_memory_space,I>& dia
					){
				cuvAssert( dst.w()==dia.w() );
				cuvAssert( dst.h()==dia.h() );

				typedef I index_type;
				typedef T value_type;

				unsigned int w = dst.w()/dst.output_maps();

				for( int d=0;d<dia.num_dia();d++ ){
					const int off = dia.get_offset( d );  //diagonal offset
					const int virtual_om  = rintf( off/float(w) ); // "output_map" which can be outside matrix (=negative)
					const int virtual_off = off - virtual_om*w;    // the offset relative to the virtual output_map

					const index_type i_start = std::max((int)0,-virtual_off);
					const index_type j_start = std::max((int)0, virtual_off);

					const index_type N = std::min(w - i_start, w - j_start);

					for( int im=0;im<dst.input_maps();im++ ){
						int startx=i_start + im*w;              // x-coordinate of  upper left corner of this diagonal
						if( startx + N > dst.w() )
						   break;
						const T* data = dia.vec().ptr() + d*dia.stride() + startx;
						T avg=std::accumulate(data,data+N,(T)0)/N;
                        /*
						 *for( int i=0;i<N;i++ )
						 *    const_cast<T*>( data )[ i ] = i;
                         */
						dst.vec().set( d*dst.input_maps() + im, avg );
					}
				}
			}
	}

	template<class T, class M, class I>
	void avg_diagonals( cuv::vector<T,M,I>& dst, const cuv::dia_matrix<T,M>& dia ){
		avg_diagonals_impl::avg_diagonals( dst, dia );
	}

	template<class T, class M, class I>
	void avg_diagonals( cuv::toeplitz_matrix<T,M,I>& dst, const cuv::dia_matrix<T,M>& dia ){
		avg_diagonals_impl::avg_diagonals( dst, dia );
	}

	template void
	avg_diagonals(cuv::vector<float,host_memory_space,unsigned int>&, const cuv::dia_matrix<float,host_memory_space>&);
	template void
	avg_diagonals(cuv::vector<float,dev_memory_space,unsigned int>&, const cuv::dia_matrix<float,dev_memory_space>&);

	template void
	avg_diagonals(cuv::toeplitz_matrix<float,host_memory_space,unsigned int>&, const cuv::dia_matrix<float,host_memory_space>&);
	template void
	avg_diagonals(cuv::toeplitz_matrix<float,dev_memory_space,unsigned int>&, const cuv::dia_matrix<float,dev_memory_space>&);
}
