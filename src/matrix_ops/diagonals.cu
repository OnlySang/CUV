#include "hip/hip_runtime.h"
#include <memory>
#include <numeric>
#include "vector_ops/vector_ops.hpp"
#include "diagonals.hpp"

namespace cuv{

	namespace avg_diagonals_impl{

		/***********************************************************
		 * With vector result
		 ***********************************************************/
		template<class T,class I>
			void avg_diagonals(
					cuv::vector<T,dev_memory_space,I>& dst,
					const cuv::dia_matrix<T,dev_memory_space,I>& dia
					){
				for( int i=0;i<dia.num_dia();i++ ){
					std::auto_ptr<const vector<T,dev_memory_space> > diagonal ( dia.get_dia( dia.get_offset( i ) ));
					dst.set( i , mean( *(diagonal.get()) ) );
				}
			}
		template<class T,class I>
			void avg_diagonals(
					cuv::vector<T,host_memory_space,I>& dst,
					const cuv::dia_matrix<T,host_memory_space,I>& dia
					){
				cuvAssert( dia.row_fact( )==1 );
				cuvAssert( dia.num_dia( )==dst.size( ));
				typedef I index_type;
				typedef T value_type;
				unsigned int A_stride = dia.stride();
				unsigned int A_h      = dia.h();
				unsigned int A_w      = dia.w();
				const cuv::vector<int,host_memory_space>& offsets = dia.get_offsets();
				for( unsigned int i=0; i<dia.num_dia(); i++ ){
					T sum=0;
					const int k = offsets[i];  //diagonal offset
					const index_type i_start = std::max((int)0,-k);
					const index_type j_start = std::max((int)0, k);

					//number of elements to process
					const index_type N = std::min(A_h - i_start, (A_w - j_start));
					const value_type * d = dia.vec().ptr() + i*A_stride + i_start;
					for( int j=0;j<N;j++)
						sum += *d++;
					dst.set( i , sum/N);
				}
			}
	}

	template<class T, class M, class I>
	void avg_diagonals( cuv::vector<T,M,I>& dst, const cuv::dia_matrix<T,M>& dia ){
		avg_diagonals_impl::avg_diagonals( dst, dia );
	}

	template void
	avg_diagonals(cuv::vector<float,host_memory_space,unsigned int>&, const cuv::dia_matrix<float,host_memory_space>&);
	template void
	avg_diagonals(cuv::vector<float,dev_memory_space,unsigned int>&, const cuv::dia_matrix<float,dev_memory_space>&);

}
