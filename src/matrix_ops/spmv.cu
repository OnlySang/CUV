#include "hip/hip_runtime.h"
//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*





#include <iostream>
#include <boost/any.hpp>
#include <basics/dia_matrix.hpp>
#include <basics/toeplitz_matrix.hpp>
#include "matrix_ops.hpp"
#include <texture.h>
#include <boost/preprocessor/arithmetic/inc.hpp>
#include <boost/preprocessor/cat.hpp>
#include <boost/preprocessor/repetition/repeat.hpp>
#include <boost/preprocessor/seq/for_each_product.hpp>
#include <boost/preprocessor/seq/to_tuple.hpp>


using namespace std;

// stuff from NVIDIA SDK
#define DIVIDE_INTO(x,y) ((x + y - 1)/y)
#define small_grid_thread_id(void) ((__umul24(blockDim.x, blockIdx.x) + threadIdx.x))
#define large_grid_thread_id(void) ((__umul24(blockDim.x,blockIdx.x + __umul24(blockIdx.y,gridDim.x)) + threadIdx.x))
#define large_grid_thread_num(void) ((__umul24(blockDim.x,gridDim.x + __umul24(blockDim.y,gridDim.y))))

#define MAX_NUM_IMGS_AT_ONCE 14
#define SEQ_ROW_FACT         1
#define SPMM_BLOCK_SIZE      256


namespace cuv{
	namespace spmv_impl{
		/*
		 *  For a given number of blocks, return a 2D grid large enough to contain them
		 *  FROM NVIDIA SDK
		 */
		dim3 make_large_grid(const unsigned int num_blocks){
			if (num_blocks <= 65535){
				return dim3(num_blocks);
			} else {
				unsigned int side = (unsigned int) ceil(sqrt((double)num_blocks));
				return dim3(side,side);
			}
		}

		dim3 make_large_grid(const unsigned int num_threads, const unsigned int blocksize){
			const unsigned int num_blocks = DIVIDE_INTO(num_threads, blocksize);
			if (num_blocks <= 65535){
				//fits in a 1D grid
				return dim3(num_blocks);
			} else {
				//2D grid is required
				const unsigned int side = (unsigned int) ceil(sqrt((double)num_blocks));
				return dim3(side,side);
			}
		}

// this file is generated using a perl-script from spmv_kernel.cuh
#include "spmv_dia_kernel_inst.cuh"
#include "spmv_toeplitz_kernel_inst.cuh"

		template <typename value_type, typename index_type>
			void spmv_dia_device(const dia_matrix<value_type,dev_memory_space,index_type>& A, 
					const vector<value_type,dev_memory_space>& v, 
					vector<value_type,dev_memory_space>& dst, 
					char transA,
					const value_type& factAv,
					const value_type& factC)
			{
				const unsigned int toff = bind_x(v.ptr(), v.size());
				spmm_device_dia_dispatch(A,v,dst,transA,factAv,factC,toff);
				cuvSafeCall(hipDeviceSynchronize());
				unbind_x(v.ptr());
			}

		template <typename value_type, typename index_type>
			void spmv_toeplitz_device(const toeplitz_matrix<value_type,dev_memory_space,index_type>& A, 
					const vector<value_type,dev_memory_space>& v, 
					vector<value_type,dev_memory_space>& dst, 
					char transA,
					const value_type& factAv,
					const value_type& factC)
			{
				const unsigned int toff = bind_x(v.ptr(), v.size());
				spmm_device_toeplitz_dispatch(A,v,dst,transA,factAv,factC,toff);
				cuvSafeCall(hipDeviceSynchronize());
				unbind_x(v.ptr());
			}

		/*template <bool transA, typename value_type, typename index_type>*/
		/*    void spmv_dia_tex_device(const dia_matrix<value_type,dev_memory_space,index_type>& A, */
		/*            const vector<value_type,dev_memory_space>& v, */
		/*            vector<value_type,dev_memory_space>& dst)*/
		/*    {*/
		/*        const unsigned int BLOCK_SIZE = 256;*/
		/*        const dim3 grid = make_large_grid(A.h(),BLOCK_SIZE);*/

		/*        cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals*/

		/*        bind_x(v.ptr());*/

		/*        if(!transA){*/
		/*            const unsigned int BLOCK_SIZE = 256;*/
		/*            const dim3 grid = make_large_grid(A.h(),BLOCK_SIZE);*/
		/*            cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals*/
		/*            spmv_dia_kernel<value_type, index_type, BLOCK_SIZE, true> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec().ptr(), v.ptr(), dst.ptr());*/
		/*        }else{*/
		/*            const unsigned int BLOCK_SIZE = 256;*/
		/*            const dim3 grid = make_large_grid(A.w(),BLOCK_SIZE);*/
		/*            cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals*/
		/*            spmv_dia_kernel_trans<value_type, index_type, BLOCK_SIZE, true> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec().ptr(), v.ptr(), dst.ptr());*/
		/*        }*/

		/*        unbind_x(v.ptr());*/
		/*    }*/
		template<class value_type, class index_type>
			void spmv(vector<value_type,dev_memory_space,index_type>& dst, toeplitz_matrix<value_type,dev_memory_space,index_type>& A, vector<value_type,dev_memory_space,index_type>& v, char transA, const float& factAv, const float& factC){
				// TODO: find a good assert
				/*if(transA=='t'){*/
					/*cuvAssert(A.w() == dst.size());*/
				/*}else{*/
					/*cuvAssert(A.h() == dst.size());*/
				/*}*/
				spmv_toeplitz_device(A,v,dst,transA,factAv,factC);
			}
		template<class value_type, class index_type>
			void spmv(vector<value_type,dev_memory_space,index_type>& dst, dia_matrix<value_type,dev_memory_space,index_type>& A, vector<value_type,dev_memory_space,index_type>& v, char transA, const float& factAv, const float& factC){
				// TODO: find a good assert
				/*if(transA=='t'){*/
					/*cuvAssert(A.w() == dst.size());*/
				/*}else{*/
					/*cuvAssert(A.h() == dst.size());*/
				/*}*/
				spmv_dia_device(A,v,dst,transA,factAv,factC);
			}


		/****************************************************************
		 *  Host Code
		 ****************************************************************/
		template<class value_type, class index_type>
			void spmv(vector<value_type,host_memory_space,index_type>& dst, dia_matrix<value_type,host_memory_space,index_type>& A, vector<value_type,host_memory_space,index_type>& v, char transA, const float& factAv, const float& factC){
				const vector<int,host_memory_space>& offsets = A.get_offsets();
				const int num_diags             = A.num_dia();
				const int A_h                   = A.h();
				const int A_w                   = A.w();
				const int A_stride              = A.stride();
				index_type max_dst = ((transA=='t') ? A_w : A_h);
				if(factC==0.f)
					for(int i=0;i<max_dst;i++) dst.set(i, 0);
				else
					for(int i=0;i<max_dst;i++) dst.set(i, dst[i] * factC);
				const int rf = A.row_fact();
				if(transA == 't'){
					cuvAssert(A_h == v.size());
					cuvAssert(A_w == dst.size());
					for(index_type i = 0; i < num_diags; i++){
						const int k = offsets[i];  //diagonal offset

						const index_type i_start =  1 * std::max((int)0, k);
						const index_type j_start = rf * std::max((int)0,-k); // the matrix is now _wider_ than high --> stretch columns!

						//number of elements to process
						const index_type N = std::min((A_h - j_start)/rf, A_w - i_start);

						const value_type * d_ = A.vec().ptr() + i*A_stride + j_start;
						const value_type * x_ = v.ptr() + j_start;
						value_type * y_ = dst.ptr() + i_start;

						for(index_type n = 0; n < N; n++,y_++){
							for(int k=0;k<rf;k++,x_++,d_++)
								*y_ += factAv * *d_ * *x_;
						}
					}
				}else{
					cuvAssert(A_w == v.size());
					cuvAssert(A_h == dst.size());
					for(index_type i = 0; i < num_diags; i++){
						const int k = offsets[i];  //diagonal offset

						const index_type i_start = rf*std::max((int)0,-k);
						const index_type j_start =  1*std::max((int)0, k);

						//number of elements to process
						const index_type N = std::min(A_h - i_start, rf*(A_w - j_start));

						const value_type * d_ = A.vec().ptr() + i*A_stride + i_start;
						const value_type * x_ = v.ptr() + j_start;
						value_type * y_ = dst.ptr() + i_start;

						for(index_type n = 0; n < N; n++){
							*y_++ += factAv * *d_++ * x_[n/rf];
						}
					}
				}
			}

		template<class value_type, class index_type>
			void spmv(vector<value_type,host_memory_space,index_type>& dst, toeplitz_matrix<value_type,host_memory_space,index_type>& A, vector<value_type,host_memory_space,index_type>& v, char transA, const float& factAv, const float& factC){
				const vector<int,host_memory_space>& offsets = A.get_offsets();
				const int num_diags             = A.num_dia();
				const int A_h                   = A.h();
				const int A_w                   = A.w();
				index_type max_dst = ((transA=='t') ? A_w : A_h);
				if(factC==0.f)
					for(int i=0;i<max_dst;i++) dst.set(i, 0);
				else
					for(int i=0;i<max_dst;i++) dst.set(i, dst[i] * factC);
				if(transA == 't'){
					cuvAssert(A_h == v.size());
					cuvAssert(A_w == dst.size());
					for( unsigned int j=0; j<A.w(); j++ ){
						value_type sum = (value_type) 0;
						for( unsigned int i=0; i<A.h();i++ )
							sum += A( i,j ) * v[i];
						dst.set( j, dst[ j ] + factAv*sum );
					}

/*
 *                    for(index_type i = 0; i < num_diags; i++){
 *                        const int k = offsets[i];  //diagonal offset
 *
 *                        const index_type i_start =  1 * std::max((int)0, k);
 *                        const index_type j_start =  1 * std::max((int)0,-k); // the matrix is now _wider_ than high --> stretch columns!
 *
 *                        //number of elements to process
 *                        const index_type N = std::min((A_h - j_start), A_w - i_start);
 *
 *                        const value_type * d_ = A.vec().ptr() + i;
 *                        const value_type * x_ = v.ptr() + j_start;
 *                        value_type * y_ = dst.ptr() + i_start;
 *
 *                        for(index_type n = 0; n < N; n++,y_++,x_++){
 *                            *y_ += factAv * *d_ * *x_;
 *                        }
 *                    }
 */
				}else{
					cuvAssert(A_w == v.size());
					cuvAssert(A_h == dst.size());
					for( int i=0; i<A.h();i++ ){ 
						value_type sum = ( value_type ) 0;
						for( int j=0; j<A.w();j++ )
							sum += A( i,j ) * v[j];
						dst.set( i, dst[ i ] + factAv*sum );
					}
/*
 *                    for(index_type i = 0; i < num_diags; i++){
 *                        const int k = offsets[i];  //diagonal offset
 *
 *                        const index_type i_start =  1*std::max((int)0,-k);
 *                        const index_type j_start =  1*std::max((int)0, k);
 *
 *                        //number of elements to process
 *                        const index_type N = std::min(A_h - i_start, (A_w - j_start));
 *
 *                        const value_type * d_ = A.vec().ptr() + i;
 *                        const value_type * x_ = v.ptr() + j_start;
 *                        value_type * y_ = dst.ptr() + i_start;
 *
 *                        for(index_type n = 0; n < N; n++){
 *                            *y_++ += factAv * *d_ * x_[n];
 *                        }
 *                    }
 */
				}
			}
	}

	template<>
		void prod(dense_matrix<float,column_major,host_memory_space>& dst,
				  dia_matrix<float,host_memory_space>&                  A,
				  dense_matrix<float,column_major,host_memory_space>&   B,
				  char transA,
				  char transB,
				  const float& factAB,
				  const float& factC){
			cuvAssert(transB == 'n');
			cuvAssert(dst.w() == B.w());
			for(int i=0;i<dst.w();i++){
				vector<float,host_memory_space> dst_v(dst.h(), dst.vec().ptr()+i*dst.h(), true);
				vector<float,host_memory_space> src_v(B.h(),   B.vec().ptr()+i*B.h(), true);
				spmv(dst_v,A,src_v,transA,factAB,factC);
			}
		}
	template<>
		void prod(dense_matrix<float,column_major,host_memory_space>& dst,
				  toeplitz_matrix<float,host_memory_space>&             A,
				  dense_matrix<float,column_major,host_memory_space>&   B,
				  char transA,
				  char transB,
				  const float& factAB,
				  const float& factC){
			cuvAssert(transB == 'n');
			cuvAssert(dst.w() == B.w());
			for(int i=0;i<dst.w();i++){
				vector<float,host_memory_space> dst_v(dst.h(), dst.vec().ptr()+i*dst.h(), true);
				vector<float,host_memory_space> src_v(B.h(),   B.vec().ptr()+i*B.h(), true);
				spmv(dst_v,A,src_v,transA,factAB,factC);
			}
		}
	template<>
		void prod(dense_matrix<float,column_major,dev_memory_space>& dst,
				  dia_matrix<float,dev_memory_space>&                  A,
				  dense_matrix<float,column_major,dev_memory_space>&   B,
				  char transA,
				  char transB,
				  const float& factAB,
				  const float& factC){
			cuvAssert(transB == 'n');
			cuvAssert(dst.w() == B.w());
			cuvAssert(dst.vec_ptr());
			cuvAssert(A.vec_ptr());
			cuvAssert(B.vec_ptr());
			if(transA=='t'){
				cuvAssert(A.w() == dst.h());
			}else{
				cuvAssert(A.h() == dst.h());
			}
			const int num_at_same_time = min(MAX_NUM_IMGS_AT_ONCE, B.w());
			for(int i=0; i<dst.w(); i += num_at_same_time){
				vector<float,dev_memory_space> dst_v(dst.h() * min(dst.w()-i,num_at_same_time), dst.vec().ptr()+i*dst.h(), true);
				vector<float,dev_memory_space> src_v(B.h()   * min(B.w()-i,  num_at_same_time), B.vec().ptr()+i*B.h(), true);
				spmv(dst_v,A,src_v,transA,factAB,factC);
			}
		}
	template<>
		void prod(dense_matrix<float,column_major,dev_memory_space>& dst,
				  toeplitz_matrix<float,dev_memory_space>&             A,
				  dense_matrix<float,column_major,dev_memory_space>&   B,
				  char transA,
				  char transB,
				  const float& factAB,
				  const float& factC){
			cuvAssert(transB == 'n');
			cuvAssert(dst.w() == B.w());
			cuvAssert(dst.vec_ptr());
			cuvAssert(A.vec_ptr());
			cuvAssert(B.vec_ptr());
			if(transA=='t'){
				cuvAssert(A.w() == dst.h());
			}else{
				cuvAssert(A.h() == dst.h());
			}
			const int num_at_same_time = min(MAX_NUM_IMGS_AT_ONCE, B.w());
			for(int i=0; i<dst.w(); i += num_at_same_time){
				vector<float,dev_memory_space> dst_v(dst.h() * min(dst.w()-i,num_at_same_time), dst.vec().ptr()+i*dst.h(), true);
				vector<float,dev_memory_space> src_v(B.h()   * min(B.w()-i,  num_at_same_time), B.vec().ptr()+i*B.h(), true);
				spmv(dst_v,A,src_v,transA,factAB,factC);
			}
		}
	template<class __matrix_type, class __vector_type>
		void spmv(__vector_type& dst, __matrix_type& A, __vector_type& v, char transA, const float& factAv, const float& factC){
			spmv_impl::spmv(dst,A,v,transA,factAv,factC);
		}
	template void spmv<dia_matrix<float,host_memory_space>, vector<float,host_memory_space> >(vector<float,host_memory_space>&dst, dia_matrix<float,host_memory_space>& A, vector<float,host_memory_space>& v, char, const float&, const float&);
	template void spmv<dia_matrix<float,dev_memory_space>, vector<float,dev_memory_space> >(vector<float,dev_memory_space>&dst, dia_matrix<float,dev_memory_space>& A, vector<float,dev_memory_space>& v, char, const float&, const float&);
	template void spmv<toeplitz_matrix<float,host_memory_space>, vector<float,host_memory_space> >(vector<float,host_memory_space>&dst, toeplitz_matrix<float,host_memory_space>& A, vector<float,host_memory_space>& v, char, const float&, const float&);
	template void spmv<toeplitz_matrix<float,dev_memory_space>, vector<float,dev_memory_space> >(vector<float,dev_memory_space>&dst, toeplitz_matrix<float,dev_memory_space>& A, vector<float,dev_memory_space>& v, char, const float&, const float&);
}
