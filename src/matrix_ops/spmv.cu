#include "hip/hip_runtime.h"
#include <iostream>
#include <host_dia_matrix.hpp>
#include <dev_dia_matrix.hpp>
#include "matrix_ops.hpp"
#include <texture.h>

using namespace std;

// stuff from NVIDIA SDK
#define DIVIDE_INTO(x,y) ((x + y - 1)/y)
#define small_grid_thread_id(void) ((__umul24(blockDim.x, blockIdx.x) + threadIdx.x))
#define large_grid_thread_id(void) ((__umul24(blockDim.x,blockIdx.x + __umul24(blockIdx.y,gridDim.x)) + threadIdx.x))
#define large_grid_thread_num(void) ((__umul24(blockDim.x,gridDim.x + __umul24(blockDim.y,gridDim.y))))


namespace cuv{
	namespace spmv_impl{
		/*
		 *  For a given number of blocks, return a 2D grid large enough to contain them
		 *  FROM NVIDIA SDK
		 */
		dim3 make_large_grid(const unsigned int num_blocks){
			if (num_blocks <= 65535){
				return dim3(num_blocks);
			} else {
				unsigned int side = (unsigned int) ceil(sqrt((double)num_blocks));
				return dim3(side,side);
			}
		}

		dim3 make_large_grid(const unsigned int num_threads, const unsigned int blocksize){
			const unsigned int num_blocks = DIVIDE_INTO(num_threads, blocksize);
			if (num_blocks <= 65535){
				//fits in a 1D grid
				return dim3(num_blocks);
			} else {
				//2D grid is required
				const unsigned int side = (unsigned int) ceil(sqrt((double)num_blocks));
				return dim3(side,side);
			}
		}
		/****************************************************************
		 *   Device Code
		 ****************************************************************/
		template <typename value_type, typename index_type, unsigned int BLOCK_SIZE, bool UseCache>
			__global__ void
			spmv_dia_kernel_trans(const index_type num_rows, 
					const index_type num_cols, 
					const index_type num_diagonals,
					const index_type stride,
					const int        * diagonal_offsets,
					const value_type * values,
					const value_type * v, 
					value_type       * dst)
			{
				__shared__ int        offsets[BLOCK_SIZE];

				const index_type thread_id = large_grid_thread_id();
				const index_type grid_size = large_grid_thread_num();

				// load diagonal offsets into shared memory
				if(threadIdx.x < num_diagonals)
					offsets[threadIdx.x] = diagonal_offsets[threadIdx.x];

				__syncthreads();

				for(index_type col = thread_id; col < num_cols; col += grid_size)
				{
					value_type sum = dst[col];
					index_type offset = 0;

					for(index_type n = 0; n < num_diagonals; n++, offset+=stride)
					{
						const int row = col - offsets[n];

						if(row >= 0 && row < num_rows)
						{
							const value_type A_ij = values[       offset + row];
							sum += A_ij * fetch_x<UseCache>(row, v);
						}
					}
					dst[col] = sum;
				}
			}
		template <typename value_type, typename index_type, unsigned int BLOCK_SIZE, bool UseCache>
			__global__ void
			spmv_dia_kernel2(const index_type num_rows, 
					const index_type num_cols, 
					const index_type num_diagonals,
					const index_type stride,
					const int        * diagonal_offsets,
					const value_type * values,
					const value_type * v, 
					value_type * dst)
			{
				__shared__ int offsets[BLOCK_SIZE];

				const index_type thread_id = large_grid_thread_id();
				const index_type grid_size = large_grid_thread_num();

				// load diagonal offsets into shared memory
				if(threadIdx.x < num_diagonals)
					offsets[threadIdx.x] = diagonal_offsets[threadIdx.x];

				__syncthreads();

				for(index_type row = thread_id; row < num_rows; row += grid_size)
				{
					value_type sum = dst[row];
					index_type offset = row;
					for(index_type n = 0; n < num_diagonals; n++, offset+=stride)
					{
						const int col = row + offsets[n];
						if(col >= 0 && col < num_cols)
						{
							const value_type A_ij = values[       offset];
							sum += A_ij * fetch_x<UseCache>(col, v);
						}
					}
					dst[row] = sum;
				}
			}

		template <typename value_type, typename index_type>
			void spmv_dia_device(const dev_dia_matrix<value_type,index_type>& A, 
					const dev_vector<value_type>& v, 
					dev_vector<value_type>& dst, 
					char transA,
					const value_type& factAv,
					const value_type& factC)
			{

				const unsigned int BLOCK_SIZE = 256;
				if(transA != 't'){
					const dim3 grid = make_large_grid(A.h(),BLOCK_SIZE);
					cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals
					spmv_dia_kernel2<value_type, index_type, BLOCK_SIZE, false> <<<grid, BLOCK_SIZE>>>
						(A.h(), A.w(),  A.num_dia(),  A.stride(),
						 A.get_offsets().ptr(), A.vec()->ptr(),
						 v.ptr(), dst.ptr());
				}else{
					const dim3 grid = make_large_grid(A.w(),BLOCK_SIZE);
					cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals
					spmv_dia_kernel_trans<value_type, index_type, BLOCK_SIZE, false> <<<grid, BLOCK_SIZE>>>
						(A.h(), A.w(),  A.num_dia(),  A.stride(),
						 A.get_offsets().ptr(), A.vec()->ptr(),
						 v.ptr(), dst.ptr());
				}
				cuvSafeCall(hipDeviceSynchronize());
			}

		template <bool transA, typename value_type, typename index_type>
			void spmv_dia_tex_device(const dev_dia_matrix<value_type,index_type>& A, 
					const dev_vector<value_type>& v, 
					dev_vector<value_type>& dst)
			{
				const unsigned int BLOCK_SIZE = 256;
				const dim3 grid = make_large_grid(A.h(),BLOCK_SIZE);

				cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals

				bind_x(v.ptr());

				if(!transA){
					const unsigned int BLOCK_SIZE = 256;
					const dim3 grid = make_large_grid(A.h(),BLOCK_SIZE);
					cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals
					spmv_dia_kernel2<value_type, index_type, BLOCK_SIZE, true> <<<grid, BLOCK_SIZE>>>
						(A.h(), A.w(),  A.num_dia(),  A.stride(),
						 A.get_offsets().ptr(), A.vec()->ptr(),
						 v.ptr(), dst.ptr());
				}else{
					const unsigned int BLOCK_SIZE = 256;
					const dim3 grid = make_large_grid(A.w(),BLOCK_SIZE);
					cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals
					spmv_dia_kernel_trans<value_type, index_type, BLOCK_SIZE, true> <<<grid, BLOCK_SIZE>>>
						(A.h(), A.w(),  A.num_dia(),  A.stride(),
						 A.get_offsets().ptr(), A.vec()->ptr(),
						 v.ptr(), dst.ptr());
				}

				unbind_x(v.ptr());
			}
		template<class value_type, class index_type>
			void spmv(dev_vector<value_type,index_type>& dst, dev_dia_matrix<value_type,index_type>& A, dev_vector<value_type,index_type>& v, char transA, const float& factAv, const float& factC){
				spmv_dia_device(A,v,dst,transA,factAv,factC);
			}


		/****************************************************************
		 *  Host Code
		 ****************************************************************/
		template<class value_type, class index_type>
			void spmv(host_vector<value_type,index_type>& dst, host_dia_matrix<value_type,index_type>& A, host_vector<value_type,index_type>& v, char transA, const float& factAv, const float& factC){
				const host_vector<int>& offsets = A.get_offsets();
				const int num_diags             = A.num_dia();
				const int num_rows              = A.h();
				const int num_cols              = A.w();
				const int stride                = A.stride();
				cuvAssert(!A.transposed());
				if(transA == 't'){
					cuvAssert(num_rows == v.size());
					cuvAssert(num_cols == dst.size());
					for(index_type i = 0; i < num_diags; i++){
						const int k = offsets[i];  //diagonal offset

						const index_type i_start = std::max((int)0, k);
						const index_type j_start = std::max((int)0,-k);

						//number of elements to process
						const index_type N = std::min(num_rows - j_start, num_cols - i_start);

						const value_type * d_ = A.vec()->ptr() + i*stride + j_start;
						const value_type * x_ = v.ptr() + j_start;
						value_type * y_ = dst.ptr() + i_start;

						for(index_type n = 0; n < N; n++){
							y_[n] += d_[n] * x_[n];
						}
					}
				}else{
					cuvAssert(num_cols == v.size());
					cuvAssert(num_rows == dst.size());
					for(index_type i = 0; i < num_diags; i++){
						const int k = offsets[i];  //diagonal offset

						const index_type i_start = std::max((int)0,-k);
						const index_type j_start = std::max((int)0, k);

						//number of elements to process
						const index_type N = std::min(num_rows - i_start, num_cols - j_start);

						const value_type * d_ = A.vec()->ptr() + i*stride + i_start;
						const value_type * x_ = v.ptr() + j_start;
						value_type * y_ = dst.ptr() + i_start;

						for(index_type n = 0; n < N; n++){
							y_[n] += d_[n] * x_[n];
						}
					}
				}
			}
	}

	template<>
		void prod(host_dense_matrix<float,column_major>& dst,
				  host_dia_matrix<float>&                  A,
				  host_dense_matrix<float,column_major>&   B,
				  char transA,
				  char transB,
				  const float& factAB,
				  const float& factC){
			cuvAssert(transB == 'n');
			cuvAssert(dst.w() == B.w());
			for(int i=0;i<dst.w();i++){
				host_vector<float> dst_v(dst.h(), dst.vec().ptr()+i*dst.h(), true);
				host_vector<float> src_v(B.h(),   B.vec().ptr()+i*B.h(), true);
				spmv(dst_v,A,src_v,transA,factAB,factC);
			}
		}
	template<>
		void prod(dev_dense_matrix<float,column_major>& dst,
				  dev_dia_matrix<float>&                  A,
				  dev_dense_matrix<float,column_major>&   B,
				  char transA,
				  char transB,
				  const float& factAB,
				  const float& factC){
			cuvAssert(transB == 'n');
			cuvAssert(dst.w() == B.w());
			for(int i=0;i<dst.w();i++){
				dev_vector<float> dst_v(dst.h(), dst.vec().ptr()+i*dst.h(), true);
				dev_vector<float> src_v(B.h(),   B.vec().ptr()+i*B.h(), true);
				spmv(dst_v,A,src_v,transA,factAB,factC);
			}
		}
	template<class __matrix_type, class __vector_type>
		void spmv(__vector_type& dst, __matrix_type& A, __vector_type& v, char transA, const float& factAv, const float& factC){
			spmv_impl::spmv(dst,A,v,transA,factAv,factC);
		}
	template void spmv<host_dia_matrix<float>, host_vector<float> >(host_vector<float>&dst, host_dia_matrix<float>& A, host_vector<float>& v, char, const float&, const float&);
	template void spmv<dev_dia_matrix<float>, dev_vector<float> >(dev_vector<float>&dst, dev_dia_matrix<float>& A, dev_vector<float>& v, char, const float&, const float&);
}
