#include "hip/hip_runtime.h"
#include <iostream>
#include <host_dia_matrix.hpp>
#include <dev_dia_matrix.hpp>
#include "matrix_ops.hpp"
#include <texture.h>

using namespace std;

// stuff from NVIDIA SDK
#define DIVIDE_INTO(x,y) ((x + y - 1)/y)
#define small_grid_thread_id(void) ((__umul24(blockDim.x, blockIdx.x) + threadIdx.x))
#define large_grid_thread_id(void) ((__umul24(blockDim.x,blockIdx.x + __umul24(blockIdx.y,gridDim.x)) + threadIdx.x))
#define large_grid_thread_num(void) ((__umul24(blockDim.x,gridDim.x + __umul24(blockDim.y,gridDim.y))))


namespace cuv{
	namespace spmv_impl{
		/*
		 *  For a given number of blocks, return a 2D grid large enough to contain them
		 *  FROM NVIDIA SDK
		 */
		dim3 make_large_grid(const unsigned int num_blocks){
			if (num_blocks <= 65535){
				return dim3(num_blocks);
			} else {
				unsigned int side = (unsigned int) ceil(sqrt((double)num_blocks));
				return dim3(side,side);
			}
		}

		dim3 make_large_grid(const unsigned int num_threads, const unsigned int blocksize){
			const unsigned int num_blocks = DIVIDE_INTO(num_threads, blocksize);
			if (num_blocks <= 65535){
				//fits in a 1D grid
				return dim3(num_blocks);
			} else {
				//2D grid is required
				const unsigned int side = (unsigned int) ceil(sqrt((double)num_blocks));
				return dim3(side,side);
			}
		}
		/****************************************************************
		 *   Device Code
		 ****************************************************************/
		template <typename value_type, typename index_type, unsigned int BLOCK_SIZE, bool UseCache, bool wantFactAv, bool wantFactC>
			__global__ void
			spmv_dia_kernel_trans(const index_type A_h, 
					const index_type A_w, 
					const index_type A_nd,
					const index_type A_stride,
					const int        * A_diaoff,
					const value_type * A_data,
					const value_type * v, 
					value_type       * dst,
					const value_type factAv,
					const value_type factC)
			{
				__shared__ int        offsets[BLOCK_SIZE];

				const index_type thread_id = large_grid_thread_id();
				const index_type grid_size = large_grid_thread_num();

				// load diagonal offsets into shared memory
				if(threadIdx.x < A_nd)
					offsets[threadIdx.x] = A_diaoff[threadIdx.x];

				__syncthreads();

				for(index_type col = thread_id; col < A_w; col += grid_size)
				{
					value_type sum = wantFactC ? factC * dst[col] : 0;
					index_type offset = 0;

					for(index_type n = 0; n < A_nd; n++, offset+=A_stride)
					{
						const int row = col - offsets[n];

						if(row >= 0 && row < A_h)
						{
							const value_type A_ij = A_data[       offset + row];
							sum += (wantFactAv ? factAv : 1.f ) * A_ij * fetch_x<UseCache>(row, v);
						}
					}
					dst[col] = sum;
				}
			}
		template <typename value_type, typename index_type, unsigned int BLOCK_SIZE, bool UseCache, bool wantFactAv, bool wantFactC>
			__global__ void
			spmv_dia_kernel(
					const index_type A_h, 
					const index_type A_w, 
					const index_type A_nd,
					const index_type A_stride,
					const int        * A_diaoff,
					const value_type * A_data,
					const value_type * v, 
					value_type       * dst,
					const value_type factAv,
					const value_type factC)
			{
				__shared__ int offsets[BLOCK_SIZE];

				const index_type thread_id = large_grid_thread_id();
				const index_type grid_size = large_grid_thread_num();

				// load diagonal offsets into shared memory
				if(threadIdx.x < A_nd)
					offsets[threadIdx.x] = A_diaoff[threadIdx.x];

				__syncthreads();

				for(index_type row = thread_id; row < A_h; row += grid_size)
				{
					value_type sum = wantFactC ? factC * dst[row] : 0 ;
					index_type offset = row;
					for(index_type n = 0; n < A_nd; n++, offset+=A_stride)
					{
						const int col = row + offsets[n];
						if(col >= 0 && col < A_w)
						{
							const value_type A_ij = A_data[       offset];
							sum += (wantFactAv ? factAv : 1.f) * A_ij * fetch_x<UseCache>(col, v);
						}
					}
					dst[row] = sum;
				}
			}

		template <typename value_type, typename index_type>
			void spmv_dia_device(const dev_dia_matrix<value_type,index_type>& A, 
					const dev_vector<value_type>& v, 
					dev_vector<value_type>& dst, 
					char transA,
					const value_type& factAv,
					const value_type& factC)
			{

				const unsigned int BLOCK_SIZE = 256;
				if(transA != 't'){
					const dim3 grid = make_large_grid(A.h(),BLOCK_SIZE);
					cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals
					if(0);
					else if(factAv==1.f && factC == 0.f)
						spmv_dia_kernel<value_type, index_type, BLOCK_SIZE, false,false,false> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec()->ptr(), v.ptr(), dst.ptr(), factAv,factC);
					else if(factAv==1.f && factC != 0.f)
						spmv_dia_kernel<value_type, index_type, BLOCK_SIZE, false,false,true> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec()->ptr(), v.ptr(), dst.ptr(), factAv,factC);
					else if(factAv!=1.f && factC == 0.f)
						spmv_dia_kernel<value_type, index_type, BLOCK_SIZE, false,true,false> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec()->ptr(), v.ptr(), dst.ptr(), factAv,factC);
					else 
						spmv_dia_kernel<value_type, index_type, BLOCK_SIZE, false,true,true> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec()->ptr(), v.ptr(), dst.ptr(), factAv,factC);
				}else{
					const dim3 grid = make_large_grid(A.w(),BLOCK_SIZE);
					cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals
					if(0);
					else if(factAv==1.f && factC == 0.f)
						spmv_dia_kernel_trans<value_type, index_type, BLOCK_SIZE, false,false,false> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec()->ptr(), v.ptr(), dst.ptr(), factAv,factC);
					else if(factAv==1.f && factC != 0.f)
						spmv_dia_kernel_trans<value_type, index_type, BLOCK_SIZE, false,false,true> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec()->ptr(), v.ptr(), dst.ptr(), factAv,factC);
					else if(factAv!=1.f && factC == 0.f)
						spmv_dia_kernel_trans<value_type, index_type, BLOCK_SIZE, false,true,false> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec()->ptr(), v.ptr(), dst.ptr(), factAv,factC);
					else 
						spmv_dia_kernel_trans<value_type, index_type, BLOCK_SIZE, false,true,true> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec()->ptr(), v.ptr(), dst.ptr(), factAv,factC);
				}
				cuvSafeCall(hipDeviceSynchronize());
			}

		/*template <bool transA, typename value_type, typename index_type>*/
		/*    void spmv_dia_tex_device(const dev_dia_matrix<value_type,index_type>& A, */
		/*            const dev_vector<value_type>& v, */
		/*            dev_vector<value_type>& dst)*/
		/*    {*/
		/*        const unsigned int BLOCK_SIZE = 256;*/
		/*        const dim3 grid = make_large_grid(A.h(),BLOCK_SIZE);*/

		/*        cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals*/

		/*        bind_x(v.ptr());*/

		/*        if(!transA){*/
		/*            const unsigned int BLOCK_SIZE = 256;*/
		/*            const dim3 grid = make_large_grid(A.h(),BLOCK_SIZE);*/
		/*            cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals*/
		/*            spmv_dia_kernel<value_type, index_type, BLOCK_SIZE, true> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec()->ptr(), v.ptr(), dst.ptr());*/
		/*        }else{*/
		/*            const unsigned int BLOCK_SIZE = 256;*/
		/*            const dim3 grid = make_large_grid(A.w(),BLOCK_SIZE);*/
		/*            cuvAssert(A.num_dia() < BLOCK_SIZE); // kernel doesn't handle larger numbers of diagonals*/
		/*            spmv_dia_kernel_trans<value_type, index_type, BLOCK_SIZE, true> <<<grid, BLOCK_SIZE>>> (A.h(), A.w(),  A.num_dia(),  A.stride(), A.get_offsets().ptr(), A.vec()->ptr(), v.ptr(), dst.ptr());*/
		/*        }*/

		/*        unbind_x(v.ptr());*/
		/*    }*/
		template<class value_type, class index_type>
			void spmv(dev_vector<value_type,index_type>& dst, dev_dia_matrix<value_type,index_type>& A, dev_vector<value_type,index_type>& v, char transA, const float& factAv, const float& factC){
				spmv_dia_device(A,v,dst,transA,factAv,factC);
			}


		/****************************************************************
		 *  Host Code
		 ****************************************************************/
		template<class value_type, class index_type>
			void spmv(host_vector<value_type,index_type>& dst, host_dia_matrix<value_type,index_type>& A, host_vector<value_type,index_type>& v, char transA, const float& factAv, const float& factC){
				const host_vector<int>& offsets = A.get_offsets();
				const int num_diags             = A.num_dia();
				const int A_h              = A.h();
				const int A_w              = A.w();
				const int A_stride                = A.stride();
				cuvAssert(!A.transposed());
				index_type max_dst = ((transA=='t') ? A_w : A_h);
				if(factC==0.f)
					for(int i=0;i<max_dst;i++) dst[i] = 0;
				else
					for(int i=0;i<max_dst;i++) dst[i] = dst[i] * factC;
				if(transA == 't'){
					cuvAssert(A_h == v.size());
					cuvAssert(A_w == dst.size());
					for(index_type i = 0; i < num_diags; i++){
						const int k = offsets[i];  //diagonal offset

						const index_type i_start = std::max((int)0, k);
						const index_type j_start = std::max((int)0,-k);

						//number of elements to process
						const index_type N = std::min(A_h - j_start, A_w - i_start);

						const value_type * d_ = A.vec()->ptr() + i*A_stride + j_start;
						const value_type * x_ = v.ptr() + j_start;
						value_type * y_ = dst.ptr() + i_start;

						for(index_type n = 0; n < N; n++){
							y_[n] += factAv * d_[n] * x_[n];
						}
					}
				}else{
					cuvAssert(A_w == v.size());
					cuvAssert(A_h == dst.size());
					for(index_type i = 0; i < num_diags; i++){
						const int k = offsets[i];  //diagonal offset

						const index_type i_start = std::max((int)0,-k);
						const index_type j_start = std::max((int)0, k);

						//number of elements to process
						const index_type N = std::min(A_h - i_start, A_w - j_start);

						const value_type * d_ = A.vec()->ptr() + i*A_stride + i_start;
						const value_type * x_ = v.ptr() + j_start;
						value_type * y_ = dst.ptr() + i_start;

						for(index_type n = 0; n < N; n++){
							y_[n] += factAv * d_[n] * x_[n];
						}
					}
				}
			}
	}

	template<>
		void prod(host_dense_matrix<float,column_major>& dst,
				  host_dia_matrix<float>&                  A,
				  host_dense_matrix<float,column_major>&   B,
				  char transA,
				  char transB,
				  const float& factAB,
				  const float& factC){
			cuvAssert(transB == 'n');
			cuvAssert(dst.w() == B.w());
			for(int i=0;i<dst.w();i++){
				host_vector<float> dst_v(dst.h(), dst.vec().ptr()+i*dst.h(), true);
				host_vector<float> src_v(B.h(),   B.vec().ptr()+i*B.h(), true);
				spmv(dst_v,A,src_v,transA,factAB,factC);
			}
		}
	template<>
		void prod(dev_dense_matrix<float,column_major>& dst,
				  dev_dia_matrix<float>&                  A,
				  dev_dense_matrix<float,column_major>&   B,
				  char transA,
				  char transB,
				  const float& factAB,
				  const float& factC){
			cuvAssert(transB == 'n');
			cuvAssert(dst.w() == B.w());
			for(int i=0;i<dst.w();i++){
				dev_vector<float> dst_v(dst.h(), dst.vec().ptr()+i*dst.h(), true);
				dev_vector<float> src_v(B.h(),   B.vec().ptr()+i*B.h(), true);
				spmv(dst_v,A,src_v,transA,factAB,factC);
			}
		}
	template<class __matrix_type, class __vector_type>
		void spmv(__vector_type& dst, __matrix_type& A, __vector_type& v, char transA, const float& factAv, const float& factC){
			spmv_impl::spmv(dst,A,v,transA,factAv,factC);
		}
	template void spmv<host_dia_matrix<float>, host_vector<float> >(host_vector<float>&dst, host_dia_matrix<float>& A, host_vector<float>& v, char, const float&, const float&);
	template void spmv<dev_dia_matrix<float>, dev_vector<float> >(dev_vector<float>&dst, dev_dia_matrix<float>& A, dev_vector<float>& v, char, const float&, const float&);
}
