#include "hip/hip_runtime.h"
//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*


#include <stdexcept>
#include <hipblas.h>
#include <cblas.h>
#include <stdio.h>

#include <cuv_general.hpp>
#include <3rd_party/CudaConv/nvmatrix.cuh>
#include <thrust/functional.h>
#include <float.h>
#include "matrix_ops.hpp"
#include <limits>
#include <vector_ops/functors.hpp>

#ifdef __CDT_PARSER__
#define __global__
#define __shared__
#endif

#define CVT_TRANSPOSE(c) \
	(CBLAS_TRANSPOSE)(((c) == 'N' || (c) == 'n') ? CblasNoTrans : \
	 ((c) == 'T' || (c) == 't') ? CblasTrans : \
	 ((c) == 'C' || (c) == 'c') ? CblasConjTrans : \
	 -1)
	/* (mg)the idea is to place the blocks under each other starting at the upper left in the matrix. their threads
	 * add up multiples of their x position (indicated by ty - see above) in shared memory. then we have a 2-dim
	 * array in shared memory that corresponds in size to the block.
	 *
	 * in each block we now have to add up the elements of each row in the shared memory to get the final value. this is done
	 * by logarithmically adding up the elements:
	 * in the first step the second half (in x direction) of the blocks add their values to the first halfs memory locations in
	 * shared memory - then this is repeated for the first half of the threads. a short example for blockDim.y=8 and blockDim.x=1:
	 *
	 * 1st iter(offset=4): a1 a2 a3 a4 | b1 b2 b3 b4
	 *
	 * 2nd iter(offset=2): (a1+b1) (a2+b2) | (a3+b3) (a4+b4)
	 *
	 * 3rd iter(offset=1): ( (a1+b1) + (a3+b3)  ) | ( (a2+b2) +  (a4+b4) )
	 *
	 * 4th iter(offset=0): ( (a1+b1) + (a3+b3)  ) + ( (a2+b2) +  (a4+b4) )
	 *
	 * tx indicates the y-dimension in the matrix; ty indicates the x-dimension in the matrix
	 */

// "coalesced transpose" with no bank conflicts, example from SDK
// potential speedup by 5 possible for "fine-grained transpose"
template<int BLOCK_SIZE, class T>
__global__
void transpose_kernel(T* dst, const T* src, int width, int height) {
	const int bx = blockIdx.x * blockDim.x;
	const int by = blockIdx.y * blockDim.y;
	const int tx = bx + threadIdx.x;
	const int ty = by + threadIdx.y;

	__shared__
	T shared[BLOCK_SIZE][BLOCK_SIZE + 1];

	if (tx < width && ty < height) {
		shared[threadIdx.y][threadIdx.x] = src[ty * width + tx];
	}
	__syncthreads();

	if (by + threadIdx.x < height && threadIdx.y + bx < width) {
		dst[(bx + threadIdx.y) * height + by + threadIdx.x]
				= shared[threadIdx.x][threadIdx.y];
	}
}
namespace cuv {
template<class __value_type, class __memory_space_type, class __index_type>
dense_matrix<__value_type , column_major, __memory_space_type, __index_type >*blockview(
		dense_matrix<__value_type,column_major,__memory_space_type,__index_type>& matrix,
				__index_type start_rows,
				__index_type num_rows,
				__index_type start_cols,
				__index_type num_cols,
				column_major
				) {
			cuvAssert(start_rows==0);
			cuvAssert(num_rows==matrix.h())
			return new dense_matrix<__value_type,column_major,__memory_space_type,__index_type>(num_rows,num_cols, matrix.ptr()+matrix.h()*start_cols,true);
		}

template<class __value_type, class __memory_space_type, class __index_type>
dense_matrix<__value_type,row_major,__memory_space_type,__index_type>* blockview(
		dense_matrix<__value_type,row_major,__memory_space_type,__index_type>& matrix,
		__index_type start_rows,
		__index_type num_rows,
		__index_type start_cols,
		__index_type num_cols,
		row_major
) {
	cuvAssert(start_cols==0);
	cuvAssert(num_cols==matrix.w())
	return new dense_matrix<__value_type,row_major,__memory_space_type,__index_type>(num_rows,num_cols, matrix.ptr()+matrix.w()*start_rows,true);
}
template<class __value_type, class __memory_layout, class __memory_space_type, class __index_type>
dense_matrix<__value_type,__memory_layout,__memory_space_type,__index_type>* blockview(
		dense_matrix<__value_type,__memory_layout,__memory_space_type,__index_type> & matrix,
		__index_type start_rows,
		__index_type num_rows ,
		__index_type start_cols,
		__index_type num_cols) {
	return blockview(matrix,start_rows,num_rows,start_cols,num_cols, __memory_layout());
}

__global__ void bitflip_kernel(float* M, int height, int row, int n) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int off = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i < n; i += off){
		M[i * height + row] = 1 - M[i * height + row];
	}

}

namespace bitflip_row_impl{
	template<class V, class I>
	void bitflip(dense_matrix<V,column_major,dev_memory_space,I>& m, const I& row){
		int num_threads = (int) min(512.f, ceil((float)sqrt(m.w())));
		int num_blocks  = (int) ceil((float)m.w()/num_threads);
		bitflip_kernel<<<num_blocks,num_threads>>>(m.ptr(),m.h(),row, m.w());
		cuvSafeCall(hipDeviceSynchronize());
	}
	template<class V, class I>
	void bitflip(dense_matrix<V,column_major,host_memory_space,I>& m, const I& row){
		for(int i=0;i<m.w();i++)
			m.set(row,i,(V)(1.f-m(row,i)));
	}
}
// bitflip a row of a column-major matrix
template<class __value_type, class __memory_layout, class __memory_space_type, class __index_type>
void bitflip(dense_matrix<__value_type,__memory_layout,__memory_space_type,__index_type>& matrix,
		__index_type row){
		assert(row<matrix.h());
		assert(matrix.ptr());
		bitflip_row_impl::bitflip(matrix,row);
}

/// column major blas3
template<>
void prod(dense_matrix<float,column_major,dev_memory_space>& dst,
		dense_matrix<float,column_major,dev_memory_space>& A,
		dense_matrix<float,column_major,dev_memory_space>& B,
		char transA,
		char transB,
		const float& factAB,
		const float& factC) {
	int m = (transA=='t' ? A.w() : A.h());
	int k1 = (transA=='t' ? A.h() : A.w());
	int k2 = (transB=='t' ? B.w() : B.h());
	int n = (transB=='t' ? B.h() : B.w());

	cuvAssert(dst.h() == m);
	cuvAssert(dst.w() == n);
	cuvAssert(k1 == k2);
	cuvAssert(A.ptr());
	cuvAssert(B.ptr());
	cuvAssert(dst.ptr());

	hipblasSgemm(transA, transB, m, n, k1, factAB, A.ptr(), A.h(),B.ptr(), B.h(), factC, dst.ptr(), dst.h());
	cuvAssert( cublasGetError() == HIPBLAS_STATUS_SUCCESS );
	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void prod(dense_matrix<float,column_major,host_memory_space>& dst,
		dense_matrix<float,column_major,host_memory_space>& A,
		dense_matrix<float,column_major,host_memory_space>& B,
		char transA,
		char transB,
		const float& factAB,
		const float& factC) {
	int m = (transA=='t' ? A.w() : A.h());
	int k1 = (transA=='t' ? A.h() : A.w());
	int k2 = (transB=='t' ? B.w() : B.h());
	int n = (transB=='t' ? B.h() : B.w());

	cuvAssert(dst.h() == m);
	cuvAssert(dst.w() == n);
	cuvAssert(k1 == k2);
	cuvAssert(A.ptr() != NULL);
	cuvAssert(B.ptr() != NULL);
	cuvAssert(dst.ptr());

#if 1 /* CBLAS */
	cblas_sgemm(
			CblasColMajor,
			CVT_TRANSPOSE(transA),
			CVT_TRANSPOSE(transB), m, n, k1,
			factAB, A.ptr(), A.h(),B.ptr(), B.h(), factC, dst.ptr(), dst.h());
#else /* naive */
	for(int i=0; i<A.h();i++)
	for(int j=0; j<B.w(); j++) {
		float f=0;
		for(int k=0;k<A.w();k++) {
			f += A(i,k)*B(k,j);
		}
		dst.set(i,j,f);
	}
#endif
}
/// row major blas3
template<>
void prod(dense_matrix<float,row_major,dev_memory_space>& dst,
		dense_matrix<float,row_major,dev_memory_space>& A,
		dense_matrix<float,row_major,dev_memory_space>& B,
		char transA,
		char transB,
		const float& factAB,
		const float& factC) {
	// we use column major prod and just exchange width and height
	int m = (transB=='t' ? B.h() : B.w());
	int k1 = (transB=='t' ? B.w() : B.h());
	int k2 = (transA=='t' ? A.h() : A.w());
	int n = (transA=='t' ? A.w() : A.h());

	cuvAssert(dst.h() == n);
	cuvAssert(dst.w() == m);
	cuvAssert(k1 == k2);
	cuvAssert(A.ptr());
	cuvAssert(B.ptr());
	cuvAssert(dst.ptr());
	hipblasSgemm(transB, transA, m, n, k1, factAB, B.ptr(), B.w(),A.ptr(), A.w(), factC, dst.ptr(), dst.w());

	cuvAssert( cublasGetError() == HIPBLAS_STATUS_SUCCESS );
	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void prod(dense_matrix<float,row_major,host_memory_space>& dst,
		dense_matrix<float,row_major,host_memory_space>& A,
		dense_matrix<float,row_major,host_memory_space>& B,
		char transA,
		char transB,
		const float& factAB,
		const float& factC) {
	int m = (transA=='t' ? A.w() : A.h());
	int k1 = (transA=='t' ? A.h() : A.w());
	int k2 = (transB=='t' ? B.w() : B.h());
	int n = (transB=='t' ? B.h() : B.w());

	cuvAssert(dst.h() == m);
	cuvAssert(dst.w() == n);
	cuvAssert(k1 == k2);
	cuvAssert(A.ptr() != NULL);
	cuvAssert(B.ptr() != NULL);
	cuvAssert(dst.ptr());

	cblas_sgemm(
			CblasRowMajor,
			CVT_TRANSPOSE(transA),
			CVT_TRANSPOSE(transB), m, n, k1,
			factAB, A.ptr(), A.w(),B.ptr(), B.w(), factC, dst.ptr(), dst.w());
}

template<class V, class I, class V2, class OP>
__global__
void matrix_plus_vector_kernel_column_major(V*A,V2* v,I w,I h, OP op) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if(tid>h) return;
	V2 tid_v = v[tid];
	for(int i=tid;i<w;i++)
	A[i] = op(A[i],tid_v);
}
template<class V, class I, class V2, class OP>
__global__
void matrix_plus_vector_kernel_column_major2 (V *A, const V2* v, I h, I w, OP op) {
	const unsigned int idx = __mul24(blockIdx.x , blockDim.x) + threadIdx.x;
	const unsigned int numThreads = __mul24(blockDim.x , gridDim.x);

	int stop = w*h;
	for (unsigned int i = idx; i < stop; i += numThreads)
	A[i] = op(A[i] , v[i % h]);
}
template<class V, class I, class V2, class OP>
__global__
void matrix_plus_vector_kernel_row_major (V *A, V2* v, I h, I w, OP op) {
	__shared__ V scalar;
	for(unsigned int baseidx = blockIdx.x; baseidx < h; baseidx += gridDim.x) {
		if (threadIdx.x == 0) {
			scalar = (V) v[baseidx];
		}
		__syncthreads();
		for (unsigned int i = threadIdx.x; i < w; i += blockDim.x) {
			const unsigned int k = baseidx * w + i;
			A[k] = op(A[k] , scalar);
		}
		__syncthreads(); // necessary, otherwise the threads use different values of scalar!
	}
}

namespace matrix_plus_vector_impl {
	template<class V, class I, class V2, class OP>
	void matrix_plus_col(dense_matrix<V,row_major,dev_memory_space,I>& A, const vector<V2,dev_memory_space,I>& v, const OP& op) {
		cuvAssert(A.h() == v.size());
		const unsigned int num_threads = min(512,A.w());
		const unsigned int num_blocks  = min(1024,A.h());
		matrix_plus_vector_kernel_row_major<<<num_blocks,num_threads>>>(A.ptr(), v.ptr(), A.h(), A.w(), op);
		cuvSafeCall(hipDeviceSynchronize());
	}
	template<class V, class I, class V2, class OP>
	void matrix_plus_col(dense_matrix<V,column_major,dev_memory_space,I>& A, const vector<V2,dev_memory_space,I>& v, const OP& op) {
		cuvAssert(A.h() == v.size());
		const unsigned int num_threads = 512;
		const unsigned int num_blocks  = min(512,(int)ceil((float)A.n() / num_threads));
		matrix_plus_vector_kernel_column_major2<<<num_blocks,num_threads>>>(A.ptr(), v.ptr(), A.h(), A.w(), op);
		cuvSafeCall(hipDeviceSynchronize());
	}
	template<class V, class I, class V2, class OP>
	void matrix_plus_col(dense_matrix<V,column_major,host_memory_space,I>& A, const vector<V2,host_memory_space,I>& v, const OP& op) {
		cuvAssert(A.h() == v.size());
		const V2* v_ptr = v.ptr();
		V * A_ptr = A.ptr();
		for(int j=0;j<A.w();j++) {
			v_ptr = v.ptr();
			for(int i=0;i<A.h();i++,A_ptr++,v_ptr++)
			*A_ptr = op(*A_ptr,*v_ptr);
		}
	}
	template<class V, class I, class V2, class OP>
	void matrix_plus_col(dense_matrix<V,row_major,host_memory_space,I>& A, const vector<V2,host_memory_space,I>& v, const OP& op) {
		cuvAssert(A.h() == v.size());
		const V2* v_ptr = v.ptr();
		V * A_ptr = A.ptr();
		for(int i=0;i<A.h();i++, v_ptr++) {
			for(int j=0;j<A.w();j++)
			*A_ptr++ = op(*A_ptr,*v_ptr);
		}
	}
	// ====================  row ======================
	template<class V, class I, class V2, class OP>
	void matrix_plus_row(dense_matrix<V,column_major,dev_memory_space,I>& A, const vector<V2,dev_memory_space,I>& v, const OP& op) {
		cuvAssert(A.w() == v.size());
		typedef dense_matrix<V,row_major,dev_memory_space, I> other;
		other o(A.w(), A.h(), A.ptr(), true);
		matrix_plus_col(o,v,op);
	}
	template<class V, class I, class V2, class OP>
	void matrix_plus_row(dense_matrix<V,row_major,dev_memory_space,I>& A, const vector<V2,dev_memory_space,I>& v, const OP& op) {
		cuvAssert(A.w() == v.size());
		typedef dense_matrix<V,column_major,dev_memory_space, I> other;
		other o(A.w(), A.h(), A.ptr(), true);
		matrix_plus_col(o,v,op);
	}
	template<class V, class I, class V2, class OP>
	void matrix_plus_row(dense_matrix<V,row_major,host_memory_space,I>& A, const vector<V2,host_memory_space,I>& v, const OP& op) {
		cuvAssert(A.w() == v.size());
		typedef dense_matrix<V,column_major,host_memory_space, I> other;
		other o(A.w(), A.h(), A.ptr(), true);
		matrix_plus_col(o,v,op);
	}
	template<class V, class I, class V2, class OP>
	void matrix_plus_row(dense_matrix<V,column_major,host_memory_space,I>& A, const vector<V2,host_memory_space,I>& v, const OP& op) {
		cuvAssert(A.w() == v.size());
		typedef dense_matrix<V,row_major,host_memory_space, I> other;
		other o(A.w(), A.h(), A.ptr(), true);
		matrix_plus_col(o,v,op);
	}
}

// ====================  col ======================
template<class __matrix_type, class __vector_type>
void matrix_plus_col(__matrix_type& A, const __vector_type& v) {
	matrix_plus_vector_impl::matrix_plus_col(A,v, thrust::plus<typename __matrix_type::value_type>());
}
template<class __matrix_type, class __vector_type>
void matrix_times_col(__matrix_type& A, const __vector_type& v) {
	matrix_plus_vector_impl::matrix_plus_col(A,v, thrust::multiplies<typename __matrix_type::value_type>());
}
template<class __matrix_type, class __vector_type>
void matrix_divide_col(__matrix_type& A, const __vector_type& v) {
	matrix_plus_vector_impl::matrix_plus_col(A,v, thrust::divides<typename __matrix_type::value_type>());
}
// ====================  row ======================
template<class __matrix_type, class __vector_type>
void matrix_plus_row(__matrix_type& A, const __vector_type& v) {
	matrix_plus_vector_impl::matrix_plus_row(A,v, thrust::plus<typename __matrix_type::value_type>());
}
template<class __matrix_type, class __vector_type>
void matrix_times_row(__matrix_type& A, const __vector_type& v) {
	matrix_plus_vector_impl::matrix_plus_row(A,v, thrust::multiplies<typename __matrix_type::value_type>());
}
template<class __matrix_type, class __vector_type>
void matrix_divide_row(__matrix_type& A, const __vector_type& v) {
	matrix_plus_vector_impl::matrix_plus_row(A,v, thrust::divides<typename __matrix_type::value_type>());
}

namespace transpose_impl{
	template<class V, class I>
	void transpose(dense_matrix<V,column_major, dev_memory_space, I>& dst,
			 const dense_matrix<V,column_major, dev_memory_space, I>& src) {
		cuvAssert(dst.w() == src.h());
		cuvAssert(dst.h() == src.w());
		const I width = dst.w();
		const I height = dst.h();
		static const int BLOCK_SIZE = 16;
		const int numBlocksX = ceil((float)width / BLOCK_SIZE);
		const int numBlocksY = ceil((float)height / BLOCK_SIZE);
		dim3 gridSize(numBlocksX, numBlocksY, 1);
		dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
		transpose_kernel<BLOCK_SIZE><<<gridSize, blockSize>>>(dst.ptr(), src.ptr(), width, height);
		cuvSafeCall(hipDeviceSynchronize());
	}

	template<class V, class I>
	void transpose(dense_matrix<V,row_major,dev_memory_space, I>& dst,
			 const dense_matrix<V,row_major,dev_memory_space, I>& src) {
		cuvAssert(dst.w() == src.h());
		cuvAssert(dst.h() == src.w());
		const I width = dst.h();
		const I height = dst.w();
		static const int BLOCK_SIZE = 16;
		const int numBlocksX = ceil((float)width / BLOCK_SIZE);
		const int numBlocksY = ceil((float)height / BLOCK_SIZE);
		dim3 gridSize(numBlocksX, numBlocksY, 1);
		dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
		transpose_kernel<BLOCK_SIZE><<<gridSize, blockSize>>>(dst.ptr(), src.ptr(), width, height);
		cuvSafeCall(hipDeviceSynchronize());
	}

	template<class V, class I>
	void transpose(dense_matrix<V,column_major,host_memory_space, I>& dst,
			 const dense_matrix<V,column_major,host_memory_space, I>& src) {
		cuvAssert(dst.w() == src.h());
		cuvAssert(dst.h() == src.w());
		V* dst_ptr = dst.ptr();
		const V* src_ptr = src.ptr();
		for(int i=0; i<dst.w(); i++) {
			for(int j=0; j<dst.h(); j++) {
				*dst_ptr++ = src_ptr[j*src.h()];
			}
			src_ptr++;
		}
	}

	template<class V, class I>
	void transpose(dense_matrix<V,row_major,host_memory_space, I>& dst,
			 const dense_matrix<V,row_major,host_memory_space, I>& src) {
		cuvAssert(dst.w() == src.h());
		cuvAssert(dst.h() == src.w());
		V* dst_ptr = dst.ptr();
		const V* src_ptr = src.ptr();
		for(int i=0; i<dst.h(); i++) {
			for(int j=0; j<dst.w(); j++) {
				*dst_ptr++ = src_ptr[j*src.w()];
			}
			src_ptr++;
		}
	}
} // namespace transpose_impl

template<class M>
void transpose(M& dst, const M& src){
	transpose_impl::transpose(dst,src);
}

template<class V, class T, class I>
cuv::dense_matrix<V,row_major,T,I>* transposed_view(cuv::dense_matrix<V,column_major,T,I>&  src){
	return new dense_matrix<V,row_major,T,I>(src.w(),src.h(),src.ptr(),true);
}

template<class V, class T, class I>
cuv::dense_matrix<V,column_major,T,I>* transposed_view(cuv::dense_matrix<V,row_major,T,I>&  src){
	return new dense_matrix<V,column_major,T,I>(src.w(),src.h(),src.ptr(),true);
}

#define INSTANTIATE_MV(V1,V2,M) \
  template void matrix_plus_col(dense_matrix<V1,M,dev_memory_space>&, const vector<V2,dev_memory_space>&);   \
  template void matrix_plus_col(dense_matrix<V1,M,host_memory_space>&, const vector<V2,host_memory_space>&); \
  template void matrix_times_col(dense_matrix<V1,M,dev_memory_space>&, const vector<V2,dev_memory_space>&);  \
  template void matrix_times_col(dense_matrix<V1,M,host_memory_space>&, const vector<V2,host_memory_space>&); \
  template void matrix_divide_col(dense_matrix<V1,M,dev_memory_space>&, const vector<V2,dev_memory_space>&);  \
  template void matrix_divide_col(dense_matrix<V1,M,host_memory_space>&, const vector<V2,host_memory_space>&); \
  template void matrix_plus_row(dense_matrix<V1,M,dev_memory_space>&, const vector<V2,dev_memory_space>&);   \
  template void matrix_plus_row(dense_matrix<V1,M,host_memory_space>&, const vector<V2,host_memory_space>&); \
  template void matrix_times_row(dense_matrix<V1,M,dev_memory_space>&, const vector<V2,dev_memory_space>&);  \
  template void matrix_times_row(dense_matrix<V1,M,host_memory_space>&, const vector<V2,host_memory_space>&); \
  template void matrix_divide_row(dense_matrix<V1,M,dev_memory_space>&, const vector<V2,dev_memory_space>&);  \
  template void matrix_divide_row(dense_matrix<V1,M,host_memory_space>&, const vector<V2,host_memory_space>&);


#define INSTANTIATE_BLOCKVIEW(V,M,I) \
  template dense_matrix<V,M,host_memory_space,I>* blockview(dense_matrix<V,M,host_memory_space,I>&,I,I,I,I); \
  template dense_matrix<V,M, dev_memory_space,I>* blockview(dense_matrix<V,M, dev_memory_space,I>&,I,I,I,I);

#define INSTANTIATE_TRANSPOSE(V,M,I) \
  template void transpose(dense_matrix<V,M,host_memory_space,I>&,const dense_matrix<V,M,host_memory_space,I>&); \
  template void transpose(dense_matrix<V,M,dev_memory_space,I>&,const dense_matrix<V,M,dev_memory_space,I>&); 

#define INSTANTIATE_TRANSPOSED_VIEW(V,I) \
  template dense_matrix<V,row_major,host_memory_space,I>* transposed_view(dense_matrix<V,column_major,host_memory_space,I>&);\
  template dense_matrix<V,column_major,host_memory_space,I>* transposed_view(dense_matrix<V,row_major,host_memory_space,I>&);\
  template dense_matrix<V,row_major,dev_memory_space,I>* transposed_view(dense_matrix<V,column_major,dev_memory_space,I>&);\
  template dense_matrix<V,column_major,dev_memory_space,I>* transposed_view(dense_matrix<V,row_major,dev_memory_space,I>&);

INSTANTIATE_TRANSPOSE(float,column_major,unsigned int);
INSTANTIATE_TRANSPOSE(float,row_major,unsigned int);
INSTANTIATE_TRANSPOSE(int,column_major,unsigned int);
INSTANTIATE_TRANSPOSE(int,row_major,unsigned int);

INSTANTIATE_TRANSPOSED_VIEW(float,unsigned int);
/*INSTANTIATE_TRANSPOSED_VIEW(int,unsigned int);*/
/*INSTANTIATE_TRANSPOSED_VIEW(unsigned int,unsigned int);*/
/*INSTANTIATE_TRANSPOSED_VIEW(char,unsigned int);*/
/*INSTANTIATE_TRANSPOSED_VIEW(unsigned char,unsigned int);*/

INSTANTIATE_MV(float, float, column_major);
INSTANTIATE_MV(float, float, row_major);
INSTANTIATE_MV(float, unsigned char, column_major);
INSTANTIATE_MV(float, unsigned char, row_major);

INSTANTIATE_BLOCKVIEW(float,column_major,unsigned int);
INSTANTIATE_BLOCKVIEW(float,row_major,unsigned int);

template void bitflip(dense_matrix<float,column_major,host_memory_space>&, unsigned int);
template void bitflip(dense_matrix<float,column_major,dev_memory_space>&, unsigned int);

}; // cuv
