#include <iostream>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>

#include <thrust/device_ptr.h>

#include <cuv_general.hpp>

#include "dev_vector.hpp"

namespace cuv{

template<class V,class I>
V dev_vector<V,I>::operator[](size_t idx)const{
	const thrust::device_ptr<const value_type> ptr(this->ptr());
	return (value_type) *(ptr+idx);
}
template<class V,class I>
void
dev_vector<V,I>::alloc(){
		cuvSafeCall(hipMalloc( (void**)& this->m_ptr, this->memsize() ));
}

template<class V,class I>
void
dev_vector<V,I>::dealloc(){
	  if(this->m_ptr && ! this->m_is_view){
		  cuvSafeCall(hipFree(this->m_ptr));
			this->m_ptr = NULL;
		}
}

template<class V,class I>
void
dev_vector<V,I>::set(const index_type& i, const value_type& val) {
	thrust::device_ptr<value_type> ptr(this->ptr());
	ptr[i] = val;
}

template class dev_vector<int>;
template class dev_vector<float>;
template class dev_vector<unsigned char>;
template class dev_vector<signed char>;

}
