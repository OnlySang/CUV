#include <iostream>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>
#include <cuv_general.hpp>

#include "dev_vector.hpp"

namespace cuv{

template<class V,class I>
void
dev_vector<V,I>::alloc(){
		cuvSafeCall(hipMalloc( (void**)& this->m_ptr, this->memsize() ));
}

template<class V,class I>
void
dev_vector<V,I>::dealloc(){
	  if(this->m_ptr && ! this->m_is_view){
		  cuvSafeCall(hipFree(this->m_ptr));
			this->m_ptr = NULL;
		}
}

template class dev_vector<float>;
template class dev_vector<unsigned char>;

}
