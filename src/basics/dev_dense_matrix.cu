#include <cuv_general.hpp>

#include <dev_dense_matrix.hpp>

#define DDM0 template<class __value_type, class __mem_layout, class __index_type> \
	dev_dense_matrix<__value_type,__mem_layout, __index_type>
#define DDM(RETURN_TYPE) template<class __value_type, class __mem_layout, class __index_type> \
	RETURN_TYPE dev_dense_matrix<__value_type,__mem_layout, __index_type>



namespace cuv{

	/*
	 * Life Cycle
	 *
	 */
	DDM0::dev_dense_matrix(const index_type& h, const index_type& w)
	: base_type(h,w) { alloc(); }

	DDM0 ::dev_dense_matrix(const index_type& h, const index_type& w, value_type* p, const bool& is_view)
	: base_type(h,w,p,is_view) {}

	template<class __value_type, class __mem_layout, class __index_type> 
	void dev_dense_matrix<__value_type,__mem_layout, __index_type>::alloc(){
	/*DDM(void)::alloc(){*/
		hipMalloc( (void**)& this->m_ptr, this->memsize() );
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) throw std::runtime_error("Device memory allocation failed");
		checkCudaError("allocation");
	}

	DDM0 ::~dev_dense_matrix(){
	  dealloc();
	}
	DDM(void)::dealloc(){
	  if(this->m_ptr && ! this->m_is_view){
		  cutilSafeCall(hipFree(this->m_ptr));
			this->m_ptr = NULL;
		}
	}

}
